#include "hip/hip_runtime.h"
include "hip/hip_runtime.h"
#include ""
#include "math.h"


__global__ void VMF_GPU_GLOBAL(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float  valAngulo = 0.0, r = 0.0;

	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		disteucl = 0;
		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 8; x++) {
				disteucl += abs(vectB[F] - vectB[x]) + abs(vectG[F] - vectG[x]) + abs(vectR[F] - vectR[x]);
				//disteucl += sqrt(pow(vectB[F] - vectB[x], 2) + pow(vectG[F] - vectG[x], 2) + pow(vectR[F] - vectR[x], 2));
				//disteucl +=  (vectB[F]-vectB[x]) * (vectB[F]-vectB[x]) + (vectG[F]-vectG[x]) * (vectG[F]-vectG[x])+(vectR[F]-vectR[x]) * (vectR[F]-vectR[x]);
				//disteucl +=  (vectB[F]-vectB[x]);
				//disteucl += (vectB[F] - vectB[x])*(vectB[F] - vectB[x]) + (vectG[F] - vectG[x])*(vectG[F] - vectG[x]) + (vectR[F] - vectR[x])*(vectR[F] - vectR[x]);

			}
			disteucl1[F] = disteucl;
			disteucl = 0;
		}
		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 7; x++) {
				if (disteucl1[x] > disteucl1[x + 1]) {
					hold = disteucl1[x];
					hold2 = posicion[x];
					disteucl1[x] = disteucl1[x + 1];
					posicion[x] = posicion[x + 1];
					disteucl1[x + 1] = hold;
					posicion[x + 1] = hold2;
				}
			}
		}
		d_Pout[(Row * n + Col) * 3 + 0] = vectR[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 1] = vectG[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 2] = vectB[posicion[0]];
	}
}


#define maxCUDA( a, b ) ( ((a) > (b)) ? (a) : (b) )
#define minCUDA( a, b ) ( ((a) < (b)) ? (a) : (b) )

__global__ void Detection_FuzzyMetric(unsigned char* Noise, const unsigned char* d_Pin, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1;
	const float d = .95;

	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {
			arriva = minCUDA(vectR[F], vectR[4]) + K;
			abajo = maxCUDA(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = minCUDA(vectG[F], vectG[4]) + K;
			abajo = maxCUDA(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = minCUDA(vectB[F], vectB[4]) + K;
			abajo = maxCUDA(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;

			dist_M = minCUDA(minCUDA(val1, val2), val3);
			if (dist_M>d)	P++;
		}

		if (P <= (q + 1)) {
			Noise[(Row * m + Col)] = 255;
		}
		else {
			Noise[(Row * m + Col)] = 0;
		}

	}
}

__device__ float Magnitud(unsigned char* VectR, unsigned char* VectG, unsigned char* VectB, unsigned int i, unsigned int j) {

	float distR = abs(VectR[i] - VectR[j]);
	float distG = abs(VectG[i] - VectG[j]);
	float distB = abs(VectB[i] - VectB[j]);

	return distR + distB + distG;

}
__global__ void Detection_Euclidean(unsigned char* Noise, const unsigned char* d_Pin, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0;
	unsigned int F = 0;
	unsigned char vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1, dEuclidiana = 45;
	const float d = .95;

	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {

			dist_M = Magnitud(vectR, vectG, vectB, F, 4);
			if (dist_M>45)	P++;
		}

		if (P <= (q + 1)) {
			Noise[(Row * m + Col)] = 255;
		}
		else {
			Noise[(Row * m + Col)] = 0;
		}

	}
}



__global__ void AMF_Filtering(unsigned char* d_Pout, const unsigned char* d_Pin, unsigned char* Noise, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float sumR = 0.0, sumG = 0.0, sumB = 0.0;
	unsigned int Div = 0;


	if ( (Row>1) && (Col>1) && ( Row < m - 1) && (Col < n - 1) ) {
		sumR = 0.0, sumG = 0.0, sumB = 0.0;
		
		if (Noise[(Row * m + Col)] == 255) {
			Div = 0;
			
			for (int i = -1; i <= 1; i++) {
				for (int j = -1; j <= 1; j++) {
					
					if (Noise[((Row + i) * m + (Col + j))] == 0) {//solo los que no son Noise
						
						Div++;
						sumR += d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
						sumG += d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
						sumB += d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];
						
					}
					
					
					
				}
			}


			d_Pout[((Row*m) + Col) * 3 + 0] = sumR / Div;
			d_Pout[((Row*m) + Col) * 3 + 1] = sumG / Div;
			d_Pout[((Row*m) + Col) * 3 + 2] = sumB / Div;

		}//fin de if
		else {
			d_Pout[((Row*m) + Col) * 3 + 0] = d_Pin[((Row*m) + Col) * 3 + 0];
			d_Pout[((Row*m) + Col) * 3 + 1] = d_Pin[((Row*m) + Col) * 3 + 1];
			d_Pout[((Row*m) + Col) * 3 + 2] = d_Pin[((Row*m) + Col) * 3 + 2];

		}
		
	}
}

__global__ void VMF_Filtering(unsigned char* d_Pout, const unsigned char* d_Pin, unsigned char* Noise, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	unsigned char arrayFiltradoR[9], arrayFiltradoG[9], arrayFiltradoB[9];
	float mn, mx;
	int posMin = 0;

	int c = 0, i = 0, j = 0;
	unsigned char aux = 100;

	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {
		if (Noise[(Row * m + Col)] == 255) {
			c = 0;
			F = 0;
			for (i = -1; i <= 1; i++) {
				for (j = -1; j <= 1; j++) {
					posicion[F] = 0;

					if (Noise[((Row + i) * m + (Col + j))] == 0) {//solo los que no son Noise

						arrayFiltradoR[c] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
						arrayFiltradoG[c] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
						arrayFiltradoB[c] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];
						aux = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];
						posicion[c] = c;
						c++;
					}
					F++;
				}
			}
			disteucl = 0;
			for (i = 0; i <= c - 1; i++) {
				disteucl = 0;
				for (j = 0; j <= c - 1; j++) {
					float distR = abs(arrayFiltradoR[i] - arrayFiltradoR[j]);
					float distG = abs(arrayFiltradoG[i] - arrayFiltradoG[j]);
					float distB = abs(arrayFiltradoB[i] - arrayFiltradoB[j]);
					disteucl += distR + distB + distG;

				}
				disteucl1[i] = disteucl;
			}
			mn = disteucl1[0];
			mx = disteucl1[0];
			posMin = 0;

			for (i = 0; i <= c - 1; i++)
			{
				if (mn>disteucl1[i])
				{
					mn = disteucl1[i];
					posMin = posicion[i];
				}
				else if (mx<disteucl1[i])
				{

				}
			}


			d_Pout[(Row * m + Col) * 3 + 0] = arrayFiltradoR[posMin];
			d_Pout[(Row * m + Col) * 3 + 1] = arrayFiltradoG[posMin];
			d_Pout[(Row * m + Col) * 3 + 2] = arrayFiltradoB[posMin];
		}//fin de if
		else {
			d_Pout[((Row*m) + Col) * 3 + 0] = d_Pin[((Row*m) + Col) * 3 + 0];
			d_Pout[((Row*m) + Col) * 3 + 1] = d_Pin[((Row*m) + Col) * 3 + 1];
			d_Pout[((Row*m) + Col) * 3 + 2] = d_Pin[((Row*m) + Col) * 3 + 2];

		}
	}
}




__global__ void FiltradoPropuesta(unsigned char* d_Pout, const unsigned char* d_Pin, unsigned char* Noise, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	unsigned char vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	unsigned char arrayFiltradoR[9], arrayFiltradoG[9], arrayFiltradoB[9];
	float mn, mx;
	int posMin = 0;
	int c = 0, i = 0, j = 0;
	unsigned char aux = 100;
	float D[40];
	if ((Row < m - 1) && (Col < n - 1)) {
		if (Noise[(Row * m + Col)] == 255) {
			c = 0;
			for (i = -1; i <= 1; i++) {
				for (j = -1; j <= 1; j++) {
					vectR[c] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
					vectG[c] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
					vectB[c] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

					posicion[c] = c;
					c++;
				}
			}
			//D[0]=Magnitud(vectR, vectG, vectB, i, j//i==0 y j==0 no se hace
			D[0] = (Magnitud(vectR, vectG, vectB, 0, 1));
			D[1] = (Magnitud(vectR, vectG, vectB, 0, 2));
			D[2] = (Magnitud(vectR, vectG, vectB, 0, 3));
			D[3] = (Magnitud(vectR, vectG, vectB, 0, 4));
			D[4] = (Magnitud(vectR, vectG, vectB, 0, 5));
			D[5] = (Magnitud(vectR, vectG, vectB, 0, 6));
			D[6] = (Magnitud(vectR, vectG, vectB, 0, 7));
			D[7] = (Magnitud(vectR, vectG, vectB, 0, 8));
			disteucl1[0] = D[0] + D[1] + D[2] + D[3] + D[4] + D[5] + D[6] + D[7];

			//i=1,j=0 ya esta es D[0]
			//i=1,j=1 No se hace
			D[8] = (Magnitud(vectR, vectG, vectB, 1, 2));
			D[9] = (Magnitud(vectR, vectG, vectB, 1, 3));
			D[10] = (Magnitud(vectR, vectG, vectB, 1, 4));
			D[11] = (Magnitud(vectR, vectG, vectB, 1, 5));
			D[12] = (Magnitud(vectR, vectG, vectB, 1, 6));
			D[13] = (Magnitud(vectR, vectG, vectB, 1, 7));
			D[14] = (Magnitud(vectR, vectG, vectB, 1, 8));
			disteucl1[1] = D[0] + D[8] + D[9] + D[10] + D[11] + D[12] + D[13] + D[14];

			//i=2,j=0 ya esta es D[1]
			//i=2,j=1 ya esta es D[8]
			//i=2,j=2 No se hace
			D[15] = (Magnitud(vectR, vectG, vectB, 2, 3));
			D[16] = (Magnitud(vectR, vectG, vectB, 2, 4));
			D[17] = (Magnitud(vectR, vectG, vectB, 2, 5));
			D[18] = (Magnitud(vectR, vectG, vectB, 2, 6));
			D[19] = (Magnitud(vectR, vectG, vectB, 2, 7));
			D[20] = (Magnitud(vectR, vectG, vectB, 2, 8));
			disteucl1[2] = D[1] + D[8] + D[15] + D[16] + D[17] + D[18] + D[19] + D[20];

			//i=3,j=0 ya esta es D[2]
			//i=3,j=1 ya esta es D[9]
			//i=3,j=2 ya esta es D[15]
			//i=3,j=3 No se hace
			D[21] = (Magnitud(vectR, vectG, vectB, 3, 4));
			D[22] = (Magnitud(vectR, vectG, vectB, 3, 5));
			D[23] = (Magnitud(vectR, vectG, vectB, 3, 6));
			D[24] = (Magnitud(vectR, vectG, vectB, 3, 7));
			D[25] = (Magnitud(vectR, vectG, vectB, 3, 8));
			disteucl1[3] = D[2] + D[9] + D[15] + D[21] + D[22] + D[23] + D[24] + D[25];

			//i=4,j=0 ya esta es D[3]
			//i=4,j=1 ya esta es D[10]
			//i=4,j=2 ya esta es D[16]
			//i=4,j=3 ya esta es D[21]
			//i=4,j=4 No se hace
			D[26] = (Magnitud(vectR, vectG, vectB, 4, 5));
			D[27] = (Magnitud(vectR, vectG, vectB, 4, 6));
			D[28] = (Magnitud(vectR, vectG, vectB, 4, 7));
			D[29] = (Magnitud(vectR, vectG, vectB, 4, 8));
			disteucl1[4] = D[3] + D[10] + D[16] + D[21] + D[26] + D[27] + D[28] + D[29];

			//i=5,j=0 ya esta es D[4]
			//i=5,j=1 ya esta es D[11]
			//i=5,j=2 ya esta es D[17]
			//i=5,j=3 ya esta es D[22]
			//i=5,j=4 ya esta es D[26]
			//i=5,j=5 No se hace
			D[30] = (Magnitud(vectR, vectG, vectB, 5, 6));
			D[31] = (Magnitud(vectR, vectG, vectB, 5, 7));
			D[32] = (Magnitud(vectR, vectG, vectB, 5, 8));
			disteucl1[5] = D[4] + D[11] + D[17] + D[22] + D[26] + D[30] + D[31] + D[32];

			//i=6,j=0 ya esta es D[5]
			//i=6,j=1 ya esta es D[12]
			//i=6,j=2 ya esta es D[18]
			//i=6,j=3 ya esta es D[23]
			//i=6,j=4 ya esta es D[27]
			//i=6,j=5 ya esta es D[30]
			//i=6,j=6 No se hace
			D[33] = (Magnitud(vectR, vectG, vectB, 6, 7));
			D[34] = (Magnitud(vectR, vectG, vectB, 6, 8));
			disteucl1[6] = D[5] + D[12] + D[18] + D[23] + D[27] + D[30] + D[33] + D[34];

			//i=7,j=0 ya esta es D[6]
			//i=7,j=1 ya esta es D[13]
			//i=7,j=2 ya esta es D[19]
			//i=7,j=3 ya esta es D[24]
			//i=7,j=4 ya esta es D[28]
			//i=7,j=5 ya esta es D[31]
			//i=7,j=6 ya esta es D[33]
			//i=7,j=7 No se hace
			D[35] = (Magnitud(vectR, vectG, vectB, 7, 8));
			disteucl1[7] = D[6] + D[13] + D[19] + D[24] + D[28] + D[31] + D[33] + D[35];

			//i=8,j=0 ya esta es D[7]
			//i=8,j=1 ya esta es D[14]
			//i=8,j=2 ya esta es D[20]
			//i=8,j=3 ya esta es D[25]
			//i=8,j=4 ya esta es D[29]
			//i=8,j=5 ya esta es D[32]
			//i=8,j=6 ya esta es D[34]
			//i=8,j=7 ya esta es D[35]
			//i=8,j=8 No se hace
			disteucl1[8] = D[7] + D[14] + D[20] + D[25] + D[29] + D[32] + D[34] + D[35];

			mn = disteucl1[0];
			mx = disteucl1[0];

			posMin = 0;

			for (int i = 0; i<8; i++)
			{
				if (mn>disteucl1[i])
				{
					mn = disteucl1[i];
					posMin = posicion[i];
				}
				else if (mx<disteucl1[i])
				{

				}
			}

			d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
		}//fin de if (Noise[(Row * m + Col)] == 255)
		else {
			d_Pout[((Row*m) + Col) * 3 + 0] = d_Pin[((Row*m) + Col) * 3 + 0];
			d_Pout[((Row*m) + Col) * 3 + 1] = d_Pin[((Row*m) + Col) * 3 + 1];
			d_Pout[((Row*m) + Col) * 3 + 2] = d_Pin[((Row*m) + Col) * 3 + 2];

		}
	}
}

__global__ void FiltradoPropuesta2(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl, disteucl1[9], hold, D[40];
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0, Noise = 0.0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1;
	const float d = .95;

	float mn, mx;
	int posMin = 0;
	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {
			arriva = minCUDA(vectR[F], vectR[4]) + K;
			abajo = maxCUDA(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = minCUDA(vectG[F], vectG[4]) + K;
			abajo = maxCUDA(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = minCUDA(vectB[F], vectB[4]) + K;
			abajo = maxCUDA(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;

			dist_M = minCUDA(minCUDA(val1, val2), val3);
			if (dist_M>d)	P++;
		}

		if (P <= (q + 1)) {
			Noise = 255;
		}
		else {
			Noise = 0;
		}
		if (Noise == 255) {
			//D[0]=Magnitud(vectR, vectG, vectB, i, j//i==0 y j==0 no se hace
			D[0] = (Magnitud(vectR, vectG, vectB, 0, 1));
			D[1] = (Magnitud(vectR, vectG, vectB, 0, 2));
			D[2] = (Magnitud(vectR, vectG, vectB, 0, 3));
			D[3] = (Magnitud(vectR, vectG, vectB, 0, 4));
			D[4] = (Magnitud(vectR, vectG, vectB, 0, 5));
			D[5] = (Magnitud(vectR, vectG, vectB, 0, 6));
			D[6] = (Magnitud(vectR, vectG, vectB, 0, 7));
			D[7] = (Magnitud(vectR, vectG, vectB, 0, 8));
			disteucl1[0] = D[0] + D[1] + D[2] + D[3] + D[4] + D[5] + D[6] + D[7];

			//i=1,j=0 ya esta es D[0]
			//i=1,j=1 No se hace
			D[8] = (Magnitud(vectR, vectG, vectB, 1, 2));
			D[9] = (Magnitud(vectR, vectG, vectB, 1, 3));
			D[10] = (Magnitud(vectR, vectG, vectB, 1, 4));
			D[11] = (Magnitud(vectR, vectG, vectB, 1, 5));
			D[12] = (Magnitud(vectR, vectG, vectB, 1, 6));
			D[13] = (Magnitud(vectR, vectG, vectB, 1, 7));
			D[14] = (Magnitud(vectR, vectG, vectB, 1, 8));
			disteucl1[1] = D[0] + D[8] + D[9] + D[10] + D[11] + D[12] + D[13] + D[14];

			//i=2,j=0 ya esta es D[1]
			//i=2,j=1 ya esta es D[8]
			//i=2,j=2 No se hace
			D[15] = (Magnitud(vectR, vectG, vectB, 2, 3));
			D[16] = (Magnitud(vectR, vectG, vectB, 2, 4));
			D[17] = (Magnitud(vectR, vectG, vectB, 2, 5));
			D[18] = (Magnitud(vectR, vectG, vectB, 2, 6));
			D[19] = (Magnitud(vectR, vectG, vectB, 2, 7));
			D[20] = (Magnitud(vectR, vectG, vectB, 2, 8));
			disteucl1[2] = D[1] + D[8] + D[15] + D[16] + D[17] + D[18] + D[19] + D[20];

			//i=3,j=0 ya esta es D[2]
			//i=3,j=1 ya esta es D[9]
			//i=3,j=2 ya esta es D[15]
			//i=3,j=3 No se hace
			D[21] = (Magnitud(vectR, vectG, vectB, 3, 4));
			D[22] = (Magnitud(vectR, vectG, vectB, 3, 5));
			D[23] = (Magnitud(vectR, vectG, vectB, 3, 6));
			D[24] = (Magnitud(vectR, vectG, vectB, 3, 7));
			D[25] = (Magnitud(vectR, vectG, vectB, 3, 8));
			disteucl1[3] = D[2] + D[9] + D[15] + D[21] + D[22] + D[23] + D[24] + D[25];

			//i=4,j=0 ya esta es D[3]
			//i=4,j=1 ya esta es D[10]
			//i=4,j=2 ya esta es D[16]
			//i=4,j=3 ya esta es D[21]
			//i=4,j=4 No se hace
			D[26] = (Magnitud(vectR, vectG, vectB, 4, 5));
			D[27] = (Magnitud(vectR, vectG, vectB, 4, 6));
			D[28] = (Magnitud(vectR, vectG, vectB, 4, 7));
			D[29] = (Magnitud(vectR, vectG, vectB, 4, 8));
			disteucl1[4] = D[3] + D[10] + D[16] + D[21] + D[26] + D[27] + D[28] + D[29];

			//i=5,j=0 ya esta es D[4]
			//i=5,j=1 ya esta es D[11]
			//i=5,j=2 ya esta es D[17]
			//i=5,j=3 ya esta es D[22]
			//i=5,j=4 ya esta es D[26]
			//i=5,j=5 No se hace
			D[30] = (Magnitud(vectR, vectG, vectB, 5, 6));
			D[31] = (Magnitud(vectR, vectG, vectB, 5, 7));
			D[32] = (Magnitud(vectR, vectG, vectB, 5, 8));
			disteucl1[5] = D[4] + D[11] + D[17] + D[22] + D[26] + D[30] + D[31] + D[32];

			//i=6,j=0 ya esta es D[5]
			//i=6,j=1 ya esta es D[12]
			//i=6,j=2 ya esta es D[18]
			//i=6,j=3 ya esta es D[23]
			//i=6,j=4 ya esta es D[27]
			//i=6,j=5 ya esta es D[30]
			//i=6,j=6 No se hace
			D[33] = (Magnitud(vectR, vectG, vectB, 6, 7));
			D[34] = (Magnitud(vectR, vectG, vectB, 6, 8));
			disteucl1[6] = D[5] + D[12] + D[18] + D[23] + D[27] + D[30] + D[33] + D[34];

			//i=7,j=0 ya esta es D[6]
			//i=7,j=1 ya esta es D[13]
			//i=7,j=2 ya esta es D[19]
			//i=7,j=3 ya esta es D[24]
			//i=7,j=4 ya esta es D[28]
			//i=7,j=5 ya esta es D[31]
			//i=7,j=6 ya esta es D[33]
			//i=7,j=7 No se hace
			D[35] = (Magnitud(vectR, vectG, vectB, 7, 8));
			disteucl1[7] = D[6] + D[13] + D[19] + D[24] + D[28] + D[31] + D[33] + D[35];

			//i=8,j=0 ya esta es D[7]
			//i=8,j=1 ya esta es D[14]
			//i=8,j=2 ya esta es D[20]
			//i=8,j=3 ya esta es D[25]
			//i=8,j=4 ya esta es D[29]
			//i=8,j=5 ya esta es D[32]
			//i=8,j=6 ya esta es D[34]
			//i=8,j=7 ya esta es D[35]
			//i=8,j=8 No se hace
			disteucl1[8] = D[7] + D[14] + D[20] + D[25] + D[29] + D[32] + D[34] + D[35];

			mn = disteucl1[0];
			mx = disteucl1[0];

			posMin = 0;

			for (int i = 0; i<8; i++)
			{
				if (mn>disteucl1[i])
				{
					mn = disteucl1[i];
					posMin = posicion[i];
				}
				else if (mx < disteucl1[i])
				{

				}
			}

			d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
		}
		else {

			d_Pout[((Row*m) + Col) * 3 + 0] = vectR[4];
			d_Pout[((Row*m) + Col) * 3 + 1] = vectG[4];
			d_Pout[((Row*m) + Col) * 3 + 2] = vectB[4];
		}

	}//if de Row y Col

}//cierre de funcion



__device__ inline void s(unsigned char* a, unsigned char*b)
{
	int tmp;
	if (*a>*b) {//si a es mayor a b, se intercambian a y b.
		tmp = *b;
		*b = *a;
		*a = tmp;
	}
}

#define min3(a,b,c) s(a, b); s(a,c);
#define max3(a,b,c) s(b, c); s(a,c);

#define minmax3(a,b,c)			max3(a, b, c); s(a,b);
#define minmax4(a,b,c,d)		s(a, b); s(c,d);s(a, c); s(b,d);
#define minmax5(a,b,c,d,e)		s(a, b); s(c,d);min3(a,c,e);max3(b,d,e);

#define minmax6(a,b,c,d,e,f)	s(a,d);s(b,e);s(c,f);min3(a,b,c);max3(d,e,f);

__global__ void MarginalMedianFilter_Global_Forgetfull(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	//Calculate the row # of the d_Pin and d_Pout element to process 
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	// Calculate the column # of the d_Pin and d_Pout element to process 
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	// each thread computes one element of d_Pout if in range 
	// Se debe de checar si los pixeles esta dentro del intervalo de 8 bits
	int x = 0, c = 0, d = 0, F, canal;

	int i, j;
	unsigned char vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	unsigned char swap;
	unsigned char a0, a1, a2, a3, a4, a5;

	if ((Row < m - 1) && (Col < n - 1)) {

		vectR[0] = d_Pin[((Row - 1) * n + (Col - 1)) * 3 + 0];
		vectG[0] = d_Pin[((Row - 1) * n + (Col - 1)) * 3 + 1];
		vectB[0] = d_Pin[((Row - 1) * n + (Col - 1)) * 3 + 2];

		vectR[1] = d_Pin[((Row - 1) * n + (Col + 0)) * 3 + 0];
		vectG[1] = d_Pin[((Row - 1) * n + (Col + 0)) * 3 + 1];
		vectB[1] = d_Pin[((Row - 1) * n + (Col + 0)) * 3 + 2];

		vectR[2] = d_Pin[((Row - 1) * n + (Col + 1)) * 3 + 0];
		vectG[2] = d_Pin[((Row - 1) * n + (Col + 1)) * 3 + 1];
		vectB[2] = d_Pin[((Row - 1) * n + (Col + 1)) * 3 + 2];

		vectR[3] = d_Pin[((Row + 0) * n + (Col - 1)) * 3 + 0];
		vectG[3] = d_Pin[((Row + 0) * n + (Col - 1)) * 3 + 1];
		vectB[3] = d_Pin[((Row + 0) * n + (Col - 1)) * 3 + 2];

		vectR[4] = d_Pin[((Row + 0) * n + (Col + 0)) * 3 + 0];
		vectG[4] = d_Pin[((Row + 0) * n + (Col + 0)) * 3 + 1];//central
		vectB[4] = d_Pin[((Row + 0) * n + (Col + 0)) * 3 + 2];

		vectR[5] = d_Pin[((Row + 0) * n + (Col + 1)) * 3 + 0];
		vectG[5] = d_Pin[((Row + 0) * n + (Col + 1)) * 3 + 1];
		vectB[5] = d_Pin[((Row + 0) * n + (Col + 1)) * 3 + 2];

		minmax6(&vectR[0], &vectR[1], &vectR[2], &vectR[3], &vectR[4], &vectR[5]);
		minmax6(&vectG[0], &vectG[1], &vectG[2], &vectG[3], &vectG[4], &vectG[5]);
		minmax6(&vectB[0], &vectB[1], &vectB[2], &vectB[3], &vectB[4], &vectB[5]);
		vectR[5] = d_Pin[((Row + 1) * n + (Col - 1)) * 3 + 0];
		vectG[5] = d_Pin[((Row + 1) * n + (Col - 1)) * 3 + 1];
		vectB[5] = d_Pin[((Row + 1) * n + (Col - 1)) * 3 + 2];

		minmax5(&vectR[1], &vectR[2], &vectR[3], &vectR[4], &vectR[5]);
		minmax5(&vectG[1], &vectG[2], &vectG[3], &vectG[4], &vectG[5]);
		minmax5(&vectB[1], &vectB[2], &vectB[3], &vectB[4], &vectB[5]);
		vectR[5] = d_Pin[((Row + 1) * n + (Col + 0)) * 3 + 0];
		vectG[5] = d_Pin[((Row + 1) * n + (Col + 0)) * 3 + 1];
		vectB[5] = d_Pin[((Row + 1) * n + (Col + 0)) * 3 + 2];

		minmax4(&vectR[2], &vectR[3], &vectR[4], &vectR[5]);
		minmax4(&vectG[2], &vectG[3], &vectG[4], &vectG[5]);
		minmax4(&vectB[2], &vectB[3], &vectB[4], &vectB[5]);
		vectR[5] = d_Pin[((Row + 1) * n + (Col + 1)) * 3 + 0];
		vectG[5] = d_Pin[((Row + 1) * n + (Col + 1)) * 3 + 1];
		vectB[5] = d_Pin[((Row + 1) * n + (Col + 1)) * 3 + 2];

		minmax3(&vectR[3], &vectR[4], &vectR[5]);
		minmax3(&vectG[3], &vectG[4], &vectG[5]);
		minmax3(&vectB[3], &vectB[4], &vectB[5]);


		d_Pout[(Row * m + Col) * 3 + 0] = vectR[4]; // ojo aqui va desde 0 a 8
		d_Pout[(Row * m + Col) * 3 + 1] = vectG[4];
		d_Pout[(Row * m + Col) * 3 + 2] = vectB[4];

	}

}


//este es la propuesta
__global__ void FiltradoPropuesta_MMF(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl, disteucl1[9], hold, D[40];
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0, Noise = 0.0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1;
	const float d = .95;

	float mn, mx;
	int posMin = 0;

	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {
			arriva = minCUDA(vectR[F], vectR[4]) + K;
			abajo = maxCUDA(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = minCUDA(vectG[F], vectG[4]) + K;
			abajo = maxCUDA(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = minCUDA(vectB[F], vectB[4]) + K;
			abajo = maxCUDA(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;

			dist_M = minCUDA(minCUDA(val1, val2), val3);
			if (dist_M>d)	P++;
		}

		if (P <= (q + 1)) {
			Noise = 255;
		}
		else {
			Noise = 0;
		}
		if (Noise == 255) {

			minmax6(&vectR[0], &vectR[1], &vectR[2], &vectR[3], &vectR[4], &vectR[5]);
			minmax6(&vectG[0], &vectG[1], &vectG[2], &vectG[3], &vectG[4], &vectG[5]);
			minmax6(&vectB[0], &vectB[1], &vectB[2], &vectB[3], &vectB[4], &vectB[5]);
			vectR[5] = vectR[6];
			vectG[5] = vectG[6];
			vectB[5] = vectB[6];

			minmax5(&vectR[1], &vectR[2], &vectR[3], &vectR[4], &vectR[5]);
			minmax5(&vectG[1], &vectG[2], &vectG[3], &vectG[4], &vectG[5]);
			minmax5(&vectB[1], &vectB[2], &vectB[3], &vectB[4], &vectB[5]);
			vectR[5] = vectR[7];
			vectG[5] = vectG[7];
			vectB[5] = vectB[7];

			minmax4(&vectR[2], &vectR[3], &vectR[4], &vectR[5]);
			minmax4(&vectG[2], &vectG[3], &vectG[4], &vectG[5]);
			minmax4(&vectB[2], &vectB[3], &vectB[4], &vectB[5]);
			vectR[5] = vectR[8];
			vectG[5] = vectG[8];
			vectB[5] = vectB[8];

			minmax3(&vectR[3], &vectR[4], &vectR[5]);
			minmax3(&vectG[3], &vectG[4], &vectG[5]);
			minmax3(&vectB[3], &vectB[4], &vectB[5]);


			d_Pout[(Row * m + Col) * 3 + 0] = vectR[4]; // ojo aqui va desde 0 a 8
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[4];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[4];

		}
		else {

			d_Pout[((Row*m) + Col) * 3 + 0] = vectR[4];
			d_Pout[((Row*m) + Col) * 3 + 1] = vectG[4];
			d_Pout[((Row*m) + Col) * 3 + 2] = vectB[4];
		}

	}//if de Row y Col

}//cierre de funcion
 //propuesta con filtrado VMF
__global__ void FiltradoPropuesta_VMF(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0, i = 0, c = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl, disteucl1[9], hold, D[40];
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0, Noise = 0.0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1;
	const float d = .95;

	float mn, mx;
	int posMin = 0;

	if ((Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {
			arriva = minCUDA(vectR[F], vectR[4]) + K;
			abajo = maxCUDA(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = minCUDA(vectG[F], vectG[4]) + K;
			abajo = maxCUDA(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = minCUDA(vectB[F], vectB[4]) + K;
			abajo = maxCUDA(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;

			dist_M = minCUDA(minCUDA(val1, val2), val3);
			if (dist_M>d)	P++;
		}

		if (P <= (q + 1)) {
			Noise = 255;
		}
		else {
			Noise = 0;
		}
		if (Noise == 255) {
			disteucl = 0;
			for (F = 0; F <= 8; F++) {
				for (x = 0; x <= 8; x++) {
					//disteucl += abs(vectB[F] - vectB[x]) + abs(vectG[F] - vectG[x]) + abs(vectR[F] - vectR[x]);
					//disteucl += sqrt(pow(vectB[F] - vectB[x], 2) + pow(vectG[F] - vectG[x], 2) + pow(vectR[F] - vectR[x], 2));
					//disteucl +=  (vectB[F]-vectB[x]) * (vectB[F]-vectB[x]) + (vectG[F]-vectG[x]) * (vectG[F]-vectG[x])+(vectR[F]-vectR[x]) * (vectR[F]-vectR[x]);
					//disteucl +=  (vectB[F]-vectB[x]);
					//disteucl += (vectB[F] - vectB[x])*(vectB[F] - vectB[x]) + (vectG[F] - vectG[x])*(vectG[F] - vectG[x]) + (vectR[F] - vectR[x])*(vectR[F] - vectR[x]);
					float distR = abs(vectR[F] - vectR[x]);
					float distG = abs(vectG[F] - vectG[x]);
					float distB = abs(vectB[F] - vectB[x]);
					disteucl += distR + distB + distG;
				}
				disteucl1[F] = disteucl;
				disteucl = 0;
			}

			mn = disteucl1[0];
			mx = disteucl1[0];

			posMin = 0;

			for (i = 0; i<8; i++)
			{
				if (mn>disteucl1[i])
				{
					mn = disteucl1[i];
					posMin = posicion[i];
				}
				else if (mx<disteucl1[i])
				{

				}
			}

			d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin]; // ojo aqui va desde 0 a 8
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];



		}
		else {

			d_Pout[((Row*m) + Col) * 3 + 0] = vectR[4];
			d_Pout[((Row*m) + Col) * 3 + 1] = vectG[4];
			d_Pout[((Row*m) + Col) * 3 + 2] = vectB[4];
		}

	}//if de Row y Col

}//cierre de funcion
__global__ void FiltradoPropuesta_AMF(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0, i = 0, c = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl, disteucl1[9], hold, D[40];
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0, Noise = 0.0;
	unsigned int P = 0;
	const unsigned int K = 1024, q = 1;
	const float d = .95;
	unsigned int Div = 0;
	float mn, mx;
	int posMin = 0;
	float sumR = 0.0, sumG = 0.0, sumB = 0.0;

	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		for (F = 0; F <= 8; F++) {
			arriva = minCUDA(vectR[F], vectR[4]) + K;
			abajo = maxCUDA(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = minCUDA(vectG[F], vectG[4]) + K;
			abajo = maxCUDA(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = minCUDA(vectB[F], vectB[4]) + K;
			abajo = maxCUDA(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;

			dist_M = minCUDA(minCUDA(val1, val2), val3);
			if (dist_M>d)	P++;
		}

		if (P <= (q + 1)) {
			Noise = 255;
		}
		else {
			Noise = 0;
		}
		if (Noise == 255) {

			for (int i = -1; i <= 1; i++) {
				for (int j = -1; j <= 1; j++) {

					sumR += d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
					sumG += d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
					sumB += d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				}
			}


			d_Pout[((Row*m) + Col) * 3 + 0] = sumR / 9;
			d_Pout[((Row*m) + Col) * 3 + 1] = sumG / 9;
			d_Pout[((Row*m) + Col) * 3 + 2] = sumB / 9;
		}
		else {

			d_Pout[((Row*m) + Col) * 3 + 0] = vectR[4];
			d_Pout[((Row*m) + Col) * 3 + 1] = vectG[4];
			d_Pout[((Row*m) + Col) * 3 + 2] = vectB[4];
		}

	}//if de Row y Col

}//cierre de funcion


__global__ void VMF_Global_Forgetfull_Reuse(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl = 0.0, disteucl1[9], hold;
	float D[40];
	float mn, mx;
	int posMin = 0;


	if ((Row < m - 1) && (Col < n - 1)) {
		F = 0;

		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		//D[0]=Magnitud(vectR, vectG, vectB, i, j//i==0 y j==0 no se hace
		D[0] = (Magnitud(vectR, vectG, vectB, 0, 1));
		D[1] = (Magnitud(vectR, vectG, vectB, 0, 2));
		D[2] = (Magnitud(vectR, vectG, vectB, 0, 3));
		D[3] = (Magnitud(vectR, vectG, vectB, 0, 4));
		D[4] = (Magnitud(vectR, vectG, vectB, 0, 5));
		D[5] = (Magnitud(vectR, vectG, vectB, 0, 6));
		D[6] = (Magnitud(vectR, vectG, vectB, 0, 7));
		D[7] = (Magnitud(vectR, vectG, vectB, 0, 8));
		disteucl1[0] = D[0] + D[1] + D[2] + D[3] + D[4] + D[5] + D[6] + D[7];

		//i=1,j=0 ya esta es D[0]
		//i=1,j=1 No se hace
		D[8] = (Magnitud(vectR, vectG, vectB, 1, 2));
		D[9] = (Magnitud(vectR, vectG, vectB, 1, 3));
		D[10] = (Magnitud(vectR, vectG, vectB, 1, 4));
		D[11] = (Magnitud(vectR, vectG, vectB, 1, 5));
		D[12] = (Magnitud(vectR, vectG, vectB, 1, 6));
		D[13] = (Magnitud(vectR, vectG, vectB, 1, 7));
		D[14] = (Magnitud(vectR, vectG, vectB, 1, 8));
		disteucl1[1] = D[0] + D[8] + D[9] + D[10] + D[11] + D[12] + D[13] + D[14];

		//i=2,j=0 ya esta es D[1]
		//i=2,j=1 ya esta es D[8]
		//i=2,j=2 No se hace
		D[15] = (Magnitud(vectR, vectG, vectB, 2, 3));
		D[16] = (Magnitud(vectR, vectG, vectB, 2, 4));
		D[17] = (Magnitud(vectR, vectG, vectB, 2, 5));
		D[18] = (Magnitud(vectR, vectG, vectB, 2, 6));
		D[19] = (Magnitud(vectR, vectG, vectB, 2, 7));
		D[20] = (Magnitud(vectR, vectG, vectB, 2, 8));
		disteucl1[2] = D[1] + D[8] + D[15] + D[16] + D[17] + D[18] + D[19] + D[20];

		//i=3,j=0 ya esta es D[2]
		//i=3,j=1 ya esta es D[9]
		//i=3,j=2 ya esta es D[15]
		//i=3,j=3 No se hace
		D[21] = (Magnitud(vectR, vectG, vectB, 3, 4));
		D[22] = (Magnitud(vectR, vectG, vectB, 3, 5));
		D[23] = (Magnitud(vectR, vectG, vectB, 3, 6));
		D[24] = (Magnitud(vectR, vectG, vectB, 3, 7));
		D[25] = (Magnitud(vectR, vectG, vectB, 3, 8));
		disteucl1[3] = D[2] + D[9] + D[15] + D[21] + D[22] + D[23] + D[24] + D[25];

		//i=4,j=0 ya esta es D[3]
		//i=4,j=1 ya esta es D[10]
		//i=4,j=2 ya esta es D[16]
		//i=4,j=3 ya esta es D[21]
		//i=4,j=4 No se hace
		D[26] = (Magnitud(vectR, vectG, vectB, 4, 5));
		D[27] = (Magnitud(vectR, vectG, vectB, 4, 6));
		D[28] = (Magnitud(vectR, vectG, vectB, 4, 7));
		D[29] = (Magnitud(vectR, vectG, vectB, 4, 8));
		disteucl1[4] = D[3] + D[10] + D[16] + D[21] + D[26] + D[27] + D[28] + D[29];

		//i=5,j=0 ya esta es D[4]
		//i=5,j=1 ya esta es D[11]
		//i=5,j=2 ya esta es D[17]
		//i=5,j=3 ya esta es D[22]
		//i=5,j=4 ya esta es D[26]
		//i=5,j=5 No se hace
		D[30] = (Magnitud(vectR, vectG, vectB, 5, 6));
		D[31] = (Magnitud(vectR, vectG, vectB, 5, 7));
		D[32] = (Magnitud(vectR, vectG, vectB, 5, 8));
		disteucl1[5] = D[4] + D[11] + D[17] + D[22] + D[26] + D[30] + D[31] + D[32];

		//i=6,j=0 ya esta es D[5]
		//i=6,j=1 ya esta es D[12]
		//i=6,j=2 ya esta es D[18]
		//i=6,j=3 ya esta es D[23]
		//i=6,j=4 ya esta es D[27]
		//i=6,j=5 ya esta es D[30]
		//i=6,j=6 No se hace
		D[33] = (Magnitud(vectR, vectG, vectB, 6, 7));
		D[34] = (Magnitud(vectR, vectG, vectB, 6, 8));
		disteucl1[6] = D[5] + D[12] + D[18] + D[23] + D[27] + D[30] + D[33] + D[34];

		//i=7,j=0 ya esta es D[6]
		//i=7,j=1 ya esta es D[13]
		//i=7,j=2 ya esta es D[19]
		//i=7,j=3 ya esta es D[24]
		//i=7,j=4 ya esta es D[28]
		//i=7,j=5 ya esta es D[31]
		//i=7,j=6 ya esta es D[33]
		//i=7,j=7 No se hace
		D[35] = (Magnitud(vectR, vectG, vectB, 7, 8));
		disteucl1[7] = D[6] + D[13] + D[19] + D[24] + D[28] + D[31] + D[33] + D[35];

		//i=8,j=0 ya esta es D[7]
		//i=8,j=1 ya esta es D[14]
		//i=8,j=2 ya esta es D[20]
		//i=8,j=3 ya esta es D[25]
		//i=8,j=4 ya esta es D[29]
		//i=8,j=5 ya esta es D[32]
		//i=8,j=6 ya esta es D[34]
		//i=8,j=7 ya esta es D[35]
		//i=8,j=8 No se hace
		disteucl1[8] = D[7] + D[14] + D[20] + D[25] + D[29] + D[32] + D[34] + D[35];

		mn = disteucl1[0];
		mx = disteucl1[0];

		posMin = 0;

		for (int i = 0; i<8; i++)
		{
			if (mn>disteucl1[i])
			{
				mn = disteucl1[i];
				posMin = posicion[i];
			}
			else if (mx<disteucl1[i])
			{

			}
		}

		d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
		d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
		d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
	}
}


/*
__global__ void Idea_VMF_FuzzyPeer(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m)
{
int Row = blockIdx.y*blockDim.y + threadIdx.y;
int Col = blockIdx.x*blockDim.x + threadIdx.x;

int x = 0, posicion[9], hold2 = 0, F = 0;
unsigned char vectR[9], vectG[9], vectB[9];
float disteucl = 0.0, disteucl1[9], hold;
float D[40];
float mn, mx;
int posMin = 0;
float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0;

if ((Row < m - 1) && (Col < n - 1)){
F = 0;

for (int i = -1; i <= 1; i++){
for (int j = -1; j <= 1; j++){
vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

posicion[F] = F;
F++;
}
}
//D[0]=Magnitud(vectR, vectG, vectB, i, j//i==0 y j==0 no se hace
D[0] = (Magnitud(vectR, vectG, vectB, 0, 1));
D[1] = (Magnitud(vectR, vectG, vectB, 0, 2));
D[2] = (Magnitud(vectR, vectG, vectB, 0, 3));
D[3] = (Magnitud(vectR, vectG, vectB, 0, 4));
D[4] = (Magnitud(vectR, vectG, vectB, 0, 5));
D[5] = (Magnitud(vectR, vectG, vectB, 0, 6));
D[6] = (Magnitud(vectR, vectG, vectB, 0, 7));
D[7] = (Magnitud(vectR, vectG, vectB, 0, 8));
disteucl1[0] = D[0] + D[1] + D[2] + D[3] + D[4] + D[5] + D[6] + D[7];

//i=1,j=0 ya esta es D[0]
//i=1,j=1 No se hace
D[8] = (Magnitud(vectR, vectG, vectB, 1, 2));
D[9] = (Magnitud(vectR, vectG, vectB, 1, 3));
D[10] = (Magnitud(vectR, vectG, vectB, 1, 4));
D[11] = (Magnitud(vectR, vectG, vectB, 1, 5));
D[12] = (Magnitud(vectR, vectG, vectB, 1, 6));
D[13] = (Magnitud(vectR, vectG, vectB, 1, 7));
D[14] = (Magnitud(vectR, vectG, vectB, 1, 8));
disteucl1[1] = D[0] + D[8] + D[9] + D[10] + D[11] + D[12] + D[13] + D[14];

//i=2,j=0 ya esta es D[1]
//i=2,j=1 ya esta es D[8]
//i=2,j=2 No se hace
D[15] = (Magnitud(vectR, vectG, vectB, 2, 3));
D[16] = (Magnitud(vectR, vectG, vectB, 2, 4));
D[17] = (Magnitud(vectR, vectG, vectB, 2, 5));
D[18] = (Magnitud(vectR, vectG, vectB, 2, 6));
D[19] = (Magnitud(vectR, vectG, vectB, 2, 7));
D[20] = (Magnitud(vectR, vectG, vectB, 2, 8));
disteucl1[2] = D[1] + D[8] + D[15] + D[16] + D[17] + D[18] + D[19] + D[20];

//i=3,j=0 ya esta es D[2]
//i=3,j=1 ya esta es D[9]
//i=3,j=2 ya esta es D[15]
//i=3,j=3 No se hace
D[21] = (Magnitud(vectR, vectG, vectB, 3, 4));
D[22] = (Magnitud(vectR, vectG, vectB, 3, 5));
D[23] = (Magnitud(vectR, vectG, vectB, 3, 6));
D[24] = (Magnitud(vectR, vectG, vectB, 3, 7));
D[25] = (Magnitud(vectR, vectG, vectB, 3, 8));
disteucl1[3] = D[2] + D[9] + D[15] + D[21] + D[22] + D[23] + D[24] + D[25];

//i=4,j=0 ya esta es D[3]
//i=4,j=1 ya esta es D[10]
//i=4,j=2 ya esta es D[16]
//i=4,j=3 ya esta es D[21]
//i=4,j=4 No se hace
D[26] = (Magnitud(vectR, vectG, vectB, 4, 5));
D[27] = (Magnitud(vectR, vectG, vectB, 4, 6));
D[28] = (Magnitud(vectR, vectG, vectB, 4, 7));
D[29] = (Magnitud(vectR, vectG, vectB, 4, 8));
disteucl1[4] = D[3] + D[10] + D[16] + D[21] + D[26] + D[27] + D[28] + D[29];

//i=5,j=0 ya esta es D[4]
//i=5,j=1 ya esta es D[11]
//i=5,j=2 ya esta es D[17]
//i=5,j=3 ya esta es D[22]
//i=5,j=4 ya esta es D[26]
//i=5,j=5 No se hace
D[30] = (Magnitud(vectR, vectG, vectB, 5, 6));
D[31] = (Magnitud(vectR, vectG, vectB, 5, 7));
D[32] = (Magnitud(vectR, vectG, vectB, 5, 8));
disteucl1[5] = D[4] + D[11] + D[17] + D[22] + D[26] + D[30] + D[31] + D[32];

//i=6,j=0 ya esta es D[5]
//i=6,j=1 ya esta es D[12]
//i=6,j=2 ya esta es D[18]
//i=6,j=3 ya esta es D[23]
//i=6,j=4 ya esta es D[27]
//i=6,j=5 ya esta es D[30]
//i=6,j=6 No se hace
D[33] = (Magnitud(vectR, vectG, vectB, 6, 7));
D[34] = (Magnitud(vectR, vectG, vectB, 6, 8));
disteucl1[6] = D[5] + D[12] + D[18] + D[23] + D[27] + D[30] + D[33] + D[34];

//i=7,j=0 ya esta es D[6]
//i=7,j=1 ya esta es D[13]
//i=7,j=2 ya esta es D[19]
//i=7,j=3 ya esta es D[24]
//i=7,j=4 ya esta es D[28]
//i=7,j=5 ya esta es D[31]
//i=7,j=6 ya esta es D[33]
//i=7,j=7 No se hace
D[35] = (Magnitud(vectR, vectG, vectB, 7, 8));
disteucl1[7] = D[6] + D[13] + D[19] + D[24] + D[28] + D[31] + D[33] + D[35];

//i=8,j=0 ya esta es D[7]
//i=8,j=1 ya esta es D[14]
//i=8,j=2 ya esta es D[20]
//i=8,j=3 ya esta es D[25]
//i=8,j=4 ya esta es D[29]
//i=8,j=5 ya esta es D[32]
//i=8,j=6 ya esta es D[34]
//i=8,j=7 ya esta es D[35]
//i=8,j=8 No se hace
disteucl1[8] = D[7] + D[14] + D[20] + D[25] + D[29] + D[32] + D[34] + D[35];

mn = disteucl1[0];
mx = disteucl1[0];

posMin = 0;

for (int i = 0; i<8; i++)
{
if (mn>disteucl1[i])
{
mn = disteucl1[i];
posMin = posicion[i];
}
else if (mx<disteucl1[i])
{

}
}
vectR[4] = vectR[posMin];
vectG[4] = vectG[posMin];
vectB[4] = vectB[posMin];
for (F = 0; F <= 8; F++){
arriva = minCUDA(vectR[F], vectR[4]) + K;
abajo = maxCUDA(vectR[F], vectR[4]) + K;
val1 = arriva / abajo;

arriva = minCUDA(vectG[F], vectG[4]) + K;
abajo = maxCUDA(vectG[F], vectG[4]) + K;
val2 = arriva / abajo;

arriva = minCUDA(vectB[F], vectB[4]) + K;
abajo = maxCUDA(vectB[F], vectB[4]) + K;
val3 = arriva / abajo;

dist_M = minCUDA(minCUDA(val1, val2), val3);
if (dist_M>d)	P++;
}

if (P <= (q + 1)){
Noise[(Row * m + Col)] = 255;
}
else{
Noise[(Row * m + Col)] = 0;
}




d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
}
}
*/

__device__ float MagnitudL1(float* VectR, float* VectG, float* VectB, unsigned int i, unsigned int j) {

	float distR = abs(VectR[i] - VectR[j]);
	float distG = abs(VectG[i] - VectG[j]);
	float distB = abs(VectB[i] - VectB[j]);

	//return sqrt((distR)*(distR)+(distG)*(distG)+(distB)*(distB));
	return distR + distB + distG;

}

//Gran
__device__ float S_shape(float Nabla, unsigned int a, unsigned int b) {

	if (Nabla <= a)		return 0;

	if (a <= Nabla && Nabla <= ((a + b) / 2)) {
		float aux = (Nabla - a) / (b - a);
		return 2 * aux*aux;
	}

	if (((a + b) / 2) <= Nabla && Nabla <= b) {
		float aux = ((Nabla - b) / (b - a));
		return 1 - (2 * aux*aux);
	}

	if (Nabla >= b)		return 1;

}
//Peque
__device__ float Z_shape(float Nabla, unsigned int a, unsigned int b) {

	if (Nabla <= a)		return 1;
	if (a <= Nabla && Nabla <= ((a + b) / 2)) {
		float aux = (Nabla - a) / (b - a);
		return 1 - (2 * aux*aux);
	}
	if (((a + b) / 2) <= Nabla && Nabla <= b) {
		float aux = (Nabla - b) / (b - a);
		return 2 * aux*aux;
	}
	if (Nabla >= b)		return 0;
}



__global__ void FTSCF_GPU
(unsigned char* d_Pout, const unsigned char* d_Pin, const unsigned int a,
	const unsigned int b, const unsigned int THS, int n, int m) {

	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], posMin = 6, F = 0, i = 0, j = 0;

	float vectR[25], vectG[25], vectB[25];
	float D[45], disteucl1[9], uGran[3], uPeque[3], rs[9], r = 0;

	float mn, mx;

	posicion[0] = 6; posicion[1] = 7; posicion[2] = 8; posicion[3] = 11; posicion[4] = 12;
	posicion[5] = 13; posicion[6] = 16; posicion[7] = 17; posicion[8] = 18;

	if ((Row < m - 3) && (Col < n - 3)) {
		for (i = -2; i <= 2; i++) {
			for (j = -2; j <= 2; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];
				F++;
			}
		}


		//NW
		D[0] = (MagnitudL1(vectR, vectG, vectB, 12, 6));
		uGran[0] = S_shape(D[0], a, b);

		D[1] = (MagnitudL1(vectR, vectG, vectB, 12, 8));
		uGran[1] = S_shape(D[1], a, b);

		D[2] = (MagnitudL1(vectR, vectG, vectB, 12, 16));
		uGran[2] = S_shape(D[2], a, b);

		D[3] = (MagnitudL1(vectR, vectG, vectB, 16, 10));
		uPeque[0] = Z_shape(D[3], a, b);

		D[4] = (MagnitudL1(vectR, vectG, vectB, 2, 8));
		uPeque[1] = Z_shape(D[4], a, b);

		rs[0] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//N
		D[5] = (MagnitudL1(vectR, vectG, vectB, 12, 7));
		uGran[0] = S_shape(D[5], a, b);

		D[6] = (MagnitudL1(vectR, vectG, vectB, 12, 13));
		uGran[1] = S_shape(D[6], a, b);

		D[7] = (MagnitudL1(vectR, vectG, vectB, 12, 11));
		uGran[2] = S_shape(D[7], a, b);

		D[8] = (MagnitudL1(vectR, vectG, vectB, 11, 6));
		uPeque[0] = Z_shape(D[8], a, b);

		D[9] = (MagnitudL1(vectR, vectG, vectB, 8, 13));
		uPeque[1] = Z_shape(D[9], a, b);

		rs[1] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//NE
		//D[10] = (MagnitudL1(vectR, vectG, vectB, 12, 8));
		// es D[1]
		uGran[0] = S_shape(D[1], a, b);

		//D[11] = (MagnitudL1(vectR, vectG, vectB, 12, 6));
		// es D[0]
		uGran[1] = S_shape(D[0], a, b);

		D[10] = (MagnitudL1(vectR, vectG, vectB, 12, 18));
		uGran[2] = S_shape(D[10], a, b);

		D[11] = (MagnitudL1(vectR, vectG, vectB, 18, 14));
		uPeque[0] = Z_shape(D[11], a, b);

		D[12] = (MagnitudL1(vectR, vectG, vectB, 6, 2));
		uPeque[1] = Z_shape(D[12], a, b);

		rs[2] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//E			
		//D[15] = (MagnitudL1(vectR, vectG, vectB, 12, 13));
		//es D[6]
		uGran[0] = S_shape(D[6], a, b);

		//D[16] = (MagnitudL1(vectR, vectG, vectB, 12, 7));
		//es D[5]
		uGran[1] = S_shape(D[5], a, b);

		D[13] = (MagnitudL1(vectR, vectG, vectB, 12, 17));
		uGran[2] = S_shape(D[13], a, b);

		D[14] = (MagnitudL1(vectR, vectG, vectB, 7, 8));
		uPeque[0] = Z_shape(D[14], a, b);

		D[15] = (MagnitudL1(vectR, vectG, vectB, 17, 18));
		uPeque[1] = Z_shape(D[15], a, b);

		rs[3] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//SE
		//D[20] = (MagnitudL1(vectR, vectG, vectB, 12, 18));
		//es D[10]
		uGran[0] = S_shape(D[10], a, b);

		//es D[2]
		//D[21] = (MagnitudL1(vectR, vectG, vectB, 12, 16));
		uGran[1] = S_shape(D[2], a, b);

		//es D[1]
		//D[22] = (MagnitudL1(vectR, vectG, vectB, 12, 8));
		uGran[2] = S_shape(D[1], a, b);

		D[16] = (MagnitudL1(vectR, vectG, vectB, 16, 22));
		uPeque[0] = Z_shape(D[16], a, b);

		D[17] = (MagnitudL1(vectR, vectG, vectB, 8, 14));
		uPeque[1] = Z_shape(D[17], a, b);

		rs[4] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//S
		//D[18] = (MagnitudL1(vectR, vectG, vectB, 12, 17));
		//es D[13]
		uGran[0] = S_shape(D[13], a, b);
		//es D[7]
		//D[26] = (MagnitudL1(vectR, vectG, vectB, 12, 11));
		uGran[1] = S_shape(D[7], a, b);
		//es D[6]
		//D[27] = (MagnitudL1(vectR, vectG, vectB, 12, 13));
		uGran[2] = S_shape(D[6], a, b);

		D[18] = (MagnitudL1(vectR, vectG, vectB, 11, 16));
		uPeque[0] = Z_shape(D[18], a, b);

		D[19] = (MagnitudL1(vectR, vectG, vectB, 13, 18));
		uPeque[1] = Z_shape(D[19], a, b);

		rs[5] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//SW
		//es D[2]
		//D[30] = (MagnitudL1(vectR, vectG, vectB, 12, 16));
		uGran[0] = S_shape(D[2], a, b);
		//es D[0]
		//D[31] = (MagnitudL1(vectR, vectG, vectB, 12, 6));
		uGran[1] = S_shape(D[0], a, b);
		//es D[10]
		//D[32] = (MagnitudL1(vectR, vectG, vectB, 12, 18));
		uGran[2] = S_shape(D[10], a, b);

		D[20] = (MagnitudL1(vectR, vectG, vectB, 6, 10));
		uPeque[0] = Z_shape(D[20], a, b);

		D[21] = (MagnitudL1(vectR, vectG, vectB, 18, 22));
		uPeque[1] = Z_shape(D[21], a, b);

		rs[6] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		//W
		//Es D[7]
		//D[35] = (MagnitudL1(vectR, vectG, vectB, 12, 11));
		uGran[0] = S_shape(D[7], a, b);
		//Es D[5]
		//D[36] = (MagnitudL1(vectR, vectG, vectB, 12, 7));
		uGran[1] = S_shape(D[5], a, b);
		//es D[13]
		//D[37] = (MagnitudL1(vectR, vectG, vectB, 12, 17));
		uGran[2] = S_shape(D[13], a, b);

		D[21] = (MagnitudL1(vectR, vectG, vectB, 6, 7));
		uPeque[0] = Z_shape(D[21], a, b);

		D[22] = (MagnitudL1(vectR, vectG, vectB, 16, 17));
		uPeque[1] = Z_shape(D[22], a, b);

		rs[7] = uGran[0] * uGran[1] * uGran[2] * uPeque[0] * uPeque[1];

		mn = rs[0];
		r = rs[0];

		for (i = 0; i <= 7; i++)
		{
			if (r<rs[i])
			{
				r = rs[i];

			}
		}
		//Filtro VMF
		if (r > THS) {

			D[23] = (MagnitudL1(vectR, vectG, vectB, 6, 8));
			D[24] = (MagnitudL1(vectR, vectG, vectB, 6, 13));
			D[25] = (MagnitudL1(vectR, vectG, vectB, 6, 16));
			D[26] = (MagnitudL1(vectR, vectG, vectB, 6, 17));
			D[27] = (MagnitudL1(vectR, vectG, vectB, 6, 18));

			disteucl1[0] = D[0] + D[8] + D[21] + D[23] + D[24] + D[25] + D[26] + D[27];

			D[28] = (MagnitudL1(vectR, vectG, vectB, 7, 11));
			D[29] = (MagnitudL1(vectR, vectG, vectB, 7, 13));
			D[30] = (MagnitudL1(vectR, vectG, vectB, 7, 16));
			D[31] = (MagnitudL1(vectR, vectG, vectB, 7, 17));
			D[32] = (MagnitudL1(vectR, vectG, vectB, 7, 18));
			disteucl1[1] = D[21] + D[5] + D[14] + D[28] + D[29] + D[30] + D[31] + D[32];

			//es D[26] D[33] = (MagnitudL1(vectR, vectG, vectB, 8, 6));
			D[33] = (MagnitudL1(vectR, vectG, vectB, 8, 11));
			D[34] = (MagnitudL1(vectR, vectG, vectB, 8, 16));
			D[35] = (MagnitudL1(vectR, vectG, vectB, 8, 17));
			D[36] = (MagnitudL1(vectR, vectG, vectB, 8, 18));
			disteucl1[2] = D[14] + D[1] + D[9] + D[26] + D[33] + D[34] + D[35] + D[36];

			//es D[28]  D[37] = (MagnitudL1(vectR, vectG, vectB, 11, 7));
			//es D[33]     D[38] = (MagnitudL1(vectR, vectG, vectB, 11, 8)); 
			D[37] = (MagnitudL1(vectR, vectG, vectB, 11, 13));
			D[38] = (MagnitudL1(vectR, vectG, vectB, 11, 17));
			D[39] = (MagnitudL1(vectR, vectG, vectB, 11, 18));
			disteucl1[3] = D[7] + D[8] + D[18] + D[28] + D[33] + D[37] + D[38] + D[39];

			//Central ya estan todas las d calculadas
			disteucl1[4] = D[0] + D[5] + D[1] + D[7] + D[6] + D[2] + D[13] + D[10];

			D[40] = (MagnitudL1(vectR, vectG, vectB, 13, 16));
			D[41] = (MagnitudL1(vectR, vectG, vectB, 13, 17));
			disteucl1[5] = D[6] + D[19] + D[9] + D[24] + D[29] + D[37] + D[40] + D[41];

			D[42] = (MagnitudL1(vectR, vectG, vectB, 16, 18));
			disteucl1[6] = D[18] + D[2] + D[22] + D[25] + D[30] + D[34] + D[40] + D[42];

			disteucl1[7] = D[22] + D[13] + D[15] + D[26] + D[31] + D[35] + D[38] + D[41];

			disteucl1[8] = D[19] + D[10] + D[15] + D[27] + D[32] + D[36] + D[39] + D[42];

			posMin = 6;
			mn = disteucl1[0];
			for (i = 0; i <= 7; i++) {
				if (mn>disteucl1[i]) {
					mn = disteucl1[i];
					posMin = posicion[i];
				}
			}

			d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
			/*
			d_Pout[(Row * m + Col) * 3 + 0] = 255;
			d_Pout[(Row * m + Col) * 3 + 1] = 255;
			d_Pout[(Row * m + Col) * 3 + 2] = 255;
			*/
		}
		else {
			// si no es ruido la salida el el pixel central de la ventana
			d_Pout[(Row * m + Col) * 3 + 0] = vectR[12];
			d_Pout[(Row * m + Col) * 3 + 1] = vectG[12];
			d_Pout[(Row * m + Col) * 3 + 2] = vectB[12];
		}
	}



}


__global__ void VMF_Global_TwoPixels(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	unsigned char vectR[9], vectG[9], vectB[9];
	float disteucl = 0.0, disteucl1[9], hold;
	float D[40];
	float mn, mx;
	int posMin = 0;


	if ((Row < m - 1) && (Col < n - 1)) {
		F = 0;

		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		//D[0]=Magnitud(vectR, vectG, vectB, i, j//i==0 y j==0 no se hace
		D[0] = (Magnitud(vectR, vectG, vectB, 0, 1));
		D[1] = (Magnitud(vectR, vectG, vectB, 0, 2));
		D[2] = (Magnitud(vectR, vectG, vectB, 0, 3));
		D[3] = (Magnitud(vectR, vectG, vectB, 0, 4));
		D[4] = (Magnitud(vectR, vectG, vectB, 0, 5));
		D[5] = (Magnitud(vectR, vectG, vectB, 0, 6));
		D[6] = (Magnitud(vectR, vectG, vectB, 0, 7));
		D[7] = (Magnitud(vectR, vectG, vectB, 0, 8));
		disteucl1[0] = D[0] + D[1] + D[2] + D[3] + D[4] + D[5] + D[6] + D[7];

		//i=1,j=0 ya esta es D[0]
		//i=1,j=1 No se hace
		D[8] = (Magnitud(vectR, vectG, vectB, 1, 2));
		D[9] = (Magnitud(vectR, vectG, vectB, 1, 3));
		D[10] = (Magnitud(vectR, vectG, vectB, 1, 4));
		D[11] = (Magnitud(vectR, vectG, vectB, 1, 5));
		D[12] = (Magnitud(vectR, vectG, vectB, 1, 6));
		D[13] = (Magnitud(vectR, vectG, vectB, 1, 7));
		D[14] = (Magnitud(vectR, vectG, vectB, 1, 8));
		disteucl1[1] = D[0] + D[8] + D[9] + D[10] + D[11] + D[12] + D[13] + D[14];

		//i=2,j=0 ya esta es D[1]
		//i=2,j=1 ya esta es D[8]
		//i=2,j=2 No se hace
		D[15] = (Magnitud(vectR, vectG, vectB, 2, 3));
		D[16] = (Magnitud(vectR, vectG, vectB, 2, 4));
		D[17] = (Magnitud(vectR, vectG, vectB, 2, 5));
		D[18] = (Magnitud(vectR, vectG, vectB, 2, 6));
		D[19] = (Magnitud(vectR, vectG, vectB, 2, 7));
		D[20] = (Magnitud(vectR, vectG, vectB, 2, 8));
		disteucl1[2] = D[1] + D[8] + D[15] + D[16] + D[17] + D[18] + D[19] + D[20];

		//i=3,j=0 ya esta es D[2]
		//i=3,j=1 ya esta es D[9]
		//i=3,j=2 ya esta es D[15]
		//i=3,j=3 No se hace
		D[21] = (Magnitud(vectR, vectG, vectB, 3, 4));
		D[22] = (Magnitud(vectR, vectG, vectB, 3, 5));
		D[23] = (Magnitud(vectR, vectG, vectB, 3, 6));
		D[24] = (Magnitud(vectR, vectG, vectB, 3, 7));
		D[25] = (Magnitud(vectR, vectG, vectB, 3, 8));
		disteucl1[3] = D[2] + D[9] + D[15] + D[21] + D[22] + D[23] + D[24] + D[25];

		//i=4,j=0 ya esta es D[3]
		//i=4,j=1 ya esta es D[10]
		//i=4,j=2 ya esta es D[16]
		//i=4,j=3 ya esta es D[21]
		//i=4,j=4 No se hace
		D[26] = (Magnitud(vectR, vectG, vectB, 4, 5));
		D[27] = (Magnitud(vectR, vectG, vectB, 4, 6));
		D[28] = (Magnitud(vectR, vectG, vectB, 4, 7));
		D[29] = (Magnitud(vectR, vectG, vectB, 4, 8));
		disteucl1[4] = D[3] + D[10] + D[16] + D[21] + D[26] + D[27] + D[28] + D[29];

		//i=5,j=0 ya esta es D[4]
		//i=5,j=1 ya esta es D[11]
		//i=5,j=2 ya esta es D[17]
		//i=5,j=3 ya esta es D[22]
		//i=5,j=4 ya esta es D[26]
		//i=5,j=5 No se hace
		D[30] = (Magnitud(vectR, vectG, vectB, 5, 6));
		D[31] = (Magnitud(vectR, vectG, vectB, 5, 7));
		D[32] = (Magnitud(vectR, vectG, vectB, 5, 8));
		disteucl1[5] = D[4] + D[11] + D[17] + D[22] + D[26] + D[30] + D[31] + D[32];

		//i=6,j=0 ya esta es D[5]
		//i=6,j=1 ya esta es D[12]
		//i=6,j=2 ya esta es D[18]
		//i=6,j=3 ya esta es D[23]
		//i=6,j=4 ya esta es D[27]
		//i=6,j=5 ya esta es D[30]
		//i=6,j=6 No se hace
		D[33] = (Magnitud(vectR, vectG, vectB, 6, 7));
		D[34] = (Magnitud(vectR, vectG, vectB, 6, 8));
		disteucl1[6] = D[5] + D[12] + D[18] + D[23] + D[27] + D[30] + D[33] + D[34];

		//i=7,j=0 ya esta es D[6]
		//i=7,j=1 ya esta es D[13]
		//i=7,j=2 ya esta es D[19]
		//i=7,j=3 ya esta es D[24]
		//i=7,j=4 ya esta es D[28]
		//i=7,j=5 ya esta es D[31]
		//i=7,j=6 ya esta es D[33]
		//i=7,j=7 No se hace
		D[35] = (Magnitud(vectR, vectG, vectB, 7, 8));
		disteucl1[7] = D[6] + D[13] + D[19] + D[24] + D[28] + D[31] + D[33] + D[35];

		//i=8,j=0 ya esta es D[7]
		//i=8,j=1 ya esta es D[14]
		//i=8,j=2 ya esta es D[20]
		//i=8,j=3 ya esta es D[25]
		//i=8,j=4 ya esta es D[29]
		//i=8,j=5 ya esta es D[32]
		//i=8,j=6 ya esta es D[34]
		//i=8,j=7 ya esta es D[35]
		//i=8,j=8 No se hace
		disteucl1[8] = D[7] + D[14] + D[20] + D[25] + D[29] + D[32] + D[34] + D[35];

		mn = disteucl1[0];
		mx = disteucl1[0];

		posMin = 0;

		for (int i = 0; i<8; i++)
		{
			if (mn>disteucl1[i])
			{
				mn = disteucl1[i];
				posMin = posicion[i];
			}
			else if (mx<disteucl1[i])
			{

			}
		}

		d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
		d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
		d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];
	}
}


__global__ void VectorUnit_GPU_Global(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	float pixel_UR[9], pixel_UG[9], pixel_UB[9];
	unsigned char vectR[9], vectG[9], vectB[9];// esta comentado por el sqrt
	float disteucl = 0.0, disteucl1[9], hold;
	float valMag;
	float mn, mx, AuxResta = 0, aux1 = 0, aux2 = 0, aux3 = 0;
	int posMin = 0;


	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {
		

		F = 0;

		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];

				pixel_UR[F] = vectR[F];
				pixel_UG[F] = vectG[F];
				pixel_UB[F] = vectB[F];

				if (pixel_UR[F] == 0 && pixel_UG[F] == 0 && pixel_UB[F] == 0) {
					pixel_UR[F] = 10;
					pixel_UG[F] = 10;
					pixel_UB[F] = 10;
 
				}

				else {
					valMag = sqrtf((pixel_UR[F] * pixel_UR[F]) + (pixel_UG[F] * pixel_UG[F]) + (pixel_UB[F] * pixel_UB[F]));
					pixel_UR[F] = pixel_UR[F] / valMag;
					pixel_UG[F] = pixel_UG[F] / valMag;
					pixel_UB[F] = pixel_UB[F] / valMag;
				}
				posicion[F] = F;

				F++;
			}
		}

		disteucl = 0;
		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 8; x++) {
				//disteucl += abs(vectB[F]-vectB[x])+abs(vectG[F]-vectG[x])+abs(vectR[F]-vectR[x]);

				//disteucl += sqrtf( powf(pixel_UR[F] - pixel_UR[x],2)
				//			   +  powf(pixel_UG[F] - pixel_UG[x],2)
				//			   +  powf(pixel_UB[F] - pixel_UB[x],2) );

				
				//disteucl += ( fabsf(pixel_UR[F] - pixel_UR[x])
				//+ fabsf(pixel_UG[F] - pixel_UG[x])
				//+ fabsf(pixel_UB[F] - pixel_UB[x]));
				
				aux1 = pixel_UR[F] - pixel_UR[x];
				aux2 = pixel_UG[F] - pixel_UG[x];
				aux3 = pixel_UB[F] - pixel_UB[x];
				disteucl += sqrt((aux1*aux1) + (aux2*aux2) + (aux3*aux3));
				
				
				
				//disteucl += sqrt(pow(pixel_UR[F] - pixel_UR[x], 2)
				//+ pow(pixel_UG[F] - pixel_UG[x], 2)
				//+ pow(pixel_UB[F] - pixel_UB[x], 2));
				
			}
			disteucl1[F] = disteucl;
			disteucl = 0;
		}


		mn = disteucl1[0];
		mx = disteucl1[0];

		posMin = 0;

		for (int i = 0; i<8; i++)
		{
			if (mn>disteucl1[i])
			{
				mn = disteucl1[i];
				posMin = posicion[i];
			}
			else if (mx<disteucl1[i])
			{

			}
		}

		d_Pout[(Row * m + Col) * 3 + 0] = vectR[posMin];
		d_Pout[(Row * m + Col) * 3 + 1] = vectG[posMin];
		d_Pout[(Row * m + Col) * 3 + 2] = vectB[posMin];

		
	}

	

}

__global__ void BVDF_GPU_Global(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int x = 0, posicion[9], hold2 = 0, F = 0;
	//double vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	//double disteucl, disteucl1[9], hold;
	//double vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	//double disteucl, disteucl1[9], hold;
	//double	arriva = 0, abajo = 0, valAngulo = 0.0, auxCos = 0;

	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float	arriva = 0, abajo = 0, valAngulo = 0.0, auxCos = 0;


	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {
		//hacer el arreglo
		F = 0;
		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}
		valAngulo = 0;
		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 8; x++) {

				if ((vectR[F] == 0 && vectG[F] == 0 && vectB[F] == 0) || (vectR[x] == 0 && vectG[x] == 0 && vectB[x] == 0)) {
					// Es pixelZero
					valAngulo += 1000;

				}

				else {
					arriva = (vectR[F] * vectR[x]) + (vectG[F] * vectG[x]) + (vectB[F] * vectB[x]);
					abajo = sqrt((vectR[F] * vectR[F]) + (vectG[F] * vectG[F]) + (vectB[F] * vectB[F])) * sqrt((vectR[x] * vectR[x]) + (vectG[x] * vectG[x]) + (vectB[x] * vectB[x]));

					//if (abajo == 0)		abajo = .01;  //si abajo=o da inf
					//if (arriva == 0)	arriva= .01;  //si abajo=o da inf

					valAngulo += acos(arriva / abajo);
					//valAngulo += __cosf(arriva / abajo);
				}
			

			}

			disteucl1[F] = valAngulo;	
			valAngulo = 0;
		}
		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 7; x++) {
				if (disteucl1[x] > disteucl1[x + 1]) {
					hold = disteucl1[x];
					hold2 = posicion[x];
					disteucl1[x] = disteucl1[x + 1];
					posicion[x] = posicion[x + 1];
					disteucl1[x + 1] = hold;
					posicion[x + 1] = hold2;
				}
			}
		}

	
		d_Pout[(Row * n + Col) * 3 + 0] = vectR[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 1] = vectG[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 2] = vectB[posicion[0]];
	
		/*
		d_Pout[(Row * n + Col) * 3 + 0] = 255;
		d_Pout[(Row * n + Col) * 3 + 1] = 255;
		d_Pout[(Row * n + Col) * 3 + 2] = 255;
		*/


	}



}



#define min(a, b) ((a < b) ? a : b)
#define max(a, b) ((a > b) ? a : b) //estas dos funciones estan repetidas con minCUDA y maxCUDA

__global__ void FTSCF_GPU_Original
(unsigned char* d_Pout, const unsigned char* d_Pin, int n, int m) {

	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int M = 0, j = 0, x = 0;
	float vectR[9], vectG[9], vectB[9], hold;

	float gam_small_1[18] = { 0 }, med_1, med_2, var_1, gam_big_1[18] = { 0 };
	float gam_small_2[18] = { 0 }, med1, med2, var1, gam_big_2[18] = { 0 };

	float array_R[25];
	float array_G[25];
	float array_B[25];

	int F = 0, i = 0;

	const int channels = 3;

	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {

		
		
			//int tid = omp_get_thread_num();
			//hacer el arreglo
			F = 0;
			
			for (i = -2; i <= 2; i++) {
				for (j = -2; j <= 2; j++) {
					array_R[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
					array_G[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
					array_B[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];
					F++;
				}
			}


			// se copia a continuacion solo los 8-vecinos
			M = 0;
			for (F = 6; F <= 8; F++) {
				vectG[M] = (array_G[F]);
				vectR[M] = (array_R[F]);
				vectB[M] = (array_B[F]);
				M++;
			}
			for (F = 11; F <= 13; F++) {
				vectG[M] = (array_G[F]);
				vectR[M] = (array_R[F]);
				vectB[M] = (array_B[F]);
				M++;
			}
			for (F = 16; F <= 18; F++) {
				vectG[M] = (array_G[F]);
				vectR[M] = (array_R[F]);
				vectB[M] = (array_B[F]);
				M++;
			}

			
			float noreste_C_R, noreste_N1_R, noreste_N2_R, sur_C_R, sur_N1_R, sur_N2_R, noroeste_C_R, noroeste_N1_R, noroeste_N2_R;
			float este_C_R, este_N1_R, este_N2_R, oeste_C_R, oeste_N1_R, oeste_N2_R, sureste_C_R, sureste_N1_R, sureste_N2_R;
			float norte_C_R, norte_N1_R, norte_N2_R, suroeste_C_R, suroeste_N1_R, suroeste_N2_R;
			float suroeste_NW_R, suroeste_SE_R, sur_W_R, sur_E_R, sureste_SW_R, sureste_NE_R, este_S_R, este_N_R, noreste_SE_R, noreste_NW_R;
			float norte_W_R, norte_E_R, noroeste_NE_R, noroeste_SW_R, oeste_S_R, oeste_N_R;
			float noreste_C_G, noreste_N1_G, noreste_N2_G, sur_C_G, sur_N1_G, sur_N2_G, noroeste_C_G, noroeste_N1_G, noroeste_N2_G;
			float este_C_G, este_N1_G, este_N2_G, oeste_C_G, oeste_N1_G, oeste_N2_G, sureste_C_G, sureste_N1_G, sureste_N2_G;
			float norte_C_G, norte_N1_G, norte_N2_G, suroeste_C_G, suroeste_N1_G, suroeste_N2_G;
			float suroeste_NW_G, suroeste_SE_G, sur_W_G, sur_E_G, sureste_SW_G, sureste_NE_G, este_S_G, este_N_G, noreste_SE_G, noreste_NW_G;
			float norte_W_G, norte_E_G, noroeste_NE_G, noroeste_SW_G, oeste_S_G, oeste_N_G;
			float noreste_C_B, noreste_N1_B, noreste_N2_B, sur_C_B, sur_N1_B, sur_N2_B, noroeste_C_B, noroeste_N1_B, noroeste_N2_B;
			float este_C_B, este_N1_B, este_N2_B, oeste_C_B, oeste_N1_B, oeste_N2_B, sureste_C_B, sureste_N1_B, sureste_N2_B;
			float norte_C_B, norte_N1_B, norte_N2_B, suroeste_C_B, suroeste_N1_B, suroeste_N2_B;
			float suroeste_NW_B, suroeste_SE_B, sur_W_B, sur_E_B, sureste_SW_B, sureste_NE_B, este_S_B, este_N_B, noreste_SE_B, noreste_NW_B;
			float norte_W_B, norte_E_B, noroeste_NE_B, noroeste_SW_B, oeste_S_B, oeste_N_B;
			float largo[9], largo_1[9], largo_2[9], LARGO[9], LARGO_1[9], LARGO_2[9];
			float noise_R_R, noise_G_G, noise_B_B;
			int SW_C_B, SW_N1_B, SW_N2_B, SW_NW_B, SW_SE_B, S_C_B, S_N1_B, S_N2_B, S_W_B, S_E_B, SE_C_B, SE_N1_B, SE_N2_B, SE_SW_B, SE_NE_B;
			int E_C_B, E_N1_B, E_N2_B, E_S_B, E_N_B, NE_C_B, NE_N1_B, NE_N2_B, NE_SE_B, NE_NW_B, N_C_B, N_N1_B, N_N2_B, N_W_B, N_E_B;
			int NW_C_B, NW_N1_B, NW_N2_B, NW_NE_B, NW_SW_B, W_C_B, W_N1_B, W_N2_B, W_S_B, W_N_B;
			int SW_C_R, SW_N1_R, SW_N2_R, SW_NW_R, SW_SE_R, S_C_R, S_N1_R, S_N2_R, S_W_R, S_E_R, SE_C_R, SE_N1_R, SE_N2_R, SE_SW_R, SE_NE_R;
			int E_C_R, E_N1_R, E_N2_R, E_S_R, E_N_R, NE_C_R, NE_N1_R, NE_N2_R, NE_SE_R, NE_NW_R, N_C_R, N_N1_R, N_N2_R, N_W_R, N_E_R;
			int NW_C_R, NW_N1_R, NW_N2_R, NW_NE_R, NW_SW_R, W_C_R, W_N1_R, W_N2_R, W_S_R, W_N_R;
			int SW_C_G, SW_N1_G, SW_N2_G, SW_NW_G, SW_SE_G, S_C_G, S_N1_G, S_N2_G, S_W_G, S_E_G, SE_C_G, SE_N1_G, SE_N2_G, SE_SW_G, SE_NE_G;
			int E_C_G, E_N1_G, E_N2_G, E_S_G, E_N_G, NE_C_G, NE_N1_G, NE_N2_G, NE_SE_G, NE_NW_G, N_C_G, N_N1_G, N_N2_G, N_W_G, N_E_G;
			int NW_C_G, NW_N1_G, NW_N2_G, NW_NE_G, NW_SW_G, W_C_G, W_N1_G, W_N2_G, W_S_G, W_N_G;
			float cons1 = 255, cons2 = 255;
			
			
			// blue
			SW_C_B = abs(array_B[6] - array_B[12]);
			SW_N1_B = abs(array_B[10] - array_B[16]);
			SW_N2_B = abs(array_B[2] - array_B[8]);
			SW_NW_B = abs(array_B[12] - array_B[16]);
			SW_SE_B = abs(array_B[12] - array_B[8]);
			S_C_B = abs(array_B[7] - array_B[12]);
			S_N1_B = abs(array_B[6] - array_B[11]);
			S_N2_B = abs(array_B[8] - array_B[13]);
			S_W_B = abs(array_B[12] - array_B[11]);
			S_E_B = abs(array_B[12] - array_B[13]);
			SE_C_B = abs(array_B[8] - array_B[12]);
			SE_N1_B = abs(array_B[2] - array_B[6]);
			SE_N2_B = abs(array_B[14] - array_B[18]);
			SE_SW_B = abs(array_B[12] - array_B[6]);
			SE_NE_B = abs(array_B[12] - array_B[18]);
			E_C_B = abs(array_B[13] - array_B[12]);
			E_N1_B = abs(array_B[8] - array_B[7]);
			E_N2_B = abs(array_B[18] - array_B[17]);
			E_S_B = abs(array_B[12] - array_B[7]);
			E_N_B = abs(array_B[12] - array_B[17]);
			NE_C_B = abs(array_B[18] - array_B[12]);
			NE_N1_B = abs(array_B[14] - array_B[8]);
			NE_N2_B = abs(array_B[22] - array_B[16]);
			NE_SE_B = abs(array_B[12] - array_B[8]);
			NE_NW_B = abs(array_B[12] - array_B[16]);
			N_C_B = abs(array_B[17] - array_B[12]);
			N_N1_B = abs(array_B[18] - array_B[13]);
			N_N2_B = abs(array_B[16] - array_B[11]);
			N_W_B = abs(array_B[12] - array_B[11]);
			N_E_B = abs(array_B[12] - array_B[13]);
			NW_C_B = abs(array_B[16] - array_B[12]);
			NW_N1_B = abs(array_B[22] - array_B[18]);
			NW_N2_B = abs(array_B[10] - array_B[6]);
			NW_NE_B = abs(array_B[12] - array_B[18]);
			NW_SW_B = abs(array_B[12] - array_B[6]);
			W_C_B = abs(array_B[11] - array_B[12]);
			W_N1_B = abs(array_B[16] - array_B[17]);
			W_N2_B = abs(array_B[6] - array_B[7]);
			W_S_B = abs(array_B[12] - array_B[7]);
			W_N_B = abs(array_B[12] - array_B[17]);

			SW_C_G = abs(array_G[6] - array_G[12]);
			SW_N1_G = abs(array_G[10] - array_G[16]);
			SW_N2_G = abs(array_G[2] - array_G[8]);
			SW_NW_G = abs(array_G[12] - array_G[16]);
			SW_SE_G = abs(array_G[12] - array_G[8]);
			S_C_G = abs(array_G[7] - array_G[12]);
			S_N1_G = abs(array_G[6] - array_G[11]);
			S_N2_G = abs(array_G[8] - array_G[13]);
			S_W_G = abs(array_G[12] - array_G[11]);
			S_E_G = abs(array_G[12] - array_G[13]);
			SE_C_G = abs(array_G[8] - array_G[12]);
			SE_N1_G = abs(array_G[2] - array_G[6]);
			SE_N2_G = abs(array_G[14] - array_G[18]);
			SE_SW_G = abs(array_G[12] - array_G[6]);
			SE_NE_G = abs(array_G[12] - array_G[18]);
			E_C_G = abs(array_G[13] - array_G[12]);
			E_N1_G = abs(array_G[8] - array_G[7]);
			E_N2_G = abs(array_G[18] - array_G[17]);
			E_S_G = abs(array_G[12] - array_G[7]);
			E_N_G = abs(array_G[12] - array_G[17]);
			NE_C_G = abs(array_G[18] - array_G[12]);
			NE_N1_G = abs(array_G[14] - array_G[8]);
			NE_N2_G = abs(array_G[22] - array_G[16]);
			NE_SE_G = abs(array_G[12] - array_G[8]);
			NE_NW_G = abs(array_G[12] - array_G[16]);
			N_C_G = abs(array_G[17] - array_G[12]);
			N_N1_G = abs(array_G[18] - array_G[13]);
			N_N2_G = abs(array_G[16] - array_G[11]);
			N_W_G = abs(array_G[12] - array_G[11]);
			N_E_G = abs(array_G[12] - array_G[13]);
			NW_C_G = abs(array_G[16] - array_G[12]);
			NW_N1_G = abs(array_G[22] - array_G[18]);
			NW_N2_G = abs(array_G[10] - array_G[6]);
			NW_NE_G = abs(array_G[12] - array_G[18]);
			NW_SW_G = abs(array_G[12] - array_G[6]);
			W_C_G = abs(array_G[11] - array_G[12]);
			W_N1_G = abs(array_G[16] - array_G[17]);
			W_N2_G = abs(array_G[6] - array_G[7]);
			W_S_G = abs(array_G[12] - array_G[7]);
			W_N_G = abs(array_G[12] - array_G[17]);

			SW_C_R = abs(array_R[6] - array_R[12]);
			SW_N1_R = abs(array_R[10] - array_R[16]);
			SW_N2_R = abs(array_R[2] - array_R[8]);
			SW_NW_R = abs(array_R[12] - array_R[16]);
			SW_SE_R = abs(array_R[12] - array_R[8]);
			S_C_R = abs(array_R[7] - array_R[12]);
			S_N1_R = abs(array_R[6] - array_R[11]);
			S_N2_R = abs(array_R[8] - array_R[13]);
			S_W_R = abs(array_R[12] - array_R[11]);
			S_E_R = abs(array_R[12] - array_R[13]);
			SE_C_R = abs(array_R[8] - array_R[12]);
			SE_N1_R = abs(array_R[2] - array_R[6]);
			SE_N2_R = abs(array_R[14] - array_R[18]);
			SE_SW_R = abs(array_R[12] - array_R[6]);
			SE_NE_R = abs(array_R[12] - array_R[18]);
			E_C_R = abs(array_R[13] - array_R[12]);
			E_N1_R = abs(array_R[8] - array_R[7]);
			E_N2_R = abs(array_R[18] - array_R[17]);
			E_S_R = abs(array_R[12] - array_R[7]);
			E_N_R = abs(array_R[12] - array_R[17]);
			NE_C_R = abs(array_R[18] - array_R[12]);
			NE_N1_R = abs(array_R[14] - array_R[8]);
			NE_N2_R = abs(array_R[22] - array_R[16]);
			NE_SE_R = abs(array_R[12] - array_R[8]);
			NE_NW_R = abs(array_R[12] - array_R[16]);
			N_C_R = abs(array_R[17] - array_R[12]);
			N_N1_R = abs(array_R[18] - array_R[13]);
			N_N2_R = abs(array_R[16] - array_R[11]);
			N_W_R = abs(array_R[12] - array_R[11]);
			N_E_R = abs(array_R[12] - array_R[13]);
			NW_C_R = abs(array_R[16] - array_R[12]);
			NW_N1_R = abs(array_R[22] - array_R[18]);
			NW_N2_R = abs(array_R[10] - array_R[6]);
			NW_NE_R = abs(array_R[12] - array_R[18]);
			NW_SW_R = abs(array_R[12] - array_R[6]);
			W_C_R = abs(array_R[11] - array_R[12]);
			W_N1_R = abs(array_R[16] - array_R[17]);
			W_N2_R = abs(array_R[6] - array_R[7]);
			W_S_R = abs(array_R[12] - array_R[7]);
			W_N_R = abs(array_R[12] - array_R[17]);

			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) == 0) suroeste_C_R = 0;
			else	suroeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[10] * array_R[16])) == 0) suroeste_N1_R = 0;
			else   suroeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[10] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[2] * array_R[8])) == 0) suroeste_N2_R = 0;
			else   suroeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[2] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) == 0) suroeste_NW_R = 0;
			else	suroeste_NW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) == 0) suroeste_SE_R = 0;
			else	suroeste_SE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[7] * array_R[12])) == 0) sur_C_R = 0;
			else	sur_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[7] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[11])) == 0) sur_N1_R = 0;
			else	sur_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[13])) == 0) sur_N2_R = 0;
			else   sur_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) == 0) sur_W_R = 0;
			else	sur_W_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) == 0) sur_E_R = 0;
			else	sur_E_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[12])) == 0) sureste_C_R = 0;
			else	sureste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[2])) == 0) sureste_N1_R = 0;
			else	sureste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[18])) == 0) sureste_N2_R = 0;
			else	sureste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[6])) == 0) sureste_SW_R = 0;
			else	sureste_SW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) == 0) sureste_NE_R = 0;
			else	sureste_NE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[13] * array_R[12])) == 0) este_C_R = 0;
			else	este_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[13] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[7])) == 0) este_N1_R = 0;
			else	este_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[17])) == 0) este_N2_R = 0;
			else	este_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) == 0) este_S_R = 0;
			else	este_S_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) == 0) este_N_R = 0;
			else	este_N_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[12])) == 0) noreste_C_R = 0;
			else	noreste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[8])) == 0) noreste_N1_R = 0;
			else	noreste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[16])) == 0) noreste_N2_R = 0;
			else	noreste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) == 0) noreste_SE_R = 0;
			else	noreste_SE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) == 0) noreste_NW_R = 0;
			else	noreste_NW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[17] * array_R[12])) == 0) norte_C_R = 0;
			else	norte_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[17] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[13])) == 0) norte_N1_R = 0;
			else	norte_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[11])) == 0) norte_N2_R = 0;
			else	norte_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) == 0) norte_E_R = 0;
			else	norte_E_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) == 0) norte_W_R = 0;
			else	norte_W_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[12])) == 0) noroeste_C_R = 0;
			else	noroeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[18])) == 0) noroeste_N1_R = 0;
			else	noroeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[10])) == 0) noroeste_N2_R = 0;
			else	noroeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) == 0) noroeste_NE_R = 0;
			else	noroeste_NE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) == 0) noroeste_SW_R = 0;
			else	noroeste_SW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[11] * array_R[12])) == 0) oeste_C_R = 0;
			else	oeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[11] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[17])) == 0) oeste_N1_R = 0;
			else	oeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[7])) == 0) oeste_N2_R = 0;
			else	oeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) == 0) oeste_N_R = 0;
			else	oeste_N_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) == 0) oeste_S_R = 0;
			else	oeste_S_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));

			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) == 0) suroeste_C_G = 0;
			else	suroeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[10] * array_G[16])) == 0) suroeste_N1_G = 0;
			else   suroeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[10] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[2] * array_G[8])) == 0) suroeste_N2_G = 0;
			else   suroeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[2] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) == 0) suroeste_NW_G = 0;
			else	suroeste_NW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) == 0) suroeste_SE_G = 0;
			else	suroeste_SE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[7] * array_G[12])) == 0) sur_C_G = 0;
			else	sur_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[7] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[11])) == 0) sur_N1_G = 0;
			else	sur_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[13])) == 0) sur_N2_G = 0;
			else   sur_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) == 0) sur_W_G = 0;
			else	sur_W_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) == 0) sur_E_G = 0;
			else	sur_E_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[12])) == 0) sureste_C_G = 0;
			else	sureste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[2])) == 0) sureste_N1_G = 0;
			else	sureste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[18])) == 0) sureste_N2_G = 0;
			else	sureste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[6])) == 0) sureste_SW_G = 0;
			else	sureste_SW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) == 0) sureste_NE_G = 0;
			else	sureste_NE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[13] * array_G[12])) == 0) este_C_G = 0;
			else	este_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[13] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[7])) == 0) este_N1_G = 0;
			else	este_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[17])) == 0) este_N2_G = 0;
			else	este_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) == 0) este_S_G = 0;
			else	este_S_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) == 0) este_N_G = 0;
			else	este_N_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[12])) == 0) noreste_C_G = 0;
			else	noreste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[8])) == 0) noreste_N1_G = 0;
			else	noreste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[16])) == 0) noreste_N2_G = 0;
			else	noreste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) == 0) noreste_SE_G = 0;
			else	noreste_SE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) == 0) noreste_NW_G = 0;
			else	noreste_NW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[17] * array_G[12])) == 0) norte_C_G = 0;
			else	norte_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[17] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[13])) == 0) norte_N1_G = 0;
			else	norte_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[11])) == 0) norte_N2_G = 0;
			else	norte_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) == 0) norte_E_G = 0;
			else	norte_E_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) == 0) norte_W_G = 0;
			else	norte_W_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[12])) == 0) noroeste_C_G = 0;
			else	noroeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[18])) == 0) noroeste_N1_G = 0;
			else	noroeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[10])) == 0) noroeste_N2_G = 0;
			else	noroeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) == 0) noroeste_NE_G = 0;
			else	noroeste_NE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) == 0) noroeste_SW_G = 0;
			else	noroeste_SW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[11] * array_G[12])) == 0) oeste_C_G = 0;
			else	oeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[11] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[17])) == 0) oeste_N1_G = 0;
			else	oeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[7])) == 0) oeste_N2_G = 0;
			else	oeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) == 0) oeste_N_G = 0;
			else	oeste_N_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) == 0) oeste_S_G = 0;
			else	oeste_S_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));

			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) == 0) suroeste_C_B = 0;
			else	suroeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[10] * array_B[16])) == 0) suroeste_N1_B = 0;
			else   suroeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[10] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[2] * array_B[8])) == 0) suroeste_N2_B = 0;
			else   suroeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[2] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) == 0) suroeste_NW_B = 0;
			else	suroeste_NW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) == 0) suroeste_SE_B = 0;
			else	suroeste_SE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[7] * array_B[12])) == 0) sur_C_B = 0;
			else	sur_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[7] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2)))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[11])) == 0) sur_N1_B = 0;
			else	sur_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[13])) == 0) sur_N2_B = 0;
			else   sur_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) == 0) sur_W_B = 0;
			else	sur_W_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) == 0) sur_E_B = 0;
			else	sur_E_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[12])) == 0) sureste_C_B = 0;
			else	sureste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[2])) == 0) sureste_N1_B = 0;
			else	sureste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[18])) == 0) sureste_N2_B = 0;
			else	sureste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[6])) == 0) sureste_SW_B = 0;
			else	sureste_SW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) == 0) sureste_NE_B = 0;
			else	sureste_NE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[13] * array_B[12])) == 0) este_C_B = 0;
			else	este_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[13] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[7])) == 0) este_N1_B = 0;
			else	este_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[17])) == 0) este_N2_B = 0;
			else	este_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) == 0) este_S_B = 0;
			else	este_S_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) == 0) este_N_B = 0;
			else	este_N_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[12])) == 0) noreste_C_B = 0;
			else	noreste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[8])) == 0) noreste_N1_B = 0;
			else	noreste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[16])) == 0) noreste_N2_B = 0;
			else	noreste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) == 0) noreste_SE_B = 0;
			else	noreste_SE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) == 0) noreste_NW_B = 0;
			else	noreste_NW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[17] * array_B[12])) == 0) norte_C_B = 0;
			else	norte_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[17] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[13])) == 0) norte_N1_B = 0;
			else	norte_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[11])) == 0) norte_N2_B = 0;
			else	norte_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) == 0) norte_E_B = 0;
			else	norte_E_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) == 0) norte_W_B = 0;
			else	norte_W_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[12])) == 0) noroeste_C_B = 0;
			else	noroeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[18])) == 0) noroeste_N1_B = 0;
			else	noroeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[10])) == 0) noroeste_N2_B = 0;
			else	noroeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) == 0) noroeste_NE_B = 0;
			else	noroeste_NE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) == 0) noroeste_SW_B = 0;
			else	noroeste_SW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[11] * array_B[12])) == 0) oeste_C_B = 0;
			else	oeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[11] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[17])) == 0) oeste_N1_B = 0;
			else	oeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[7])) == 0) oeste_N2_B = 0;
			else	oeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) == 0) oeste_N_B = 0;
			else	oeste_N_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
			if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) == 0) oeste_S_B = 0;
			else	oeste_S_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
			//	SUROESTE	

			med_1 = 1, var_1 = 0.8;
			med_2 = 0.1;

			if (suroeste_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((suroeste_C_R)-med_1), 2) / (2 * var_1))));
			if (suroeste_N1_R < med_2) gam_small_1[0] = 1;
			else 	gam_small_1[0] = (exp(-(pow(((suroeste_N1_R)-med_2), 2) / (2 * var_1))));
			if (suroeste_N2_R < med_2) gam_small_1[1] = 1;
			else 	gam_small_1[1] = (exp(-(pow(((suroeste_N2_R)-med_2), 2) / (2 * var_1))));
			if (suroeste_NW_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_R)-med_1), 2) / (2 * var_1))));
			if (suroeste_SE_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_R)-med_1), 2) / (2 * var_1))));
			largo[0] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (sur_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sur_C_R)-med_1), 2) / (2 * var_1))));
			if (sur_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sur_N1_R)-med_2), 2) / (2 * var_1))));
			if (sur_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sur_N2_R)-med_2), 2) / (2 * var_1))));
			if (sur_W_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sur_W_R)-med_1), 2) / (2 * var_1))));
			if (sur_E_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sur_E_R)-med_1), 2) / (2 * var_1))));
			largo[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (sureste_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sureste_C_R)-med_1), 2) / (2 * var_1))));
			if (sureste_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sureste_N1_R)-med_2), 2) / (2 * var_1))));
			if (sureste_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sureste_N2_R)-med_2), 2) / (2 * var_1))));
			if (sureste_NE_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sureste_NE_R)-med_1), 2) / (2 * var_1))));
			if (sureste_SW_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sureste_SW_R)-med_1), 2) / (2 * var_1))));
			largo[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (este_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((este_C_R)-med_1), 2) / (2 * var_1))));
			if (este_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((este_N1_R)-med_2), 2) / (2 * var_1))));
			if (este_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((este_N2_R)-med_2), 2) / (2 * var_1))));
			if (este_N_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((este_N_R)-med_1), 2) / (2 * var_1))));
			if (este_S_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((este_S_R)-med_1), 2) / (2 * var_1))));
			largo[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noreste_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noreste_C_R)-med_1), 2) / (2 * var_1))));
			if (noreste_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noreste_N1_R)-med_2), 2) / (2 * var_1))));
			if (noreste_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noreste_N2_R)-med_2), 2) / (2 * var_1))));
			if (noreste_NW_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noreste_NW_R)-med_1), 2) / (2 * var_1))));
			if (noreste_SE_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noreste_SE_R)-med_1), 2) / (2 * var_1))));
			largo[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (norte_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((norte_C_R)-med_1), 2) / (2 * var_1))));
			if (norte_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((norte_N1_R)-med_2), 2) / (2 * var_1))));
			if (norte_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((norte_N2_R)-med_2), 2) / (2 * var_1))));
			if (norte_W_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((norte_W_R)-med_1), 2) / (2 * var_1))));
			if (norte_E_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((norte_E_R)-med_1), 2) / (2 * var_1))));
			largo[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noroeste_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noroeste_C_R)-med_1), 2) / (2 * var_1))));
			if (noroeste_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_R)-med_2), 2) / (2 * var_1))));
			if (noroeste_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_R)-med_2), 2) / (2 * var_1))));
			if (noroeste_NE_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_R)-med_1), 2) / (2 * var_1))));
			if (noroeste_SW_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_R)-med_1), 2) / (2 * var_1))));
			largo[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (oeste_C_R > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((oeste_C_R)-med_1), 2) / (2 * var_1))));
			if (oeste_N1_R < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((oeste_N1_R)-med_2), 2) / (2 * var_1))));
			if (oeste_N2_R < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((oeste_N2_R)-med_2), 2) / (2 * var_1))));
			if (oeste_N_R > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((oeste_N_R)-med_1), 2) / (2 * var_1))));
			if (oeste_S_R > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((oeste_S_R)-med_1), 2) / (2 * var_1))));
			largo[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (suroeste_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((suroeste_C_G)-med_1), 2) / (2 * var_1))));
			if (suroeste_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((suroeste_N1_G)-med_2), 2) / (2 * var_1))));
			if (suroeste_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((suroeste_N2_G)-med_2), 2) / (2 * var_1))));
			if (suroeste_NW_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_G)-med_1), 2) / (2 * var_1))));
			if (suroeste_SE_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_G)-med_1), 2) / (2 * var_1))));
			largo_1[0] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (sur_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sur_C_G)-med_1), 2) / (2 * var_1))));
			if (sur_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sur_N1_G)-med_2), 2) / (2 * var_1))));
			if (sur_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sur_N2_G)-med_2), 2) / (2 * var_1))));
			if (sur_W_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sur_W_G)-med_1), 2) / (2 * var_1))));
			if (sur_E_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sur_E_G)-med_1), 2) / (2 * var_1))));
			largo_1[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (sureste_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sureste_C_G)-med_1), 2) / (2 * var_1))));
			if (sureste_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sureste_N1_G)-med_2), 2) / (2 * var_1))));
			if (sureste_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sureste_N2_G)-med_2), 2) / (2 * var_1))));
			if (sureste_NE_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sureste_NE_G)-med_1), 2) / (2 * var_1))));
			if (sureste_SW_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sureste_SW_G)-med_1), 2) / (2 * var_1))));
			largo_1[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (este_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((este_C_G)-med_1), 2) / (2 * var_1))));
			if (este_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((este_N1_G)-med_2), 2) / (2 * var_1))));
			if (este_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((este_N2_G)-med_2), 2) / (2 * var_1))));
			if (este_N_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((este_N_G)-med_1), 2) / (2 * var_1))));
			if (este_S_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((este_S_G)-med_1), 2) / (2 * var_1))));
			largo_1[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noreste_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noreste_C_G)-med_1), 2) / (2 * var_1))));
			if (noreste_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noreste_N1_G)-med_2), 2) / (2 * var_1))));
			if (noreste_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noreste_N2_G)-med_2), 2) / (2 * var_1))));
			if (noreste_NW_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noreste_NW_G)-med_1), 2) / (2 * var_1))));
			if (noreste_SE_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noreste_SE_G)-med_1), 2) / (2 * var_1))));
			largo_1[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (norte_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((norte_C_G)-med_1), 2) / (2 * var_1))));
			if (norte_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((norte_N1_G)-med_2), 2) / (2 * var_1))));
			if (norte_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((norte_N2_G)-med_2), 2) / (2 * var_1))));
			if (norte_W_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((norte_W_G)-med_1), 2) / (2 * var_1))));
			if (norte_E_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((norte_E_G)-med_1), 2) / (2 * var_1))));
			largo_1[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noroeste_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noroeste_C_G)-med_1), 2) / (2 * var_1))));
			if (noroeste_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_G)-med_2), 2) / (2 * var_1))));
			if (noroeste_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_G)-med_2), 2) / (2 * var_1))));
			if (noroeste_NE_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_G)-med_1), 2) / (2 * var_1))));
			if (noroeste_SW_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_G)-med_1), 2) / (2 * var_1))));
			largo_1[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (oeste_C_G > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((oeste_C_G)-med_1), 2) / (2 * var_1))));
			if (oeste_N1_G < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((oeste_N1_G)-med_2), 2) / (2 * var_1))));
			if (oeste_N2_G < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((oeste_N2_G)-med_2), 2) / (2 * var_1))));
			if (oeste_N_G > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((oeste_N_G)-med_1), 2) / (2 * var_1))));
			if (oeste_S_G > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((oeste_S_G)-med_1), 2) / (2 * var_1))));
			largo_1[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (suroeste_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((suroeste_C_B)-med_1), 2) / (2 * var_1))));
			if (suroeste_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((suroeste_N1_B)-med_2), 2) / (2 * var_1))));
			if (suroeste_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((suroeste_N2_B)-med_2), 2) / (2 * var_1))));
			if (suroeste_NW_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_B)-med_1), 2) / (2 * var_1))));
			if (suroeste_SE_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_B)-med_1), 2) / (2 * var_1))));
			largo_2[0] = (gam_big_2[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_2[2]);
			if (sur_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sur_C_B)-med_1), 2) / (2 * var_1))));
			if (sur_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sur_N1_B)-med_2), 2) / (2 * var_1))));
			if (sur_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sur_N2_B)-med_2), 2) / (2 * var_1))));
			if (sur_W_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sur_W_B)-med_1), 2) / (2 * var_1))));
			if (sur_E_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sur_E_B)-med_1), 2) / (2 * var_1))));
			largo_2[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (sureste_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((sureste_C_B)-med_1), 2) / (2 * var_1))));
			if (sureste_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((sureste_N1_B)-med_2), 2) / (2 * var_1))));
			if (sureste_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((sureste_N2_B)-med_2), 2) / (2 * var_1))));
			if (sureste_NE_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((sureste_NE_B)-med_1), 2) / (2 * var_1))));
			if (sureste_SW_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((sureste_SW_B)-med_1), 2) / (2 * var_1))));
			largo_2[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (este_C_B > med_1) gam_big_1[0] = 1;

			else	gam_big_1[0] = (exp(-(pow(((este_C_B)-med_1), 2) / (2 * var_1))));
			if (este_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((este_N1_B)-med_2), 2) / (2 * var_1))));
			if (este_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((este_N2_B)-med_2), 2) / (2 * var_1))));
			if (este_N_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((este_N_B)-med_1), 2) / (2 * var_1))));
			if (este_S_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((este_S_B)-med_1), 2) / (2 * var_1))));
			largo_2[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noreste_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noreste_C_B)-med_1), 2) / (2 * var_1))));
			if (noreste_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noreste_N1_B)-med_2), 2) / (2 * var_1))));
			if (noreste_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noreste_N2_B)-med_2), 2) / (2 * var_1))));
			if (noreste_NW_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noreste_NW_B)-med_1), 2) / (2 * var_1))));
			if (noreste_SE_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noreste_SE_B)-med_1), 2) / (2 * var_1))));
			largo_2[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (norte_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((norte_C_B)-med_1), 2) / (2 * var_1))));
			if (norte_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((norte_N1_B)-med_2), 2) / (2 * var_1))));
			if (norte_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((norte_N2_B)-med_2), 2) / (2 * var_1))));
			if (norte_W_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((norte_W_B)-med_1), 2) / (2 * var_1))));
			if (norte_E_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((norte_E_B)-med_1), 2) / (2 * var_1))));
			largo_2[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (noroeste_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((noroeste_C_B)-med_1), 2) / (2 * var_1))));
			if (noroeste_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_B)-med_2), 2) / (2 * var_1))));
			if (noroeste_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_B)-med_2), 2) / (2 * var_1))));
			if (noroeste_NE_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_B)-med_1), 2) / (2 * var_1))));
			if (noroeste_SW_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_B)-med_1), 2) / (2 * var_1))));
			largo_2[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
			if (oeste_C_B > med_1) gam_big_1[0] = 1;
			else	gam_big_1[0] = (exp(-(pow(((oeste_C_B)-med_1), 2) / (2 * var_1))));
			if (oeste_N1_B < med_2) gam_small_1[0] = 1;
			else	gam_small_1[0] = (exp(-(pow(((oeste_N1_B)-med_2), 2) / (2 * var_1))));
			if (oeste_N2_B < med_2) gam_small_1[1] = 1;
			else	gam_small_1[1] = (exp(-(pow(((oeste_N2_B)-med_2), 2) / (2 * var_1))));
			if (oeste_N_B > med_1) gam_big_1[1] = 1;
			else	gam_big_1[1] = (exp(-(pow(((oeste_N_B)-med_1), 2) / (2 * var_1))));
			if (oeste_S_B > med_1) gam_big_1[2] = 1;
			else	gam_big_1[2] = (exp(-(pow(((oeste_S_B)-med_1), 2) / (2 * var_1))));
			largo_2[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);

			med1 = 60;
			med2 = 10;
			var1 = 1000;
			if (SW_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SW_C_R)-med1), 2) / (2 * var1))));
			if (SW_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SW_N1_R)-med2), 2) / (2 * var1))));
			if (SW_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SW_N2_R)-med2), 2) / (2 * var1))));
			if (SW_NW_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SW_NW_R)-med1), 2) / (2 * var1))));
			if (SW_SE_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SW_SE_R)-med1), 2) / (2 * var1))));
			LARGO[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (S_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((S_C_R)-med1), 2) / (2 * var1))));
			if (S_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((S_N1_R)-med2), 2) / (2 * var1))));
			if (S_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((S_N2_R)-med2), 2) / (2 * var1))));
			if (S_W_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((S_W_R)-med1), 2) / (2 * var1))));
			if (S_E_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((S_E_R)-med1), 2) / (2 * var1))));
			LARGO[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (SE_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SE_C_R)-med1), 2) / (2 * var1))));
			if (SE_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SE_N1_R)-med2), 2) / (2 * var1))));
			if (SE_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SE_N2_R)-med2), 2) / (2 * var1))));
			if (SE_NE_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SE_NE_R)-med1), 2) / (2 * var1))));
			if (SE_SW_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SE_SW_R)-med1), 2) / (2 * var1))));
			LARGO[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (E_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((E_C_R)-med1), 2) / (2 * var1))));
			if (E_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((E_N1_R)-med2), 2) / (2 * var1))));
			if (E_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((E_N2_R)-med2), 2) / (2 * var1))));
			if (E_N_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((E_N_R)-med1), 2) / (2 * var1))));
			if (E_S_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((E_S_R)-med1), 2) / (2 * var1))));
			LARGO[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NE_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NE_C_R)-med1), 2) / (2 * var1))));
			if (NE_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NE_N1_R)-med2), 2) / (2 * var1))));
			if (NE_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NE_N2_R)-med2), 2) / (2 * var1))));
			if (NE_NW_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NE_NW_R)-med1), 2) / (2 * var1))));
			if (NE_SE_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NE_SE_R)-med1), 2) / (2 * var1))));
			LARGO[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (N_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((N_C_R)-med1), 2) / (2 * var1))));
			if (N_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((N_N1_R)-med2), 2) / (2 * var1))));
			if (N_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((N_N2_R)-med2), 2) / (2 * var1))));
			if (N_W_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((N_W_R)-med1), 2) / (2 * var1))));
			if (N_E_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((N_E_R)-med1), 2) / (2 * var1))));
			LARGO[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NW_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NW_C_R)-med1), 2) / (2 * var1))));
			if (NW_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NW_N1_R)-med2), 2) / (2 * var1))));
			if (NW_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NW_N2_R)-med2), 2) / (2 * var1))));
			if (NW_NE_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NW_NE_R)-med1), 2) / (2 * var1))));
			if (NW_SW_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NW_SW_R)-med1), 2) / (2 * var1))));
			LARGO[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (W_C_R > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((W_C_R)-med1), 2) / (2 * var1))));
			if (W_N1_R < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((W_N1_R)-med2), 2) / (2 * var1))));
			if (W_N2_R < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((W_N2_R)-med2), 2) / (2 * var1))));
			if (W_N_R > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((W_N_R)-med1), 2) / (2 * var1))));
			if (W_S_R > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((W_S_R)-med1), 2) / (2 * var1))));
			LARGO[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (SW_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SW_C_G)-med1), 2) / (2 * var1))));
			if (SW_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SW_N1_G)-med2), 2) / (2 * var1))));
			if (SW_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SW_N2_G)-med2), 2) / (2 * var1))));
			if (SW_NW_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SW_NW_G)-med1), 2) / (2 * var1))));
			if (SW_SE_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SW_SE_G)-med1), 2) / (2 * var1))));
			LARGO_1[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (S_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((S_C_G)-med1), 2) / (2 * var1))));
			if (S_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((S_N1_G)-med2), 2) / (2 * var1))));
			if (S_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((S_N2_G)-med2), 2) / (2 * var1))));
			if (S_W_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((S_W_G)-med1), 2) / (2 * var1))));
			if (S_E_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((S_E_G)-med1), 2) / (2 * var1))));
			LARGO_1[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (SE_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SE_C_G)-med1), 2) / (2 * var1))));
			if (SE_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SE_N1_G)-med2), 2) / (2 * var1))));
			if (SE_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SE_N2_G)-med2), 2) / (2 * var1))));
			if (SE_NE_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SE_NE_G)-med1), 2) / (2 * var1))));
			if (SE_SW_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SE_SW_G)-med1), 2) / (2 * var1))));
			LARGO_1[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (E_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((E_C_G)-med1), 2) / (2 * var1))));
			if (E_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((E_N1_G)-med2), 2) / (2 * var1))));
			if (E_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((E_N2_G)-med2), 2) / (2 * var1))));
			if (E_N_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((E_N_G)-med1), 2) / (2 * var1))));
			if (E_S_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((E_S_G)-med1), 2) / (2 * var1))));
			LARGO_1[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NE_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NE_C_G)-med1), 2) / (2 * var1))));
			if (NE_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NE_N1_G)-med2), 2) / (2 * var1))));
			if (NE_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NE_N2_G)-med2), 2) / (2 * var1))));
			if (NE_NW_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NE_NW_G)-med1), 2) / (2 * var1))));
			if (NE_SE_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NE_SE_G)-med1), 2) / (2 * var1))));
			LARGO_1[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (N_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((N_C_G)-med1), 2) / (2 * var1))));
			if (N_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((N_N1_G)-med2), 2) / (2 * var1))));
			if (N_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((N_N2_G)-med2), 2) / (2 * var1))));
			if (N_W_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((N_W_G)-med1), 2) / (2 * var1))));
			if (N_E_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((N_E_G)-med1), 2) / (2 * var1))));
			LARGO_1[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NW_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NW_C_G)-med1), 2) / (2 * var1))));
			if (NW_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NW_N1_G)-med2), 2) / (2 * var1))));
			if (NW_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NW_N2_G)-med2), 2) / (2 * var1))));
			if (NW_NE_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NW_NE_G)-med1), 2) / (2 * var1))));
			if (NW_SW_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NW_SW_G)-med1), 2) / (2 * var1))));
			LARGO_1[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (W_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((W_C_G)-med1), 2) / (2 * var1))));
			if (W_N1_G < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((W_N1_G)-med2), 2) / (2 * var1))));
			if (W_N2_G < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((W_N2_G)-med2), 2) / (2 * var1))));
			if (W_N_G > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((W_N_G)-med1), 2) / (2 * var1))));
			if (W_S_G > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((W_S_G)-med1), 2) / (2 * var1))));
			LARGO_1[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (SW_C_G > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SW_C_B)-med1), 2) / (2 * var1))));
			if (SW_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SW_N1_B)-med2), 2) / (2 * var1))));
			if (SW_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SW_N2_B)-med2), 2) / (2 * var1))));
			if (SW_NW_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SW_NW_B)-med1), 2) / (2 * var1))));
			if (SW_SE_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SW_SE_B)-med1), 2) / (2 * var1))));
			LARGO_2[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (S_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((S_C_B)-med1), 2) / (2 * var1))));
			if (S_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((S_N1_B)-med2), 2) / (2 * var1))));
			if (S_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((S_N2_B)-med2), 2) / (2 * var1))));
			if (S_W_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((S_W_B)-med1), 2) / (2 * var1))));
			if (S_E_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((S_E_B)-med1), 2) / (2 * var1))));
			LARGO_2[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (SE_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((SE_C_B)-med1), 2) / (2 * var1))));
			if (SE_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((SE_N1_B)-med2), 2) / (2 * var1))));
			if (SE_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((SE_N2_B)-med2), 2) / (2 * var1))));
			if (SE_NE_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((SE_NE_B)-med1), 2) / (2 * var1))));
			if (SE_SW_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((SE_SW_B)-med1), 2) / (2 * var1))));
			LARGO_2[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (E_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((E_C_B)-med1), 2) / (2 * var1))));
			if (E_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((E_N1_B)-med2), 2) / (2 * var1))));
			if (E_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((E_N2_B)-med2), 2) / (2 * var1))));
			if (E_N_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((E_N_B)-med1), 2) / (2 * var1))));
			if (E_S_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((E_S_B)-med1), 2) / (2 * var1))));
			LARGO_2[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NE_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NE_C_B)-med1), 2) / (2 * var1))));
			if (NE_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NE_N1_B)-med2), 2) / (2 * var1))));
			if (NE_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NE_N2_B)-med2), 2) / (2 * var1))));
			if (NE_NW_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NE_NW_B)-med1), 2) / (2 * var1))));
			if (NE_SE_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NE_SE_B)-med1), 2) / (2 * var1))));
			LARGO_2[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (N_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((N_C_B)-med1), 2) / (2 * var1))));
			if (N_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((N_N1_B)-med2), 2) / (2 * var1))));
			if (N_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((N_N2_B)-med2), 2) / (2 * var1))));
			if (N_W_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((N_W_B)-med1), 2) / (2 * var1))));
			if (N_E_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((N_E_B)-med1), 2) / (2 * var1))));
			LARGO_2[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (NW_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((NW_C_B)-med1), 2) / (2 * var1))));
			if (NW_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((NW_N1_B)-med2), 2) / (2 * var1))));
			if (NW_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((NW_N2_B)-med2), 2) / (2 * var1))));
			if (NW_NE_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((NW_NE_B)-med1), 2) / (2 * var1))));
			if (NW_SW_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((NW_SW_B)-med1), 2) / (2 * var1))));
			LARGO_2[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
			if (W_C_B > med1) gam_big_2[0] = 1;
			else	gam_big_2[0] = (exp(-(pow(((W_C_B)-med1), 2) / (2 * var1))));
			if (W_N1_B < med2) gam_small_2[0] = 1;
			else	gam_small_2[0] = (exp(-(pow(((W_N1_B)-med2), 2) / (2 * var1))));
			if (W_N2_B < med2) gam_small_2[1] = 1;
			else	gam_small_2[1] = (exp(-(pow(((W_N2_B)-med2), 2) / (2 * var1))));
			if (W_N_B > med1) gam_big_2[1] = 1;
			else	gam_big_2[1] = (exp(-(pow(((W_N_B)-med1), 2) / (2 * var1))));
			if (W_S_B > med1) gam_big_2[2] = 1;
			else	gam_big_2[2] = (exp(-(pow(((W_S_B)-med1), 2) / (2 * var1))));
			LARGO_2[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);

			float	mu_R_R[8], mu_G_G[8], mu_B_B[8];

			mu_R_R[0] = min(largo[0], LARGO[0]);
			mu_R_R[1] = min(largo[1], LARGO[1]);
			mu_R_R[2] = min(largo[2], LARGO[2]);
			mu_R_R[3] = min(largo[3], LARGO[3]);
			mu_R_R[4] = min(largo[4], LARGO[4]);
			mu_R_R[5] = min(largo[5], LARGO[5]);
			mu_R_R[6] = min(largo[6], LARGO[6]);
			mu_R_R[7] = min(largo[7], LARGO[7]);

			mu_G_G[0] = min(largo_1[0], LARGO_1[0]);
			mu_G_G[1] = min(largo_1[1], LARGO_1[1]);
			mu_G_G[2] = min(largo_1[2], LARGO_1[2]);
			mu_G_G[3] = min(largo_1[3], LARGO_1[3]);
			mu_G_G[4] = min(largo_1[4], LARGO_1[4]);
			mu_G_G[5] = min(largo_1[5], LARGO_1[5]);
			mu_G_G[6] = min(largo_1[6], LARGO_1[6]);
			mu_G_G[7] = min(largo_1[7], LARGO_1[7]);

			mu_B_B[0] = min(largo_2[0], LARGO_2[0]);
			mu_B_B[1] = min(largo_2[1], LARGO_2[1]);
			mu_B_B[2] = min(largo_2[2], LARGO_2[2]);
			mu_B_B[3] = min(largo_2[3], LARGO_2[3]);
			mu_B_B[4] = min(largo_2[4], LARGO_2[4]);
			mu_B_B[5] = min(largo_2[5], LARGO_2[5]);
			mu_B_B[6] = min(largo_2[6], LARGO_2[6]);
			mu_B_B[7] = min(largo_2[7], LARGO_2[7]);

			noise_R_R = max(max(max(max(max(max(max(mu_R_R[0], mu_R_R[1]), mu_R_R[2]), mu_R_R[3]), mu_R_R[4]), mu_R_R[5]), mu_R_R[6]), mu_R_R[7]);
			noise_G_G = max(max(max(max(max(max(max(mu_G_G[0], mu_G_G[1]), mu_G_G[2]), mu_G_G[3]), mu_G_G[4]), mu_G_G[5]), mu_G_G[6]), mu_G_G[7]);
			noise_B_B = max(max(max(max(max(max(max(mu_B_B[0], mu_B_B[1]), mu_B_B[2]), mu_B_B[3]), mu_B_B[4]), mu_B_B[5]), mu_B_B[6]), mu_B_B[7]);

			//printf( "%f",noise_B_B);

			if ((noise_B_B >= 0.3))
			{

				float weights[9], sum_weights = 0, hold2, suma = 0;
				for (j = 0; j <= 7; j++)
				{
					sum_weights += (1 - mu_B_B[j]);
				}
				sum_weights = (sum_weights + 3 * sqrt(1 - noise_B_B)) / 2;
				weights[0] = (1 - mu_B_B[0]);
				weights[1] = (1 - mu_B_B[1]);
				weights[2] = (1 - mu_B_B[2]);
				weights[3] = (1 - mu_B_B[7]);
				weights[4] = 3 * sqrt(1 - noise_B_B);
				weights[5] = (1 - mu_B_B[3]);
				weights[6] = (1 - mu_B_B[6]);
				weights[7] = (1 - mu_B_B[5]);
				weights[8] = (1 - mu_B_B[4]);

				for (j = 0; j <= 8; j++)
				{
					for (x = 0; x <= 7; x++)
					{
						if (vectB[x] > vectB[x + 1])
						{
							hold = vectB[x];
							hold2 = weights[x];
							vectB[x] = vectB[x + 1];
							weights[x] = weights[x + 1];
							vectB[x + 1] = hold;
							weights[x + 1] = hold2;
						}
					}
				}
				for (j = 8; j >= 0; j--)
				{
					suma += weights[j];
					if (suma >= sum_weights)
					{
						if (j < 2)
						{
							sum_weights = sum_weights - (weights[0] + weights[1]);
							sum_weights = sum_weights / 2;
							suma = 0;
							for (F = 8; F >= 2; F--)
							{
								suma += weights[F];
								if (suma > sum_weights)
								{
									d_Pout[(Row * m + Col) * channels + 2] = vectB[F];
									F = -1;
								}
							}
							j = -1;
						}
						else
						{
							d_Pout[(Row * m + Col) * channels + 2] = vectB[j];
							//d_Pout[(Row * m + Col) * channels + 0] = d_Pout[(Row * m + Col) * channels + 0];
							j = -1;
						}
						suma = -1;
					}
				}
				//		fwrite (&CCC, 1, 1, header_file);
			}
			else
			{
				d_Pout[(Row * m + Col) * channels + 2] = vectB[4];
				//d_Pout[(Row * m + Col) * channels + 0] = 0;

				//		fwrite (&CCC, 1, 1, header_file);
			}

			if (noise_G_G >= 0.3)
			{

				float weights[9], sum_weights = 0, hold2, suma = 0;
				for (j = 0; j <= 7; j++)
				{
					sum_weights += (1 - mu_G_G[j]);
				}
				sum_weights = (sum_weights + 3 * sqrt(1 - noise_G_G)) / 2;
				weights[0] = (1 - mu_G_G[0]);
				weights[1] = (1 - mu_G_G[1]);
				weights[2] = (1 - mu_G_G[2]);
				weights[3] = (1 - mu_G_G[7]);
				weights[4] = 3 * sqrt(1 - noise_G_G);
				weights[5] = (1 - mu_G_G[3]);
				weights[6] = (1 - mu_G_G[6]);
				weights[7] = (1 - mu_G_G[5]);
				weights[8] = (1 - mu_G_G[4]);
				for (j = 0; j <= 8; j++)
				{
					for (x = 0; x <= 7; x++)
					{
						if (vectG[x] > vectG[x + 1])
						{
							hold = vectG[x];
							hold2 = weights[x];
							vectG[x] = vectG[x + 1];
							weights[x] = weights[x + 1];
							vectG[x + 1] = hold;
							weights[x + 1] = hold2;
						}
					}
				}
				for (j = 8; j >= 0; j--)
				{
					suma += weights[j];
					if (suma >= sum_weights)
					{
						if (j < 2)
						{
							sum_weights = sum_weights - (weights[0] + weights[1]);
							sum_weights = sum_weights / 2;
							suma = 0;
							for (F = 8; F >= 2; F--)
							{
								suma += weights[F];
								if (suma >= sum_weights)
								{
									d_Pout[(Row * m + Col) * channels + 1] = vectG[F];
									F = -1;
								}
							}
							j = -1;
						}
						else
						{
							d_Pout[(Row * m + Col) * channels + 1] = vectG[j];
							j = -1;
						}
						suma = -1;
					}
				}
				//		fwrite (&BBB, 1, 1, header_file);
			}
			else
			{
				d_Pout[(Row * m + Col) * channels + 1] = vectG[4];
				//		fwrite (&BBB, 1, 1, header_file);
			}

			if (noise_R_R >= 0.3)
			{

				float weights[9], sum_weights = 0, hold2, suma = 0;
				for (j = 0; j <= 7; j++)
				{
					sum_weights += (1 - mu_R_R[j]);
				}
				sum_weights = (sum_weights + 3 * sqrt(1 - noise_R_R)) / 2;
				weights[0] = (1 - mu_R_R[0]);
				weights[1] = (1 - mu_R_R[1]);
				weights[2] = (1 - mu_R_R[2]);
				weights[3] = (1 - mu_R_R[7]);
				weights[4] = 3 * sqrt(1 - noise_R_R);
				weights[5] = (1 - mu_R_R[3]);
				weights[6] = (1 - mu_R_R[6]);
				weights[7] = (1 - mu_R_R[5]);
				weights[8] = (1 - mu_R_R[4]);
				for (j = 0; j <= 8; j++)
				{
					for (x = 0; x <= 7; x++)
					{
						if (vectR[x] > vectR[x + 1])
						{
							hold = vectR[x];
							hold2 = weights[x];
							vectR[x] = vectR[x + 1];
							weights[x] = weights[x + 1];
							vectR[x + 1] = hold;
							weights[x + 1] = hold2;
						}
					}
				}
				for (j = 8; j >= 0; j--)
				{
					suma += weights[j];
					if (suma >= sum_weights)
					{
						if (j < 2)
						{
							sum_weights = sum_weights - (weights[0] + weights[1]);
							sum_weights = sum_weights / 2;
							suma = 0;
							for (F = 8; F >= 2; F--)
							{
								suma += weights[F];
								if (suma > sum_weights)
								{
									d_Pout[(Row * m + Col) * channels + 0] = vectR[F];
									F = -1;
								}
							}
							j = -1;
						}
						else
						{
							d_Pout[(Row * m + Col) * channels + 0] = vectR[j];
							j = -1;
						}
						suma = -1;
					}
				}
				//      fwrite (&AAA, 1, 1, header_file);
			}
			else
			{
				d_Pout[(Row * m + Col) * channels + 0] = vectR[4];
				//d_Pout[(Row * m + Col) * channels + 0] = 255;
				//		fwrite (&AAA, 1, 1, header_file);
			}
			
			
			//d_Pout[(Row * m + Col) * channels + 0] = 255;
		}
		
}

__global__ void FTSCF_GPU_Original_Params
(unsigned char* d_Pout, const unsigned char* d_Pin, int n, int m,
	
	float med_1, float var_1, float med_2, float med1, float med2, float var1, float THS) {

	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	int M = 0, j = 0, x = 0;
	float vectR[9], vectG[9], vectB[9], hold;

	float gam_small_1[18] = { 0 }, gam_big_1[18] = { 0 };
	float gam_small_2[18] = { 0 }, gam_big_2[18] = { 0 };

	float array_R[25];
	float array_G[25];
	float array_B[25];

	int F = 0, i = 0;

	const int channels = 3;

	if ((Row>1) && (Col>1) && (Row < m - 1) && (Col < n - 1)) {



		//int tid = omp_get_thread_num();
		//hacer el arreglo
		F = 0;

		for (i = -2; i <= 2; i++) {
			for (j = -2; j <= 2; j++) {
				array_R[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 0];
				array_G[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 1];
				array_B[F] = d_Pin[((Row + i) * m + (Col + j)) * 3 + 2];
				F++;
			}
		}


		// se copia a continuacion solo los 8-vecinos
		M = 0;
		for (F = 6; F <= 8; F++) {
			vectG[M] = (array_G[F]);
			vectR[M] = (array_R[F]);
			vectB[M] = (array_B[F]);
			M++;
		}
		for (F = 11; F <= 13; F++) {
			vectG[M] = (array_G[F]);
			vectR[M] = (array_R[F]);
			vectB[M] = (array_B[F]);
			M++;
		}
		for (F = 16; F <= 18; F++) {
			vectG[M] = (array_G[F]);
			vectR[M] = (array_R[F]);
			vectB[M] = (array_B[F]);
			M++;
		}


		float noreste_C_R, noreste_N1_R, noreste_N2_R, sur_C_R, sur_N1_R, sur_N2_R, noroeste_C_R, noroeste_N1_R, noroeste_N2_R;
		float este_C_R, este_N1_R, este_N2_R, oeste_C_R, oeste_N1_R, oeste_N2_R, sureste_C_R, sureste_N1_R, sureste_N2_R;
		float norte_C_R, norte_N1_R, norte_N2_R, suroeste_C_R, suroeste_N1_R, suroeste_N2_R;
		float suroeste_NW_R, suroeste_SE_R, sur_W_R, sur_E_R, sureste_SW_R, sureste_NE_R, este_S_R, este_N_R, noreste_SE_R, noreste_NW_R;
		float norte_W_R, norte_E_R, noroeste_NE_R, noroeste_SW_R, oeste_S_R, oeste_N_R;
		float noreste_C_G, noreste_N1_G, noreste_N2_G, sur_C_G, sur_N1_G, sur_N2_G, noroeste_C_G, noroeste_N1_G, noroeste_N2_G;
		float este_C_G, este_N1_G, este_N2_G, oeste_C_G, oeste_N1_G, oeste_N2_G, sureste_C_G, sureste_N1_G, sureste_N2_G;
		float norte_C_G, norte_N1_G, norte_N2_G, suroeste_C_G, suroeste_N1_G, suroeste_N2_G;
		float suroeste_NW_G, suroeste_SE_G, sur_W_G, sur_E_G, sureste_SW_G, sureste_NE_G, este_S_G, este_N_G, noreste_SE_G, noreste_NW_G;
		float norte_W_G, norte_E_G, noroeste_NE_G, noroeste_SW_G, oeste_S_G, oeste_N_G;
		float noreste_C_B, noreste_N1_B, noreste_N2_B, sur_C_B, sur_N1_B, sur_N2_B, noroeste_C_B, noroeste_N1_B, noroeste_N2_B;
		float este_C_B, este_N1_B, este_N2_B, oeste_C_B, oeste_N1_B, oeste_N2_B, sureste_C_B, sureste_N1_B, sureste_N2_B;
		float norte_C_B, norte_N1_B, norte_N2_B, suroeste_C_B, suroeste_N1_B, suroeste_N2_B;
		float suroeste_NW_B, suroeste_SE_B, sur_W_B, sur_E_B, sureste_SW_B, sureste_NE_B, este_S_B, este_N_B, noreste_SE_B, noreste_NW_B;
		float norte_W_B, norte_E_B, noroeste_NE_B, noroeste_SW_B, oeste_S_B, oeste_N_B;
		float largo[9], largo_1[9], largo_2[9], LARGO[9], LARGO_1[9], LARGO_2[9];
		float noise_R_R, noise_G_G, noise_B_B;
		int SW_C_B, SW_N1_B, SW_N2_B, SW_NW_B, SW_SE_B, S_C_B, S_N1_B, S_N2_B, S_W_B, S_E_B, SE_C_B, SE_N1_B, SE_N2_B, SE_SW_B, SE_NE_B;
		int E_C_B, E_N1_B, E_N2_B, E_S_B, E_N_B, NE_C_B, NE_N1_B, NE_N2_B, NE_SE_B, NE_NW_B, N_C_B, N_N1_B, N_N2_B, N_W_B, N_E_B;
		int NW_C_B, NW_N1_B, NW_N2_B, NW_NE_B, NW_SW_B, W_C_B, W_N1_B, W_N2_B, W_S_B, W_N_B;
		int SW_C_R, SW_N1_R, SW_N2_R, SW_NW_R, SW_SE_R, S_C_R, S_N1_R, S_N2_R, S_W_R, S_E_R, SE_C_R, SE_N1_R, SE_N2_R, SE_SW_R, SE_NE_R;
		int E_C_R, E_N1_R, E_N2_R, E_S_R, E_N_R, NE_C_R, NE_N1_R, NE_N2_R, NE_SE_R, NE_NW_R, N_C_R, N_N1_R, N_N2_R, N_W_R, N_E_R;
		int NW_C_R, NW_N1_R, NW_N2_R, NW_NE_R, NW_SW_R, W_C_R, W_N1_R, W_N2_R, W_S_R, W_N_R;
		int SW_C_G, SW_N1_G, SW_N2_G, SW_NW_G, SW_SE_G, S_C_G, S_N1_G, S_N2_G, S_W_G, S_E_G, SE_C_G, SE_N1_G, SE_N2_G, SE_SW_G, SE_NE_G;
		int E_C_G, E_N1_G, E_N2_G, E_S_G, E_N_G, NE_C_G, NE_N1_G, NE_N2_G, NE_SE_G, NE_NW_G, N_C_G, N_N1_G, N_N2_G, N_W_G, N_E_G;
		int NW_C_G, NW_N1_G, NW_N2_G, NW_NE_G, NW_SW_G, W_C_G, W_N1_G, W_N2_G, W_S_G, W_N_G;
		float cons1 = 255, cons2 = 255;


		// blue
		SW_C_B = abs(array_B[6] - array_B[12]);
		SW_N1_B = abs(array_B[10] - array_B[16]);
		SW_N2_B = abs(array_B[2] - array_B[8]);
		SW_NW_B = abs(array_B[12] - array_B[16]);
		SW_SE_B = abs(array_B[12] - array_B[8]);
		S_C_B = abs(array_B[7] - array_B[12]);
		S_N1_B = abs(array_B[6] - array_B[11]);
		S_N2_B = abs(array_B[8] - array_B[13]);
		S_W_B = abs(array_B[12] - array_B[11]);
		S_E_B = abs(array_B[12] - array_B[13]);
		SE_C_B = abs(array_B[8] - array_B[12]);
		SE_N1_B = abs(array_B[2] - array_B[6]);
		SE_N2_B = abs(array_B[14] - array_B[18]);
		SE_SW_B = abs(array_B[12] - array_B[6]);
		SE_NE_B = abs(array_B[12] - array_B[18]);
		E_C_B = abs(array_B[13] - array_B[12]);
		E_N1_B = abs(array_B[8] - array_B[7]);
		E_N2_B = abs(array_B[18] - array_B[17]);
		E_S_B = abs(array_B[12] - array_B[7]);
		E_N_B = abs(array_B[12] - array_B[17]);
		NE_C_B = abs(array_B[18] - array_B[12]);
		NE_N1_B = abs(array_B[14] - array_B[8]);
		NE_N2_B = abs(array_B[22] - array_B[16]);
		NE_SE_B = abs(array_B[12] - array_B[8]);
		NE_NW_B = abs(array_B[12] - array_B[16]);
		N_C_B = abs(array_B[17] - array_B[12]);
		N_N1_B = abs(array_B[18] - array_B[13]);
		N_N2_B = abs(array_B[16] - array_B[11]);
		N_W_B = abs(array_B[12] - array_B[11]);
		N_E_B = abs(array_B[12] - array_B[13]);
		NW_C_B = abs(array_B[16] - array_B[12]);
		NW_N1_B = abs(array_B[22] - array_B[18]);
		NW_N2_B = abs(array_B[10] - array_B[6]);
		NW_NE_B = abs(array_B[12] - array_B[18]);
		NW_SW_B = abs(array_B[12] - array_B[6]);
		W_C_B = abs(array_B[11] - array_B[12]);
		W_N1_B = abs(array_B[16] - array_B[17]);
		W_N2_B = abs(array_B[6] - array_B[7]);
		W_S_B = abs(array_B[12] - array_B[7]);
		W_N_B = abs(array_B[12] - array_B[17]);

		SW_C_G = abs(array_G[6] - array_G[12]);
		SW_N1_G = abs(array_G[10] - array_G[16]);
		SW_N2_G = abs(array_G[2] - array_G[8]);
		SW_NW_G = abs(array_G[12] - array_G[16]);
		SW_SE_G = abs(array_G[12] - array_G[8]);
		S_C_G = abs(array_G[7] - array_G[12]);
		S_N1_G = abs(array_G[6] - array_G[11]);
		S_N2_G = abs(array_G[8] - array_G[13]);
		S_W_G = abs(array_G[12] - array_G[11]);
		S_E_G = abs(array_G[12] - array_G[13]);
		SE_C_G = abs(array_G[8] - array_G[12]);
		SE_N1_G = abs(array_G[2] - array_G[6]);
		SE_N2_G = abs(array_G[14] - array_G[18]);
		SE_SW_G = abs(array_G[12] - array_G[6]);
		SE_NE_G = abs(array_G[12] - array_G[18]);
		E_C_G = abs(array_G[13] - array_G[12]);
		E_N1_G = abs(array_G[8] - array_G[7]);
		E_N2_G = abs(array_G[18] - array_G[17]);
		E_S_G = abs(array_G[12] - array_G[7]);
		E_N_G = abs(array_G[12] - array_G[17]);
		NE_C_G = abs(array_G[18] - array_G[12]);
		NE_N1_G = abs(array_G[14] - array_G[8]);
		NE_N2_G = abs(array_G[22] - array_G[16]);
		NE_SE_G = abs(array_G[12] - array_G[8]);
		NE_NW_G = abs(array_G[12] - array_G[16]);
		N_C_G = abs(array_G[17] - array_G[12]);
		N_N1_G = abs(array_G[18] - array_G[13]);
		N_N2_G = abs(array_G[16] - array_G[11]);
		N_W_G = abs(array_G[12] - array_G[11]);
		N_E_G = abs(array_G[12] - array_G[13]);
		NW_C_G = abs(array_G[16] - array_G[12]);
		NW_N1_G = abs(array_G[22] - array_G[18]);
		NW_N2_G = abs(array_G[10] - array_G[6]);
		NW_NE_G = abs(array_G[12] - array_G[18]);
		NW_SW_G = abs(array_G[12] - array_G[6]);
		W_C_G = abs(array_G[11] - array_G[12]);
		W_N1_G = abs(array_G[16] - array_G[17]);
		W_N2_G = abs(array_G[6] - array_G[7]);
		W_S_G = abs(array_G[12] - array_G[7]);
		W_N_G = abs(array_G[12] - array_G[17]);

		SW_C_R = abs(array_R[6] - array_R[12]);
		SW_N1_R = abs(array_R[10] - array_R[16]);
		SW_N2_R = abs(array_R[2] - array_R[8]);
		SW_NW_R = abs(array_R[12] - array_R[16]);
		SW_SE_R = abs(array_R[12] - array_R[8]);
		S_C_R = abs(array_R[7] - array_R[12]);
		S_N1_R = abs(array_R[6] - array_R[11]);
		S_N2_R = abs(array_R[8] - array_R[13]);
		S_W_R = abs(array_R[12] - array_R[11]);
		S_E_R = abs(array_R[12] - array_R[13]);
		SE_C_R = abs(array_R[8] - array_R[12]);
		SE_N1_R = abs(array_R[2] - array_R[6]);
		SE_N2_R = abs(array_R[14] - array_R[18]);
		SE_SW_R = abs(array_R[12] - array_R[6]);
		SE_NE_R = abs(array_R[12] - array_R[18]);
		E_C_R = abs(array_R[13] - array_R[12]);
		E_N1_R = abs(array_R[8] - array_R[7]);
		E_N2_R = abs(array_R[18] - array_R[17]);
		E_S_R = abs(array_R[12] - array_R[7]);
		E_N_R = abs(array_R[12] - array_R[17]);
		NE_C_R = abs(array_R[18] - array_R[12]);
		NE_N1_R = abs(array_R[14] - array_R[8]);
		NE_N2_R = abs(array_R[22] - array_R[16]);
		NE_SE_R = abs(array_R[12] - array_R[8]);
		NE_NW_R = abs(array_R[12] - array_R[16]);
		N_C_R = abs(array_R[17] - array_R[12]);
		N_N1_R = abs(array_R[18] - array_R[13]);
		N_N2_R = abs(array_R[16] - array_R[11]);
		N_W_R = abs(array_R[12] - array_R[11]);
		N_E_R = abs(array_R[12] - array_R[13]);
		NW_C_R = abs(array_R[16] - array_R[12]);
		NW_N1_R = abs(array_R[22] - array_R[18]);
		NW_N2_R = abs(array_R[10] - array_R[6]);
		NW_NE_R = abs(array_R[12] - array_R[18]);
		NW_SW_R = abs(array_R[12] - array_R[6]);
		W_C_R = abs(array_R[11] - array_R[12]);
		W_N1_R = abs(array_R[16] - array_R[17]);
		W_N2_R = abs(array_R[6] - array_R[7]);
		W_S_R = abs(array_R[12] - array_R[7]);
		W_N_R = abs(array_R[12] - array_R[17]);

		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) == 0) suroeste_C_R = 0;
		else	suroeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[10] * array_R[16])) == 0) suroeste_N1_R = 0;
		else   suroeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[10] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[2] * array_R[8])) == 0) suroeste_N2_R = 0;
		else   suroeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[2] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) == 0) suroeste_NW_R = 0;
		else	suroeste_NW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) == 0) suroeste_SE_R = 0;
		else	suroeste_SE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[7] * array_R[12])) == 0) sur_C_R = 0;
		else	sur_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[7] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[11])) == 0) sur_N1_R = 0;
		else	sur_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[13])) == 0) sur_N2_R = 0;
		else   sur_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) == 0) sur_W_R = 0;
		else	sur_W_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) == 0) sur_E_R = 0;
		else	sur_E_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[12])) == 0) sureste_C_R = 0;
		else	sureste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[2])) == 0) sureste_N1_R = 0;
		else	sureste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[18])) == 0) sureste_N2_R = 0;
		else	sureste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[6])) == 0) sureste_SW_R = 0;
		else	sureste_SW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) == 0) sureste_NE_R = 0;
		else	sureste_NE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[13] * array_R[12])) == 0) este_C_R = 0;
		else	este_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[13] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[7])) == 0) este_N1_R = 0;
		else	este_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[8] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[17])) == 0) este_N2_R = 0;
		else	este_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) == 0) este_S_R = 0;
		else	este_S_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) == 0) este_N_R = 0;
		else	este_N_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[12])) == 0) noreste_C_R = 0;
		else	noreste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[8])) == 0) noreste_N1_R = 0;
		else	noreste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[14] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[16])) == 0) noreste_N2_R = 0;
		else	noreste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) == 0) noreste_SE_R = 0;
		else	noreste_SE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) == 0) noreste_NW_R = 0;
		else	noreste_NW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[17] * array_R[12])) == 0) norte_C_R = 0;
		else	norte_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[17] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[13])) == 0) norte_N1_R = 0;
		else	norte_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[18] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[11])) == 0) norte_N2_R = 0;
		else	norte_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) == 0) norte_E_R = 0;
		else	norte_E_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) == 0) norte_W_R = 0;
		else	norte_W_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[12])) == 0) noroeste_C_R = 0;
		else	noroeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[18])) == 0) noroeste_N1_R = 0;
		else	noroeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[22] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[10])) == 0) noroeste_N2_R = 0;
		else	noroeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) == 0) noroeste_NE_R = 0;
		else	noroeste_NE_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) == 0) noroeste_SW_R = 0;
		else	noroeste_SW_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[11] * array_R[12])) == 0) oeste_C_R = 0;
		else	oeste_C_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[11] * array_R[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[17])) == 0) oeste_N1_R = 0;
		else	oeste_N1_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[16] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[7])) == 0) oeste_N2_R = 0;
		else	oeste_N2_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[6] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) == 0) oeste_N_R = 0;
		else	oeste_N_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) == 0) oeste_S_R = 0;
		else	oeste_S_R = acos(((cons1 + cons1) + (cons2*cons2) + (array_R[12] * array_R[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_R[12], 2))))));

		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) == 0) suroeste_C_G = 0;
		else	suroeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[10] * array_G[16])) == 0) suroeste_N1_G = 0;
		else   suroeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[10] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[2] * array_G[8])) == 0) suroeste_N2_G = 0;
		else   suroeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[2] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) == 0) suroeste_NW_G = 0;
		else	suroeste_NW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) == 0) suroeste_SE_G = 0;
		else	suroeste_SE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[7] * array_G[12])) == 0) sur_C_G = 0;
		else	sur_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[7] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[11])) == 0) sur_N1_G = 0;
		else	sur_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[13])) == 0) sur_N2_G = 0;
		else   sur_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) == 0) sur_W_G = 0;
		else	sur_W_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) == 0) sur_E_G = 0;
		else	sur_E_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[12])) == 0) sureste_C_G = 0;
		else	sureste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[2])) == 0) sureste_N1_G = 0;
		else	sureste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[18])) == 0) sureste_N2_G = 0;
		else	sureste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[6])) == 0) sureste_SW_G = 0;
		else	sureste_SW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) == 0) sureste_NE_G = 0;
		else	sureste_NE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[13] * array_G[12])) == 0) este_C_G = 0;
		else	este_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[13] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[7])) == 0) este_N1_G = 0;
		else	este_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[8] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[17])) == 0) este_N2_G = 0;
		else	este_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) == 0) este_S_G = 0;
		else	este_S_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) == 0) este_N_G = 0;
		else	este_N_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[12])) == 0) noreste_C_G = 0;
		else	noreste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[8])) == 0) noreste_N1_G = 0;
		else	noreste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[14] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[16])) == 0) noreste_N2_G = 0;
		else	noreste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) == 0) noreste_SE_G = 0;
		else	noreste_SE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) == 0) noreste_NW_G = 0;
		else	noreste_NW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[17] * array_G[12])) == 0) norte_C_G = 0;
		else	norte_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[17] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[13])) == 0) norte_N1_G = 0;
		else	norte_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[18] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[11])) == 0) norte_N2_G = 0;
		else	norte_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) == 0) norte_E_G = 0;
		else	norte_E_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) == 0) norte_W_G = 0;
		else	norte_W_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[12])) == 0) noroeste_C_G = 0;
		else	noroeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[18])) == 0) noroeste_N1_G = 0;
		else	noroeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[22] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[10])) == 0) noroeste_N2_G = 0;
		else	noroeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) == 0) noroeste_NE_G = 0;
		else	noroeste_NE_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) == 0) noroeste_SW_G = 0;
		else	noroeste_SW_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[11] * array_G[12])) == 0) oeste_C_G = 0;
		else	oeste_C_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[11] * array_G[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[17])) == 0) oeste_N1_G = 0;
		else	oeste_N1_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[16] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[7])) == 0) oeste_N2_G = 0;
		else	oeste_N2_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[6] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) == 0) oeste_N_G = 0;
		else	oeste_N_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) == 0) oeste_S_G = 0;
		else	oeste_S_G = acos(((cons1 + cons1) + (cons2*cons2) + (array_G[12] * array_G[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_G[12], 2))))));

		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) == 0) suroeste_C_B = 0;
		else	suroeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[10] * array_B[16])) == 0) suroeste_N1_B = 0;
		else   suroeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[10] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[2] * array_B[8])) == 0) suroeste_N2_B = 0;
		else   suroeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[2] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[2], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) == 0) suroeste_NW_B = 0;
		else	suroeste_NW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) == 0) suroeste_SE_B = 0;
		else	suroeste_SE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[7] * array_B[12])) == 0) sur_C_B = 0;
		else	sur_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[7] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2)))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2)))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[11])) == 0) sur_N1_B = 0;
		else	sur_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[13])) == 0) sur_N2_B = 0;
		else   sur_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) == 0) sur_W_B = 0;
		else	sur_W_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) == 0) sur_E_B = 0;
		else	sur_E_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[12])) == 0) sureste_C_B = 0;
		else	sureste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[2])) == 0) sureste_N1_B = 0;
		else	sureste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[2])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[2], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[18])) == 0) sureste_N2_B = 0;
		else	sureste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[6])) == 0) sureste_SW_B = 0;
		else	sureste_SW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[6])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) == 0) sureste_NE_B = 0;
		else	sureste_NE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[13] * array_B[12])) == 0) este_C_B = 0;
		else	este_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[13] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[7])) == 0) este_N1_B = 0;
		else	este_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[8] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[17])) == 0) este_N2_B = 0;
		else	este_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) == 0) este_S_B = 0;
		else	este_S_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) == 0) este_N_B = 0;
		else	este_N_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[12])) == 0) noreste_C_B = 0;
		else	noreste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[8])) == 0) noreste_N1_B = 0;
		else	noreste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[14] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[14], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[16])) == 0) noreste_N2_B = 0;
		else	noreste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) == 0) noreste_SE_B = 0;
		else	noreste_SE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[8])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[8], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) == 0) noreste_NW_B = 0;
		else	noreste_NW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[16])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[17] * array_B[12])) == 0) norte_C_B = 0;
		else	norte_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[17] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[13])) == 0) norte_N1_B = 0;
		else	norte_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[18] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[11])) == 0) norte_N2_B = 0;
		else	norte_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) == 0) norte_E_B = 0;
		else	norte_E_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[13])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[13], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) == 0) norte_W_B = 0;
		else	norte_W_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[11])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[12])) == 0) noroeste_C_B = 0;
		else	noroeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[18])) == 0) noroeste_N1_B = 0;
		else	noroeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[22] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[22], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[10])) == 0) noroeste_N2_B = 0;
		else	noroeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[10])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[10], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) == 0) noroeste_NE_B = 0;
		else	noroeste_NE_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[18])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[18], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) == 0) noroeste_SW_B = 0;
		else	noroeste_SW_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[11] * array_B[12])) == 0) oeste_C_B = 0;
		else	oeste_C_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[11] * array_B[12])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[11], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[17])) == 0) oeste_N1_B = 0;
		else	oeste_N1_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[16] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[16], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[7])) == 0) oeste_N2_B = 0;
		else	oeste_N2_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[6] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[6], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) == 0) oeste_N_B = 0;
		else	oeste_N_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[17])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[17], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
		if (((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) == 0) oeste_S_B = 0;
		else	oeste_S_B = acos(((cons1 + cons1) + (cons2*cons2) + (array_B[12] * array_B[7])) / ((sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[7], 2))*(sqrt(pow(cons1, 2) + pow(cons1, 2) + pow(array_B[12], 2))))));
		//	SUROESTE	

		/*
		med_1 = 1, var_1 = 0.8;
		med_2 = 0.1;
		*/
		if (suroeste_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((suroeste_C_R)-med_1), 2) / (2 * var_1))));
		if (suroeste_N1_R < med_2) gam_small_1[0] = 1;
		else 	gam_small_1[0] = (exp(-(pow(((suroeste_N1_R)-med_2), 2) / (2 * var_1))));
		if (suroeste_N2_R < med_2) gam_small_1[1] = 1;
		else 	gam_small_1[1] = (exp(-(pow(((suroeste_N2_R)-med_2), 2) / (2 * var_1))));
		if (suroeste_NW_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_R)-med_1), 2) / (2 * var_1))));
		if (suroeste_SE_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_R)-med_1), 2) / (2 * var_1))));
		largo[0] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (sur_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sur_C_R)-med_1), 2) / (2 * var_1))));
		if (sur_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sur_N1_R)-med_2), 2) / (2 * var_1))));
		if (sur_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sur_N2_R)-med_2), 2) / (2 * var_1))));
		if (sur_W_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sur_W_R)-med_1), 2) / (2 * var_1))));
		if (sur_E_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sur_E_R)-med_1), 2) / (2 * var_1))));
		largo[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (sureste_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sureste_C_R)-med_1), 2) / (2 * var_1))));
		if (sureste_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sureste_N1_R)-med_2), 2) / (2 * var_1))));
		if (sureste_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sureste_N2_R)-med_2), 2) / (2 * var_1))));
		if (sureste_NE_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sureste_NE_R)-med_1), 2) / (2 * var_1))));
		if (sureste_SW_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sureste_SW_R)-med_1), 2) / (2 * var_1))));
		largo[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (este_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((este_C_R)-med_1), 2) / (2 * var_1))));
		if (este_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((este_N1_R)-med_2), 2) / (2 * var_1))));
		if (este_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((este_N2_R)-med_2), 2) / (2 * var_1))));
		if (este_N_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((este_N_R)-med_1), 2) / (2 * var_1))));
		if (este_S_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((este_S_R)-med_1), 2) / (2 * var_1))));
		largo[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noreste_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noreste_C_R)-med_1), 2) / (2 * var_1))));
		if (noreste_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noreste_N1_R)-med_2), 2) / (2 * var_1))));
		if (noreste_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noreste_N2_R)-med_2), 2) / (2 * var_1))));
		if (noreste_NW_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noreste_NW_R)-med_1), 2) / (2 * var_1))));
		if (noreste_SE_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noreste_SE_R)-med_1), 2) / (2 * var_1))));
		largo[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (norte_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((norte_C_R)-med_1), 2) / (2 * var_1))));
		if (norte_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((norte_N1_R)-med_2), 2) / (2 * var_1))));
		if (norte_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((norte_N2_R)-med_2), 2) / (2 * var_1))));
		if (norte_W_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((norte_W_R)-med_1), 2) / (2 * var_1))));
		if (norte_E_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((norte_E_R)-med_1), 2) / (2 * var_1))));
		largo[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noroeste_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noroeste_C_R)-med_1), 2) / (2 * var_1))));
		if (noroeste_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_R)-med_2), 2) / (2 * var_1))));
		if (noroeste_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_R)-med_2), 2) / (2 * var_1))));
		if (noroeste_NE_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_R)-med_1), 2) / (2 * var_1))));
		if (noroeste_SW_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_R)-med_1), 2) / (2 * var_1))));
		largo[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (oeste_C_R > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((oeste_C_R)-med_1), 2) / (2 * var_1))));
		if (oeste_N1_R < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((oeste_N1_R)-med_2), 2) / (2 * var_1))));
		if (oeste_N2_R < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((oeste_N2_R)-med_2), 2) / (2 * var_1))));
		if (oeste_N_R > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((oeste_N_R)-med_1), 2) / (2 * var_1))));
		if (oeste_S_R > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((oeste_S_R)-med_1), 2) / (2 * var_1))));
		largo[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (suroeste_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((suroeste_C_G)-med_1), 2) / (2 * var_1))));
		if (suroeste_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((suroeste_N1_G)-med_2), 2) / (2 * var_1))));
		if (suroeste_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((suroeste_N2_G)-med_2), 2) / (2 * var_1))));
		if (suroeste_NW_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_G)-med_1), 2) / (2 * var_1))));
		if (suroeste_SE_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_G)-med_1), 2) / (2 * var_1))));
		largo_1[0] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (sur_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sur_C_G)-med_1), 2) / (2 * var_1))));
		if (sur_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sur_N1_G)-med_2), 2) / (2 * var_1))));
		if (sur_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sur_N2_G)-med_2), 2) / (2 * var_1))));
		if (sur_W_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sur_W_G)-med_1), 2) / (2 * var_1))));
		if (sur_E_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sur_E_G)-med_1), 2) / (2 * var_1))));
		largo_1[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (sureste_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sureste_C_G)-med_1), 2) / (2 * var_1))));
		if (sureste_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sureste_N1_G)-med_2), 2) / (2 * var_1))));
		if (sureste_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sureste_N2_G)-med_2), 2) / (2 * var_1))));
		if (sureste_NE_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sureste_NE_G)-med_1), 2) / (2 * var_1))));
		if (sureste_SW_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sureste_SW_G)-med_1), 2) / (2 * var_1))));
		largo_1[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (este_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((este_C_G)-med_1), 2) / (2 * var_1))));
		if (este_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((este_N1_G)-med_2), 2) / (2 * var_1))));
		if (este_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((este_N2_G)-med_2), 2) / (2 * var_1))));
		if (este_N_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((este_N_G)-med_1), 2) / (2 * var_1))));
		if (este_S_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((este_S_G)-med_1), 2) / (2 * var_1))));
		largo_1[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noreste_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noreste_C_G)-med_1), 2) / (2 * var_1))));
		if (noreste_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noreste_N1_G)-med_2), 2) / (2 * var_1))));
		if (noreste_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noreste_N2_G)-med_2), 2) / (2 * var_1))));
		if (noreste_NW_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noreste_NW_G)-med_1), 2) / (2 * var_1))));
		if (noreste_SE_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noreste_SE_G)-med_1), 2) / (2 * var_1))));
		largo_1[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (norte_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((norte_C_G)-med_1), 2) / (2 * var_1))));
		if (norte_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((norte_N1_G)-med_2), 2) / (2 * var_1))));
		if (norte_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((norte_N2_G)-med_2), 2) / (2 * var_1))));
		if (norte_W_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((norte_W_G)-med_1), 2) / (2 * var_1))));
		if (norte_E_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((norte_E_G)-med_1), 2) / (2 * var_1))));
		largo_1[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noroeste_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noroeste_C_G)-med_1), 2) / (2 * var_1))));
		if (noroeste_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_G)-med_2), 2) / (2 * var_1))));
		if (noroeste_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_G)-med_2), 2) / (2 * var_1))));
		if (noroeste_NE_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_G)-med_1), 2) / (2 * var_1))));
		if (noroeste_SW_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_G)-med_1), 2) / (2 * var_1))));
		largo_1[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (oeste_C_G > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((oeste_C_G)-med_1), 2) / (2 * var_1))));
		if (oeste_N1_G < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((oeste_N1_G)-med_2), 2) / (2 * var_1))));
		if (oeste_N2_G < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((oeste_N2_G)-med_2), 2) / (2 * var_1))));
		if (oeste_N_G > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((oeste_N_G)-med_1), 2) / (2 * var_1))));
		if (oeste_S_G > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((oeste_S_G)-med_1), 2) / (2 * var_1))));
		largo_1[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (suroeste_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((suroeste_C_B)-med_1), 2) / (2 * var_1))));
		if (suroeste_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((suroeste_N1_B)-med_2), 2) / (2 * var_1))));
		if (suroeste_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((suroeste_N2_B)-med_2), 2) / (2 * var_1))));
		if (suroeste_NW_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((suroeste_NW_B)-med_1), 2) / (2 * var_1))));
		if (suroeste_SE_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((suroeste_SE_B)-med_1), 2) / (2 * var_1))));
		largo_2[0] = (gam_big_2[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_2[2]);
		if (sur_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sur_C_B)-med_1), 2) / (2 * var_1))));
		if (sur_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sur_N1_B)-med_2), 2) / (2 * var_1))));
		if (sur_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sur_N2_B)-med_2), 2) / (2 * var_1))));
		if (sur_W_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sur_W_B)-med_1), 2) / (2 * var_1))));
		if (sur_E_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sur_E_B)-med_1), 2) / (2 * var_1))));
		largo_2[1] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (sureste_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((sureste_C_B)-med_1), 2) / (2 * var_1))));
		if (sureste_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((sureste_N1_B)-med_2), 2) / (2 * var_1))));
		if (sureste_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((sureste_N2_B)-med_2), 2) / (2 * var_1))));
		if (sureste_NE_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((sureste_NE_B)-med_1), 2) / (2 * var_1))));
		if (sureste_SW_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((sureste_SW_B)-med_1), 2) / (2 * var_1))));
		largo_2[2] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (este_C_B > med_1) gam_big_1[0] = 1;

		else	gam_big_1[0] = (exp(-(pow(((este_C_B)-med_1), 2) / (2 * var_1))));
		if (este_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((este_N1_B)-med_2), 2) / (2 * var_1))));
		if (este_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((este_N2_B)-med_2), 2) / (2 * var_1))));
		if (este_N_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((este_N_B)-med_1), 2) / (2 * var_1))));
		if (este_S_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((este_S_B)-med_1), 2) / (2 * var_1))));
		largo_2[3] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noreste_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noreste_C_B)-med_1), 2) / (2 * var_1))));
		if (noreste_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noreste_N1_B)-med_2), 2) / (2 * var_1))));
		if (noreste_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noreste_N2_B)-med_2), 2) / (2 * var_1))));
		if (noreste_NW_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noreste_NW_B)-med_1), 2) / (2 * var_1))));
		if (noreste_SE_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noreste_SE_B)-med_1), 2) / (2 * var_1))));
		largo_2[4] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (norte_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((norte_C_B)-med_1), 2) / (2 * var_1))));
		if (norte_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((norte_N1_B)-med_2), 2) / (2 * var_1))));
		if (norte_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((norte_N2_B)-med_2), 2) / (2 * var_1))));
		if (norte_W_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((norte_W_B)-med_1), 2) / (2 * var_1))));
		if (norte_E_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((norte_E_B)-med_1), 2) / (2 * var_1))));
		largo_2[5] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (noroeste_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((noroeste_C_B)-med_1), 2) / (2 * var_1))));
		if (noroeste_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((noroeste_N1_B)-med_2), 2) / (2 * var_1))));
		if (noroeste_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((noroeste_N2_B)-med_2), 2) / (2 * var_1))));
		if (noroeste_NE_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((noroeste_NE_B)-med_1), 2) / (2 * var_1))));
		if (noroeste_SW_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((noroeste_SW_B)-med_1), 2) / (2 * var_1))));
		largo_2[6] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);
		if (oeste_C_B > med_1) gam_big_1[0] = 1;
		else	gam_big_1[0] = (exp(-(pow(((oeste_C_B)-med_1), 2) / (2 * var_1))));
		if (oeste_N1_B < med_2) gam_small_1[0] = 1;
		else	gam_small_1[0] = (exp(-(pow(((oeste_N1_B)-med_2), 2) / (2 * var_1))));
		if (oeste_N2_B < med_2) gam_small_1[1] = 1;
		else	gam_small_1[1] = (exp(-(pow(((oeste_N2_B)-med_2), 2) / (2 * var_1))));
		if (oeste_N_B > med_1) gam_big_1[1] = 1;
		else	gam_big_1[1] = (exp(-(pow(((oeste_N_B)-med_1), 2) / (2 * var_1))));
		if (oeste_S_B > med_1) gam_big_1[2] = 1;
		else	gam_big_1[2] = (exp(-(pow(((oeste_S_B)-med_1), 2) / (2 * var_1))));
		largo_2[7] = (gam_big_1[0] * gam_small_1[0] * gam_small_1[1] * gam_big_1[1] * gam_big_1[2]);

		/*
		med1 = 60;
		med2 = 10;
		var1 = 1000;
		*/

		if (SW_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SW_C_R)-med1), 2) / (2 * var1))));
		if (SW_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SW_N1_R)-med2), 2) / (2 * var1))));
		if (SW_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SW_N2_R)-med2), 2) / (2 * var1))));
		if (SW_NW_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SW_NW_R)-med1), 2) / (2 * var1))));
		if (SW_SE_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SW_SE_R)-med1), 2) / (2 * var1))));
		LARGO[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (S_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((S_C_R)-med1), 2) / (2 * var1))));
		if (S_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((S_N1_R)-med2), 2) / (2 * var1))));
		if (S_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((S_N2_R)-med2), 2) / (2 * var1))));
		if (S_W_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((S_W_R)-med1), 2) / (2 * var1))));
		if (S_E_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((S_E_R)-med1), 2) / (2 * var1))));
		LARGO[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (SE_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SE_C_R)-med1), 2) / (2 * var1))));
		if (SE_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SE_N1_R)-med2), 2) / (2 * var1))));
		if (SE_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SE_N2_R)-med2), 2) / (2 * var1))));
		if (SE_NE_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SE_NE_R)-med1), 2) / (2 * var1))));
		if (SE_SW_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SE_SW_R)-med1), 2) / (2 * var1))));
		LARGO[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (E_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((E_C_R)-med1), 2) / (2 * var1))));
		if (E_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((E_N1_R)-med2), 2) / (2 * var1))));
		if (E_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((E_N2_R)-med2), 2) / (2 * var1))));
		if (E_N_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((E_N_R)-med1), 2) / (2 * var1))));
		if (E_S_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((E_S_R)-med1), 2) / (2 * var1))));
		LARGO[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NE_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NE_C_R)-med1), 2) / (2 * var1))));
		if (NE_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NE_N1_R)-med2), 2) / (2 * var1))));
		if (NE_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NE_N2_R)-med2), 2) / (2 * var1))));
		if (NE_NW_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NE_NW_R)-med1), 2) / (2 * var1))));
		if (NE_SE_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NE_SE_R)-med1), 2) / (2 * var1))));
		LARGO[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (N_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((N_C_R)-med1), 2) / (2 * var1))));
		if (N_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((N_N1_R)-med2), 2) / (2 * var1))));
		if (N_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((N_N2_R)-med2), 2) / (2 * var1))));
		if (N_W_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((N_W_R)-med1), 2) / (2 * var1))));
		if (N_E_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((N_E_R)-med1), 2) / (2 * var1))));
		LARGO[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NW_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NW_C_R)-med1), 2) / (2 * var1))));
		if (NW_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NW_N1_R)-med2), 2) / (2 * var1))));
		if (NW_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NW_N2_R)-med2), 2) / (2 * var1))));
		if (NW_NE_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NW_NE_R)-med1), 2) / (2 * var1))));
		if (NW_SW_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NW_SW_R)-med1), 2) / (2 * var1))));
		LARGO[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (W_C_R > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((W_C_R)-med1), 2) / (2 * var1))));
		if (W_N1_R < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((W_N1_R)-med2), 2) / (2 * var1))));
		if (W_N2_R < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((W_N2_R)-med2), 2) / (2 * var1))));
		if (W_N_R > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((W_N_R)-med1), 2) / (2 * var1))));
		if (W_S_R > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((W_S_R)-med1), 2) / (2 * var1))));
		LARGO[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (SW_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SW_C_G)-med1), 2) / (2 * var1))));
		if (SW_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SW_N1_G)-med2), 2) / (2 * var1))));
		if (SW_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SW_N2_G)-med2), 2) / (2 * var1))));
		if (SW_NW_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SW_NW_G)-med1), 2) / (2 * var1))));
		if (SW_SE_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SW_SE_G)-med1), 2) / (2 * var1))));
		LARGO_1[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (S_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((S_C_G)-med1), 2) / (2 * var1))));
		if (S_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((S_N1_G)-med2), 2) / (2 * var1))));
		if (S_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((S_N2_G)-med2), 2) / (2 * var1))));
		if (S_W_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((S_W_G)-med1), 2) / (2 * var1))));
		if (S_E_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((S_E_G)-med1), 2) / (2 * var1))));
		LARGO_1[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (SE_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SE_C_G)-med1), 2) / (2 * var1))));
		if (SE_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SE_N1_G)-med2), 2) / (2 * var1))));
		if (SE_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SE_N2_G)-med2), 2) / (2 * var1))));
		if (SE_NE_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SE_NE_G)-med1), 2) / (2 * var1))));
		if (SE_SW_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SE_SW_G)-med1), 2) / (2 * var1))));
		LARGO_1[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (E_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((E_C_G)-med1), 2) / (2 * var1))));
		if (E_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((E_N1_G)-med2), 2) / (2 * var1))));
		if (E_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((E_N2_G)-med2), 2) / (2 * var1))));
		if (E_N_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((E_N_G)-med1), 2) / (2 * var1))));
		if (E_S_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((E_S_G)-med1), 2) / (2 * var1))));
		LARGO_1[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NE_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NE_C_G)-med1), 2) / (2 * var1))));
		if (NE_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NE_N1_G)-med2), 2) / (2 * var1))));
		if (NE_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NE_N2_G)-med2), 2) / (2 * var1))));
		if (NE_NW_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NE_NW_G)-med1), 2) / (2 * var1))));
		if (NE_SE_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NE_SE_G)-med1), 2) / (2 * var1))));
		LARGO_1[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (N_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((N_C_G)-med1), 2) / (2 * var1))));
		if (N_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((N_N1_G)-med2), 2) / (2 * var1))));
		if (N_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((N_N2_G)-med2), 2) / (2 * var1))));
		if (N_W_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((N_W_G)-med1), 2) / (2 * var1))));
		if (N_E_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((N_E_G)-med1), 2) / (2 * var1))));
		LARGO_1[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NW_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NW_C_G)-med1), 2) / (2 * var1))));
		if (NW_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NW_N1_G)-med2), 2) / (2 * var1))));
		if (NW_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NW_N2_G)-med2), 2) / (2 * var1))));
		if (NW_NE_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NW_NE_G)-med1), 2) / (2 * var1))));
		if (NW_SW_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NW_SW_G)-med1), 2) / (2 * var1))));
		LARGO_1[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (W_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((W_C_G)-med1), 2) / (2 * var1))));
		if (W_N1_G < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((W_N1_G)-med2), 2) / (2 * var1))));
		if (W_N2_G < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((W_N2_G)-med2), 2) / (2 * var1))));
		if (W_N_G > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((W_N_G)-med1), 2) / (2 * var1))));
		if (W_S_G > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((W_S_G)-med1), 2) / (2 * var1))));
		LARGO_1[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (SW_C_G > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SW_C_B)-med1), 2) / (2 * var1))));
		if (SW_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SW_N1_B)-med2), 2) / (2 * var1))));
		if (SW_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SW_N2_B)-med2), 2) / (2 * var1))));
		if (SW_NW_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SW_NW_B)-med1), 2) / (2 * var1))));
		if (SW_SE_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SW_SE_B)-med1), 2) / (2 * var1))));
		LARGO_2[0] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (S_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((S_C_B)-med1), 2) / (2 * var1))));
		if (S_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((S_N1_B)-med2), 2) / (2 * var1))));
		if (S_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((S_N2_B)-med2), 2) / (2 * var1))));
		if (S_W_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((S_W_B)-med1), 2) / (2 * var1))));
		if (S_E_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((S_E_B)-med1), 2) / (2 * var1))));
		LARGO_2[1] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (SE_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((SE_C_B)-med1), 2) / (2 * var1))));
		if (SE_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((SE_N1_B)-med2), 2) / (2 * var1))));
		if (SE_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((SE_N2_B)-med2), 2) / (2 * var1))));
		if (SE_NE_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((SE_NE_B)-med1), 2) / (2 * var1))));
		if (SE_SW_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((SE_SW_B)-med1), 2) / (2 * var1))));
		LARGO_2[2] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (E_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((E_C_B)-med1), 2) / (2 * var1))));
		if (E_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((E_N1_B)-med2), 2) / (2 * var1))));
		if (E_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((E_N2_B)-med2), 2) / (2 * var1))));
		if (E_N_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((E_N_B)-med1), 2) / (2 * var1))));
		if (E_S_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((E_S_B)-med1), 2) / (2 * var1))));
		LARGO_2[3] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NE_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NE_C_B)-med1), 2) / (2 * var1))));
		if (NE_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NE_N1_B)-med2), 2) / (2 * var1))));
		if (NE_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NE_N2_B)-med2), 2) / (2 * var1))));
		if (NE_NW_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NE_NW_B)-med1), 2) / (2 * var1))));
		if (NE_SE_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NE_SE_B)-med1), 2) / (2 * var1))));
		LARGO_2[4] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (N_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((N_C_B)-med1), 2) / (2 * var1))));
		if (N_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((N_N1_B)-med2), 2) / (2 * var1))));
		if (N_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((N_N2_B)-med2), 2) / (2 * var1))));
		if (N_W_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((N_W_B)-med1), 2) / (2 * var1))));
		if (N_E_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((N_E_B)-med1), 2) / (2 * var1))));
		LARGO_2[5] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (NW_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((NW_C_B)-med1), 2) / (2 * var1))));
		if (NW_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((NW_N1_B)-med2), 2) / (2 * var1))));
		if (NW_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((NW_N2_B)-med2), 2) / (2 * var1))));
		if (NW_NE_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((NW_NE_B)-med1), 2) / (2 * var1))));
		if (NW_SW_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((NW_SW_B)-med1), 2) / (2 * var1))));
		LARGO_2[6] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);
		if (W_C_B > med1) gam_big_2[0] = 1;
		else	gam_big_2[0] = (exp(-(pow(((W_C_B)-med1), 2) / (2 * var1))));
		if (W_N1_B < med2) gam_small_2[0] = 1;
		else	gam_small_2[0] = (exp(-(pow(((W_N1_B)-med2), 2) / (2 * var1))));
		if (W_N2_B < med2) gam_small_2[1] = 1;
		else	gam_small_2[1] = (exp(-(pow(((W_N2_B)-med2), 2) / (2 * var1))));
		if (W_N_B > med1) gam_big_2[1] = 1;
		else	gam_big_2[1] = (exp(-(pow(((W_N_B)-med1), 2) / (2 * var1))));
		if (W_S_B > med1) gam_big_2[2] = 1;
		else	gam_big_2[2] = (exp(-(pow(((W_S_B)-med1), 2) / (2 * var1))));
		LARGO_2[7] = (gam_big_2[0] * gam_small_2[0] * gam_small_2[1] * gam_big_2[1] * gam_big_2[2]);

		float	mu_R_R[8], mu_G_G[8], mu_B_B[8];

		mu_R_R[0] = min(largo[0], LARGO[0]);
		mu_R_R[1] = min(largo[1], LARGO[1]);
		mu_R_R[2] = min(largo[2], LARGO[2]);
		mu_R_R[3] = min(largo[3], LARGO[3]);
		mu_R_R[4] = min(largo[4], LARGO[4]);
		mu_R_R[5] = min(largo[5], LARGO[5]);
		mu_R_R[6] = min(largo[6], LARGO[6]);
		mu_R_R[7] = min(largo[7], LARGO[7]);

		mu_G_G[0] = min(largo_1[0], LARGO_1[0]);
		mu_G_G[1] = min(largo_1[1], LARGO_1[1]);
		mu_G_G[2] = min(largo_1[2], LARGO_1[2]);
		mu_G_G[3] = min(largo_1[3], LARGO_1[3]);
		mu_G_G[4] = min(largo_1[4], LARGO_1[4]);
		mu_G_G[5] = min(largo_1[5], LARGO_1[5]);
		mu_G_G[6] = min(largo_1[6], LARGO_1[6]);
		mu_G_G[7] = min(largo_1[7], LARGO_1[7]);

		mu_B_B[0] = min(largo_2[0], LARGO_2[0]);
		mu_B_B[1] = min(largo_2[1], LARGO_2[1]);
		mu_B_B[2] = min(largo_2[2], LARGO_2[2]);
		mu_B_B[3] = min(largo_2[3], LARGO_2[3]);
		mu_B_B[4] = min(largo_2[4], LARGO_2[4]);
		mu_B_B[5] = min(largo_2[5], LARGO_2[5]);
		mu_B_B[6] = min(largo_2[6], LARGO_2[6]);
		mu_B_B[7] = min(largo_2[7], LARGO_2[7]);

		noise_R_R = max(max(max(max(max(max(max(mu_R_R[0], mu_R_R[1]), mu_R_R[2]), mu_R_R[3]), mu_R_R[4]), mu_R_R[5]), mu_R_R[6]), mu_R_R[7]);
		noise_G_G = max(max(max(max(max(max(max(mu_G_G[0], mu_G_G[1]), mu_G_G[2]), mu_G_G[3]), mu_G_G[4]), mu_G_G[5]), mu_G_G[6]), mu_G_G[7]);
		noise_B_B = max(max(max(max(max(max(max(mu_B_B[0], mu_B_B[1]), mu_B_B[2]), mu_B_B[3]), mu_B_B[4]), mu_B_B[5]), mu_B_B[6]), mu_B_B[7]);

		//printf( "%f",noise_B_B);

		if ((noise_B_B >= THS))
		{

			float weights[9], sum_weights = 0, hold2, suma = 0;
			for (j = 0; j <= 7; j++)
			{
				sum_weights += (1 - mu_B_B[j]);
			}
			sum_weights = (sum_weights + 3 * sqrt(1 - noise_B_B)) / 2;
			weights[0] = (1 - mu_B_B[0]);
			weights[1] = (1 - mu_B_B[1]);
			weights[2] = (1 - mu_B_B[2]);
			weights[3] = (1 - mu_B_B[7]);
			weights[4] = 3 * sqrt(1 - noise_B_B);
			weights[5] = (1 - mu_B_B[3]);
			weights[6] = (1 - mu_B_B[6]);
			weights[7] = (1 - mu_B_B[5]);
			weights[8] = (1 - mu_B_B[4]);

			for (j = 0; j <= 8; j++)
			{
				for (x = 0; x <= 7; x++)
				{
					if (vectB[x] > vectB[x + 1])
					{
						hold = vectB[x];
						hold2 = weights[x];
						vectB[x] = vectB[x + 1];
						weights[x] = weights[x + 1];
						vectB[x + 1] = hold;
						weights[x + 1] = hold2;
					}
				}
			}
			for (j = 8; j >= 0; j--)
			{
				suma += weights[j];
				if (suma >= sum_weights)
				{
					if (j < 2)
					{
						sum_weights = sum_weights - (weights[0] + weights[1]);
						sum_weights = sum_weights / 2;
						suma = 0;
						for (F = 8; F >= 2; F--)
						{
							suma += weights[F];
							if (suma > sum_weights)
							{
								d_Pout[(Row * m + Col) * channels + 2] = vectB[F];
								F = -1;
							}
						}
						j = -1;
					}
					else
					{
						d_Pout[(Row * m + Col) * channels + 2] = vectB[j];
						//d_Pout[(Row * m + Col) * channels + 0] = d_Pout[(Row * m + Col) * channels + 0];
						j = -1;
					}
					suma = -1;
				}
			}
			//		fwrite (&CCC, 1, 1, header_file);
		}
		else
		{
			d_Pout[(Row * m + Col) * channels + 2] = vectB[4];
			//d_Pout[(Row * m + Col) * channels + 0] = 0;

			//		fwrite (&CCC, 1, 1, header_file);
		}

		if (noise_G_G >= THS)
		{

			float weights[9], sum_weights = 0, hold2, suma = 0;
			for (j = 0; j <= 7; j++)
			{
				sum_weights += (1 - mu_G_G[j]);
			}
			sum_weights = (sum_weights + 3 * sqrt(1 - noise_G_G)) / 2;
			weights[0] = (1 - mu_G_G[0]);
			weights[1] = (1 - mu_G_G[1]);
			weights[2] = (1 - mu_G_G[2]);
			weights[3] = (1 - mu_G_G[7]);
			weights[4] = 3 * sqrt(1 - noise_G_G);
			weights[5] = (1 - mu_G_G[3]);
			weights[6] = (1 - mu_G_G[6]);
			weights[7] = (1 - mu_G_G[5]);
			weights[8] = (1 - mu_G_G[4]);
			for (j = 0; j <= 8; j++)
			{
				for (x = 0; x <= 7; x++)
				{
					if (vectG[x] > vectG[x + 1])
					{
						hold = vectG[x];
						hold2 = weights[x];
						vectG[x] = vectG[x + 1];
						weights[x] = weights[x + 1];
						vectG[x + 1] = hold;
						weights[x + 1] = hold2;
					}
				}
			}
			for (j = 8; j >= 0; j--)
			{
				suma += weights[j];
				if (suma >= sum_weights)
				{
					if (j < 2)
					{
						sum_weights = sum_weights - (weights[0] + weights[1]);
						sum_weights = sum_weights / 2;
						suma = 0;
						for (F = 8; F >= 2; F--)
						{
							suma += weights[F];
							if (suma >= sum_weights)
							{
								d_Pout[(Row * m + Col) * channels + 1] = vectG[F];
								F = -1;
							}
						}
						j = -1;
					}
					else
					{
						d_Pout[(Row * m + Col) * channels + 1] = vectG[j];
						j = -1;
					}
					suma = -1;
				}
			}
			//		fwrite (&BBB, 1, 1, header_file);
		}
		else
		{
			d_Pout[(Row * m + Col) * channels + 1] = vectG[4];
			//		fwrite (&BBB, 1, 1, header_file);
		}

		if (noise_R_R >= THS)
		{

			float weights[9], sum_weights = 0, hold2, suma = 0;
			for (j = 0; j <= 7; j++)
			{
				sum_weights += (1 - mu_R_R[j]);
			}
			sum_weights = (sum_weights + 3 * sqrt(1 - noise_R_R)) / 2;
			weights[0] = (1 - mu_R_R[0]);
			weights[1] = (1 - mu_R_R[1]);
			weights[2] = (1 - mu_R_R[2]);
			weights[3] = (1 - mu_R_R[7]);
			weights[4] = 3 * sqrt(1 - noise_R_R);
			weights[5] = (1 - mu_R_R[3]);
			weights[6] = (1 - mu_R_R[6]);
			weights[7] = (1 - mu_R_R[5]);
			weights[8] = (1 - mu_R_R[4]);
			for (j = 0; j <= 8; j++)
			{
				for (x = 0; x <= 7; x++)
				{
					if (vectR[x] > vectR[x + 1])
					{
						hold = vectR[x];
						hold2 = weights[x];
						vectR[x] = vectR[x + 1];
						weights[x] = weights[x + 1];
						vectR[x + 1] = hold;
						weights[x + 1] = hold2;
					}
				}
			}
			for (j = 8; j >= 0; j--)
			{
				suma += weights[j];
				if (suma >= sum_weights)
				{
					if (j < 2)
					{
						sum_weights = sum_weights - (weights[0] + weights[1]);
						sum_weights = sum_weights / 2;
						suma = 0;
						for (F = 8; F >= 2; F--)
						{
							suma += weights[F];
							if (suma > sum_weights)
							{
								d_Pout[(Row * m + Col) * channels + 0] = vectR[F];
								F = -1;
							}
						}
						j = -1;
					}
					else
					{
						d_Pout[(Row * m + Col) * channels + 0] = vectR[j];
						j = -1;
					}
					suma = -1;
				}
			}
			//      fwrite (&AAA, 1, 1, header_file);
		}
		else
		{
			d_Pout[(Row * m + Col) * channels + 0] = vectR[4];
			//d_Pout[(Row * m + Col) * channels + 0] = 255;
			//		fwrite (&AAA, 1, 1, header_file);
		}


		//d_Pout[(Row * m + Col) * channels + 0] = 255;
	}

}

