#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/objdetect/objdetect.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include <iostream>
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <time.h>
#include <iostream> // library that contain basic input/output functions
#include <fstream> 
#include <iomanip>
#include <string.h>
#include <algorithm>    // std::sort, max min


#include "FuncionesAux.h"
#include "FiltrosColor.cuh"


#define	M	512// horizontal, x   //estan bien estos columnnas
#define N	512// verticual, y   // filas

/*
#define	M	768// horizontal, x   //estan bien estos columnnas
#define N	512// verticual, y   // filas
*/

#define nChannels 3

#define Mask_width  3
#define Mask_radius Mask_width / 2
#define TILE_WIDTH  16
#define SIZE        (TILE_WIDTH + Mask_width - 1)





hipError_t addWithCuda(unsigned char *dev_out, unsigned char *dev_in, int size); //salida,entrada y tamaño

using namespace cv;
using namespace std;


struct pixel
{
	float alpha;
	unsigned char R;
	unsigned char G;
	unsigned char B;
	int index;

};

__device__ int compareVMF(const struct pixel *a, const struct pixel *b)
{
	if (a->alpha < b->alpha) return -1;
	if (a->alpha == b->alpha) return 0;
	if (a->alpha > b->alpha) return 1;
}

double getPSNR(const Mat& I1, const Mat& I2)
{
	Mat s1;
	absdiff(I1, I2, s1);       // |I1 - I2|
	s1.convertTo(s1, CV_32F);  // cannot make a square on 8 bits
	s1 = s1.mul(s1);           // |I1 - I2|^2

	Scalar s = sum(s1);        // sum elements per channel

	double sse = s.val[0] + s.val[1] + s.val[2]; // sum channels

	if (sse <= 1e-10) // for small values return zero
		return 0;
	else
	{
		double mse = sse / (double)(I1.channels() * I1.total());
		double psnr = 10.0 * log10((255 * 255) / mse);
		return psnr;
	}
}
double getMAE(const Mat& I1, const Mat& I2)
{
	Mat s1;
	absdiff(I1, I2, s1);       // |I1 - I2|

	Scalar s = sum(s1);        // sum elements per channel

	double sse = s.val[0] + s.val[1] + s.val[2]; // sum channels

	if (sse <= 1e-10) // for small values return zero
		return 0;
	else
	{
		double mae = sse / (double)(I1.channels() * I1.total());

		return mae;
	}
}
float getMCRE(const Mat& I1, const Mat& I2)
{
	long int JJ;
	double XYZ[9], x, y, z, x1, y1, z1, X, Y, Z, dist = 0;
	int k;
	XYZ[0] = 0.489989; XYZ[1] = 0.310008; XYZ[2] = 0.2; XYZ[3] = 0.176962;
	XYZ[4] = 0.81240; XYZ[5] = 0.01; XYZ[6] = 0.0; XYZ[7] = 0.01; XYZ[8] = 0.99;

	unsigned char *datosI1, *datosI2;
	datosI1 = (unsigned char*)(I1.data); datosI2 = (unsigned char*)(I2.data);

	for (int Row = 1; Row < I1.rows - 1; Row++) {
		for (int Col = 1; Col < I1.cols - 1; Col++) {
			if (datosI1[(Row * I1.rows + Col) * 3 + 0] == 0 & datosI1[(Row * I1.rows + Col) * 3 + 1] == 0 & datosI1[(Row * I1.rows + Col) * 3 + 2] == 0)
			{
				x = 0, y = 0, z = 0;
			}
			else {
				X = XYZ[0] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[1] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[2] * datosI1[(Row * I1.rows + Col) * 3 + 0];
				Y = XYZ[3] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[4] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[5] * datosI1[(Row * I1.rows + Col) * 3 + 0];
				Z = XYZ[6] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[7] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[8] * datosI1[(Row * I1.rows + Col) * 3 + 0];
				x = X / (X + Y + Z);
				y = Y / (X + Y + Z);
				z = Z / (X + Y + Z);
			}
			//apartir de aqui son R1,B1 y G1
			if (datosI2[(Row * I1.rows + Col) * 3 + 0] + datosI2[(Row * I1.rows + Col) * 3 + 1] + datosI2[(Row * I1.rows + Col) * 3 + 2] == 0)
			{
				x1 = 0, y1 = 0, z1 = 0;
			}
			else
			{
				X = XYZ[0] * datosI2[(Row * I1.rows + Col) * 3 + 2] + XYZ[1] * datosI2[(Row * I1.rows + Col) * 3 + 1] + XYZ[2] * datosI2[(Row * I1.rows + Col) * 3 + 0];
				Y = XYZ[3] * datosI2[(Row * I1.rows + Col) * 3 + 2] + XYZ[4] * datosI2[(Row * I1.rows + Col) * 3 + 1] + XYZ[5] * datosI2[(Row * I1.rows + Col) * 3 + 0];
				Z = XYZ[6] * datosI2[(Row * I1.rows + Col) * 3 + 2] + XYZ[7] * datosI2[(Row * I1.rows + Col) * 3 + 1] + XYZ[8] * datosI2[(Row * I1.rows + Col) * 3 + 0];
				x1 = X / (X + Y + Z);
				y1 = Y / (X + Y + Z);
				z1 = Z / (X + Y + Z);
			}
			dist = sqrt(pow(x - x1, 2) + pow(y - y1, 2) + pow(z - z1, 2)) + dist;
		}
	}
	//printf("%f\n", dist / (I1.rows*I1.cols));
	return dist / (I1.rows*I1.cols);
}
double getMCRE_Mio(const Mat& I1, const Mat& I2)
{
	float pixelR1, pixelG1, pixelB1, pixelR2, pixelG2, pixelB2;
	float valMag1, valMag2;
	double distancia = 0.0, aux = 0.0;
	float pixelUnit1[3], pixelUnit2[3];
	unsigned char *datosI1, *datosI2;
	datosI1 = (unsigned char*)(I1.data);
	datosI2 = (unsigned char*)(I2.data);

	for (int Col = 2; Col <= I1.rows - 2; Col++) {
		for (int Row = 2; Row <= I1.rows - 2; Row++) {
			pixelR1 = datosI1[((Row)* I1.rows + (Col)) * 3 + 0];
			pixelG1 = datosI1[((Row)* I1.rows + (Col)) * 3 + 1];
			pixelB1 = datosI1[((Row)* I1.rows + (Col)) * 3 + 2];

			valMag1 = sqrt((pixelR1*pixelR1) + (pixelG1*pixelG1) + (pixelB1*pixelB1));



			pixelR2 = datosI2[((Row)* I1.rows + (Col)) * 3 + 0];
			pixelG2 = datosI2[((Row)* I1.rows + (Col)) * 3 + 1];
			pixelB2 = datosI2[((Row)* I1.rows + (Col)) * 3 + 2];

			valMag2 = sqrt((pixelR2*pixelR2) + (pixelG2*pixelG2) + (pixelB2*pixelB2));


			if (valMag1 == 0 || valMag2 == 0) {
				distancia += 0;
				//printf("divicion por cero\n");
			}
			else {
				pixelUnit1[0] = (pixelR1 / valMag1);
				pixelUnit1[1] = (pixelG1 / valMag1);
				pixelUnit1[2] = (pixelB1 / valMag1);

				pixelUnit2[0] = (pixelR2 / valMag2);
				pixelUnit2[1] = (pixelG2 / valMag2);
				pixelUnit2[2] = (pixelB2 / valMag2);

				distancia += sqrt(pow((pixelUnit1[0] * 255) - (pixelUnit2[0] * 255), 2)
					+ pow((pixelUnit1[1] * 255) - (pixelUnit2[1] * 255), 2)
					+ pow((pixelUnit1[2] * 255) - (pixelUnit2[2] * 255), 2));
			}

		}
	}
	aux = distancia / (I1.rows*I1.cols);
	return aux;
}
float getNCD(const Mat& I1, const Mat& I2)
{
	long int JJ;
	int k;
	float xyz[20], XYZ[20], WPQ[20];
	float un, vn, L, L1, u, v, ul, vl, v_1, u_1, L_1, u_2, v_2;
	float suma = 0, sumar = 0, NCD = 0;

	xyz[0] = 0.412453;
	xyz[1] = 0.357580;
	xyz[2] = 0.180423;
	xyz[3] = 0.212671;
	xyz[4] = 0.715160;
	xyz[5] = 0.072169;
	xyz[6] = 0.019334;
	xyz[7] = 0.119193;
	xyz[8] = 0.950227;

	xyz[9] = xyz[0] + xyz[1] + xyz[2];
	xyz[10] = xyz[3] + xyz[4] + xyz[5];
	xyz[11] = xyz[6] + xyz[7] + xyz[8];

	un = (4 * xyz[9]) / (xyz[9] + 15 * xyz[10] + 3 * xyz[11]);
	vn = (9 * xyz[10]) / (xyz[9] + 15 * xyz[10] + 3 * xyz[11]);

	XYZ[0] = 0.412453;
	XYZ[1] = 0.357580;
	XYZ[2] = 0.180423;
	XYZ[3] = 0.212671;
	XYZ[4] = 0.715160;
	XYZ[5] = 0.072169;
	XYZ[6] = 0.019334;
	XYZ[7] = 0.119193;
	XYZ[8] = 0.950227;

	WPQ[0] = 0.412453;
	WPQ[1] = 0.357580;
	WPQ[2] = 0.180423;
	WPQ[3] = 0.212671;
	WPQ[4] = 0.715160;
	WPQ[5] = 0.072169;
	WPQ[6] = 0.019334;
	WPQ[7] = 0.119193;
	WPQ[8] = 0.950227;

	unsigned char *datosI1, *datosI2;
	datosI1 = (unsigned char*)(I1.data); datosI2 = (unsigned char*)(I2.data);

	for (int Row = 1; Row < I1.rows - 1; Row++) {
		for (int Col = 1; Col < I1.cols - 1; Col++) {
			XYZ[9] = XYZ[0] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[1] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[2] * datosI1[(Row * I1.rows + Col) * 3 + 0];
			XYZ[10] = XYZ[3] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[4] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[5] * datosI1[(Row * I1.rows + Col) * 3 + 0];
			XYZ[11] = XYZ[6] * datosI1[(Row * I1.rows + Col) * 3 + 2] + XYZ[7] * datosI1[(Row * I1.rows + Col) * 3 + 1] + XYZ[8] * datosI1[(Row * I1.rows + Col) * 3 + 0];
			L = XYZ[10] / xyz[10];

			if (L >= 0.008856) {
				L1 = 116 * (pow(L, 0.333)) - 16;
			}
			else {
				L1 = 903.3 * L;
			}
			if ((XYZ[9] + 15 * XYZ[10] + 3 * XYZ[11]) == 0) {
				u = 0;
				v = 0;
			}
			else {
				u = 4 * XYZ[9] / (XYZ[9] + 15 * XYZ[10] + 3 * XYZ[11]);
				v = 9 * XYZ[10] / (XYZ[9] + 15 * XYZ[10] + 3 * XYZ[11]);
			}
			ul = 13 * L1*(u - un);
			vl = 13 * L1*(v - vn);

			WPQ[9] = WPQ[0] * datosI2[(Row * I1.rows + Col) * 3 + 2] + WPQ[1] * datosI2[(Row * I1.rows + Col) * 3 + 1] + WPQ[2] * datosI2[(Row * I1.rows + Col) * 3 + 0];
			WPQ[10] = WPQ[3] * datosI2[(Row * I1.rows + Col) * 3 + 2] + WPQ[4] * datosI2[(Row * I1.rows + Col) * 3 + 1] + WPQ[5] * datosI2[(Row * I1.rows + Col) * 3 + 0];
			WPQ[11] = WPQ[6] * datosI2[(Row * I1.rows + Col) * 3 + 2] + WPQ[7] * datosI2[(Row * I1.rows + Col) * 3 + 1] + WPQ[8] * datosI2[(Row * I1.rows + Col) * 3 + 0];
			L = WPQ[10] / xyz[10];
			if (L > 0.008856) {
				L_1 = 116 * (pow(L, 0.333)) - 16;
			}
			else {
				L_1 = 903.3 * L;
			}

			if ((WPQ[9] + 15 * WPQ[10] + 3 * WPQ[11]) == 0) {
				u_1 = 0;
				v_1 = 0;
			}
			else {
				if (XYZ[9] == 0 && XYZ[10] == 0 && XYZ[11] == 0) {
					u_1 = 0;
					v_1 = 0;
				}
				else {
					u_1 = 4 * XYZ[9] / (XYZ[9] + 15 * XYZ[10] + 3 * XYZ[11]);
					v_1 = 9 * XYZ[10] / (XYZ[9] + 15 * XYZ[10] + 3 * XYZ[11]);
				}
			}
			u_2 = 13 * L_1*(u_1 - un);
			v_2 = 13 * L_1*(v_1 - vn);

			suma = sqrt(pow(L_1, 2) + pow(u_2, 2) + pow(v_2, 2)) + suma;
			sumar = sqrt(pow(L1 - L_1, 2) + pow(ul - u_2, 2) + pow(vl - v_2, 2)) + sumar;
		}
	}
	NCD = sumar / suma;
	return NCD;
}


__global__ void PeerGroup(unsigned char* d_Pout, unsigned char* d_Pin, int n, int m) {
	//Calculate the row # of the d_Pin and d_Pout element to process 
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	// Calculate the column # of the d_Pin and d_Pout element to process 
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	// each thread computes one element of d_Pout if in range 
	// Se debe de checar si los pixeles esta dentro del intervalo de 8 bits
	int x = 0, posicion[9], hold2 = 0, F = 0;
	int K = 1024;
	float vectR[9], vectG[9], vectB[9]; // si el tipo de dato es double, no ay recursos para la ejecusion)
	float disteucl, disteucl1[9], hold;
	float  valAngulo = 0.0, r = 0.0;
	float arriva = 0.0, abajo = 0.0, val1, val2, val3, dist_M = 0;


	if ((Row < m - 1) && (Col < n - 1)) {


		//hacer el arreglo
		F = 0;

		for (int i = -1; i <= 1; i++) {
			for (int j = -1; j <= 1; j++) {
				vectR[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 0];
				vectG[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 1];
				vectB[F] = d_Pin[((Row + i) * n + (Col + j)) * 3 + 2];

				posicion[F] = F;
				F++;
			}
		}


		disteucl = 0;
		for (F = 0; F <= 8; F++) {
			arriva = min(vectR[F], vectR[4]) + K;
			abajo = max(vectR[F], vectR[4]) + K;
			val1 = arriva / abajo;

			arriva = min(vectG[F], vectG[4]) + K;
			abajo = max(vectG[F], vectG[4]) + K;
			val2 = arriva / abajo;

			arriva = min(vectB[F], vectB[4]) + K;
			abajo = max(vectB[F], vectB[4]) + K;
			val3 = arriva / abajo;
			dist_M = min(min(val1, val2), val3);

		}

		for (F = 0; F <= 8; F++) {
			for (x = 0; x <= 7; x++) {
				if (disteucl1[x] > disteucl1[x + 1]) {
					hold = disteucl1[x];
					hold2 = posicion[x];
					disteucl1[x] = disteucl1[x + 1];
					posicion[x] = posicion[x + 1];
					disteucl1[x + 1] = hold;
					posicion[x + 1] = hold2;
				}
			}
		}



		d_Pout[(Row * n + Col) * 3 + 0] = vectR[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 1] = vectG[posicion[0]];
		d_Pout[(Row * n + Col) * 3 + 2] = vectB[posicion[0]];

	}



}


int main()
{


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int msecMenor = 10000;
	float msec = 0, sumaT = 0;
	int nExperimentos = 60;
	double valPSNR[100], valMCRE[100], valMCREMio[100], valNCD[100], valMAE[100];



	Mat imageOriginal = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512/0.bmp", IMREAD_UNCHANGED);
	//Mat imageOriginal = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512/0.bmp", IMREAD_UNCHANGED);
	//Mat imageOriginal = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512_Aleatorio/0.bmp", IMREAD_UNCHANGED);

	//Mat imageOriginal = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/caps_768x512.png", IMREAD_UNCHANGED);
	//Mat imageOriginal = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena320x320_Aleatorio/0.bmp", IMREAD_UNCHANGED);
	if (!imageOriginal.data)          // Check for invalid input
	{
		cout << "No esta la imagen1" << std::endl;
		_getch();
		return -1;
	}

	int size = (N)*(M)* sizeof(unsigned char)* nChannels;

	//Se usa malloc para poder procesar imagenes grandes
	unsigned char *h_in;
	//h_in = (unsigned char *)malloc(size);
	//h_in = (unsigned char*)(imageOriginal.data);					// puntero a los datos de la imagenIn

	Mat imagenOut(N, M, CV_8UC3, Scalar(255));
	unsigned char *h_out;	h_out = (unsigned char *)malloc(size);
	imagenOut.data = h_out;

	//Noise
	Mat imagenOut_Noise(N, M, CV_8UC1, Scalar(255));
	unsigned char *h_out_Noise;	h_out_Noise = (unsigned char *)malloc(size);
	imagenOut_Noise.data = h_out_Noise;

	imshow("imagen de Original", imageOriginal);

	//obtencion de imagen de prueba e imagen de salida y puntero
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512_Aleatorio/0.bmp", CV_LOAD_IMAGE_UNCHANGED);

	char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512/";
	//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512_Aleatorio/";
	//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena320x320_Aleatorio/";
	//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512/";
	
	//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512_Aleatorio/";
	ObtenerPath(Dir, 10);

	
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/caps_10SyP.png", CV_LOAD_IMAGE_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	Mat image = imread(Dir, IMREAD_UNCHANGED); if (!image.data) { cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }

	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena1024x1024_Ruido10.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }

	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena2048x2048.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena2048x2048_Ruido10.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena4096x4096_Ruido10.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena8192x8192_Ruido10.bmp", IMREAD_UNCHANGED); if (!image.data) { cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena320x320_Ruido10.bmp", IMREAD_UNCHANGED); if (!image.data) { cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }


	h_in = (unsigned char*)(image.data);
	//
	unsigned char *dev_in, *dev_out, *Noise;

	if (hipSuccess != hipMalloc((void **)&dev_in, size))
	{
		printf("Error en hipMalloc!\n");		_getch();
	}

	if (hipSuccess != hipMalloc((void **)&dev_out, size))
	{
		printf("Error en hipMalloc!\n");		_getch();
	}
	if (hipSuccess != hipMalloc((void **)&Noise, size))
	{
		printf("Error en hipMalloc!\n");		_getch();
	}
	

	//int nHilosporBloque = 4;
	//int nHilosporBloque = 8;//Tenia este
	int nHilosporBloque = 16;//con este funciona
	//int nHilosporBloque = 32;
	//int nHilosporBloque = 64;
	
	dim3 nThreads(nHilosporBloque, nHilosporBloque, 1);		// numeros de Hilos por bloque  (se selecciono asi aqui, tiene que ser un multiplo de 32)
	dim3 nBloques((M / nHilosporBloque) + 1, (N / nHilosporBloque) + 1, 1);

	//Copiar datos de Host a Device
	if (hipSuccess != hipMemcpy(dev_in, h_in, size, hipMemcpyHostToDevice))
	{
		printf("Error!\n");		_getch();
	}

	//image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512/10.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen2" << std::endl; _getch(); return -1; }
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena2048x2048.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena1024x1024.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
	//image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena2048x2048.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }	
	//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/caps_20SyP.png", IMREAD_UNCHANGED);

	//h_in = (unsigned char *)malloc(size);
	//h_in = (unsigned char*)(image.data);

	/*
	if (hipSuccess != hipMemcpy(dev_in, h_in, size, hipMemcpyHostToDevice))
	{
	printf("Error!\n");		_getch();
	}
	*/

	//MarginalMedianFilter_Global_Forgetfull << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//VMF_Global_Forgetfull_Reuse << <nBloques, nThreads >> >(dev_out, dev_in, N,M );

	//Detection_FuzzyMetric << <nBloques, nThreads >> >(Noise, dev_in, N, M);
	//Detection_Euclidean << <nBloques, nThreads >> >(Noise, dev_in, N, M);
	//AMF_Filtering <<<nBloques, nThreads >>>(dev_out, dev_in, Noise, N, M);
	//VMF_Filtering << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
	
	//FiltradoPropuesta << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
	//VMF_GPU_GLOBAL << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//FTSCF_GPU << <nBloques, nThreads >> >(dev_out, dev_in,10,60,.5, M, N);

	//FiltradoPropuesta_MMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//FiltradoPropuesta_VMF   <<<nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//FiltradoPropuesta_AMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	VectorUnit_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	

	//FTSCF_GPU_Original << <nBloques, nThreads >> >(dev_out, dev_in, N, M);

	BVDF_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	
	
	if (hipSuccess != hipMemcpy(h_out, dev_out, size, hipMemcpyDeviceToHost))
	{
	printf("Error en copiar de Device a host!\n");	_getch();
	}

	imagenOut.data = h_out;
	imshow("Imagen Filtrada", imagenOut); waitKey();

	//Noise
	if (hipSuccess != hipMemcpy(h_out_Noise, Noise, size, hipMemcpyDeviceToHost))
	{
	printf("Error en copiar de Device a host!\n");	_getch();
	}

	imagenOut_Noise.data = h_out_Noise;
	
	/*
	/////////tiempo de ejecucion
	for (int contador = 0; contador <= nExperimentos; contador++){

	hipEventRecord(start);

	//Detection_FuzzyMetric << <nBloques, nThreads >> >(Noise, dev_in, N, M);
	//VMF_Filtering << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
	//AMF_Filtering << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
	//FiltradoPropuesta << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
	//FiltradoPropuesta2 << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//MarginalMedianFilter_Global_Forgetfull << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//VMF_Global_Forgetfull_Reuse << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//FiltradoPropuesta_MMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//VMF_GPU_GLOBAL << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
	//FTSCF_GPU << <nBloques, nThreads >> >(dev_out, dev_in, 0, 120, .5, M, N);

	VectorUnit_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//BVDF_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	
	//FTSCF_GPU_Original << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//FiltradoPropuesta_MMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//FiltradoPropuesta_VMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
	//FiltradoPropuesta_AMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	//if (milliseconds<msecMenor) msecMenor = milliseconds;
	printf("\n%f",milliseconds);

	}
	//printf("         Time taken %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	//printf("\n%f",msecMenor);

	if ( hipSuccess != hipMemcpy(h_out, dev_out, size, hipMemcpyDeviceToHost) )
	{printf( "Error en copiar de Device a host!\n" );	_getch();	}
	
	
	//Esto es para escribir imagenes en disco
	vector<int> compression_params;
	compression_params.push_back(IMWRITE_PNG_COMPRESSION);
	compression_params.push_back(9);
	
	
	*/
	
	// Obtencion de PSNR
	for (int contador = 0; contador <= nExperimentos; contador++) {

		char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512/";
		//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena512x512_Aleatorio/";
		//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512/";
		//char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Mandrill512x512_Aleatorio/";
		ObtenerPath(Dir, contador);
		Mat image = imread(Dir, IMREAD_UNCHANGED); if (!image.data) { cout << "No esta la imagen2" << std::endl; _getch(); return -1; }
		//Mat image = imread("C:/Users/AgustinQuadro4000/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena2048x2048.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }
		h_in = (unsigned char *)malloc(size);
		h_in = (unsigned char*)(image.data);
		if (hipSuccess != hipMemcpy(dev_in, h_in, size, hipMemcpyHostToDevice))
		{
			printf("Error!\n");		_getch();
		}

		//Detection_FuzzyMetric << <nBloques, nThreads >> >(Noise, dev_in, N, M);
		//Detection_Euclidean << <nBloques, nThreads >> >(Noise, dev_in, N, M);
		//AMF_Filtering <<<nBloques, nThreads >>>(dev_out, dev_in, Noise, N, M);
		//VMF_Filtering << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
		//FiltradoPropuesta << <nBloques, nThreads >> >(dev_out, dev_in, Noise, N, M);
		//FiltradoPropuesta2 << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
		
		//MarginalMedianFilter_Global_Forgetfull << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
		//VMF_Global_Forgetfull_Reuse << <nBloques, nThreads >> >(dev_out, dev_in, M, N);

		//FiltradoPropuesta_MMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
		//FTSCF_GPU_Original << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
		//FiltradoPropuesta_VMF   <<<nBloques, nThreads >> >(dev_out, dev_in, N, M);
		//FiltradoPropuesta_AMF << <nBloques, nThreads >> >(dev_out, dev_in, N, M);

		//FTSCF_GPU << <nBloques, nThreads >> >(dev_out, dev_in, 0, 120, .5, M, N);
		
		//FTSCF_GPU_Original << <nBloques, nThreads >> >(dev_out, dev_in, M, N);
		//FTSCF_GPU_Original_Params << <nBloques, nThreads >> >(dev_out, dev_in, M, N, 1, .8, .1, 60, 10, 1000);

		VectorUnit_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);
		//BVDF_GPU_Global << <nBloques, nThreads >> >(dev_out, dev_in, N, M);

		if (hipSuccess != hipMemcpy(h_out, dev_out, size, hipMemcpyDeviceToHost))
		{
			printf("Error en copiar de Device a host!\n");	_getch();
		}

		imagenOut.data = h_out;


		//Escribir una a una las imagenes resultantes

		//char Dir2[] = "D:/Google Drive/Trabajo Doctorado/Resultados/Imagenes Filtradas/SSIM_VSNR/PropuestaPeerGroup_Mandril512x512_RA/";
		//ObtenerPathWrite(Dir2, contador);
		//imwrite(Dir2, imagenOut, compression_params);


		valPSNR[contador] = getPSNR(imageOriginal, imagenOut);
		valMCRE[contador] = getMCRE(imageOriginal, imagenOut);
		//valMCRE[contador] = getMCRE_Mio(imageOriginal, imagenOut);
		valNCD[contador] = getNCD(imageOriginal, imagenOut);
		valMAE[contador] = getMAE(imageOriginal, imagenOut);
		//valMCREMio[contador] = getMCRE_Mio(imageOriginal, imagenOut);
		printf("%d\n", contador);

	}
	

	

	/*
	//valores optimos

	float med_1, var_1, med_2, med1, med2, var1, THS=0;
	float med_1_MAX, var_1_MAX, med_2_MAX, med1_MAX, med2_MAX, var1_MAX, THS_MAX;

	int THS_contador = 0, int contador =0;

	float valorPSNR[60];
	float MAX_PSNR = 0, Sum_PSNR = 0, Mejor_Sum_PSNR =0;
	int Iteracion = 0;
	
	unsigned char *h_inOpt; 
	unsigned char *h_outOpt;

	for (med_1 = 0.2; med_1 <= 1; med_1= med_1 + .05) {
		for (var_1 = 0.2; var_1 <= .8; var_1= var_1+ .05) {
			for (med_2 = 0.2; med_2 <= .8; med_2=med_2 + .05) {
				for (med1 = 10; med1 <= 150; med1++) {
					for (med2 = 10; med2 <= 150; med2++){
						for (var1 = 500; var1 <= 1000; var1 = var1 + 10){
							for (THS = 0; THS <= .5; THS=THS+0.1){
								printf("Iteracion=%d\n", Iteracion++);
								for (contador = 0; contador <= 60; contador++){
									Mat imagenOutOpt(N, M, CV_8UC3, Scalar(255));

									//unsigned char *h_inOpt = new(nothrow) unsigned char[ N * M * nChannels ];
									//unsigned char *h_outOpt = new(nothrow) unsigned char[N * M * nChannels];

									char Dir[] = "D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/Lena320x320_Aleatorio/";
									ObtenerPath(Dir, contador);
									Mat imageOpt = imread(Dir, IMREAD_UNCHANGED); if (!imageOpt.data){ cout << "No esta la imagen2" << std::endl; _getch(); return -1; }
									//Mat image = imread("D:/Google Drive/Trabajo Doctorado/VisualStudio y MAtlab/Matlab/lena2048x2048.bmp", IMREAD_UNCHANGED); if (!image.data){ cout << "No esta la imagen Dir11" << std::endl; _getch(); return -1; }

									h_inOpt = (unsigned char*)(imageOpt.data);
									h_outOpt = (unsigned char *)imagenOutOpt.data;
									
									if (hipSuccess != hipMemcpy(dev_in, h_inOpt, size, hipMemcpyHostToDevice)){	printf("Error!\n");		_getch();}
									imageOpt.release();

									FTSCF_GPU_Original_Params << <nBloques, nThreads >> >(dev_out, dev_in, M, N, med_1, var_1, med_2, med1, med2, var1, THS);
		
									if (hipSuccess != hipMemcpy(h_outOpt, dev_out, size, hipMemcpyDeviceToHost)){	printf("Error en copiar de Device a host!\n");	_getch();}
		
									valorPSNR[contador] = getPSNR(imageOriginal, imagenOutOpt);

									imageOpt.release();
									imagenOutOpt.release();

									//delete[] h_inOpt;
									//delete[] h_outOpt;

									Sum_PSNR = valorPSNR[contador] + Sum_PSNR;
									//printf("valorPSNR[%d] = %f\n", contador, valorPSNR[contador]);
																	
								}//Contador
								if (Sum_PSNR > Mejor_Sum_PSNR) {
									med_1_MAX = med_1;
									var_1_MAX = var_1;
									med_2_MAX = med_2;
									med1_MAX = med1;
									med2_MAX = med2;
									var1_MAX = var1;
									THS_MAX = THS;
						
									Mejor_Sum_PSNR = Sum_PSNR;

									printf("med_1=%f\n", med_1);
									printf("var_1=%f\n", var_1);
									printf("med_2=%f\n", med_2);
									printf("med1=%f\n", med1);
									printf("med2=%f\n", med2);
									printf("var1=%f\n", var1);

									printf("THS=%f  ", THS);

									printf("Sum_PSNR=%f\n", Sum_PSNR);
								}//if impresion
								Sum_PSNR = 0;
							}//THS
						}//var1
					}//med2
				}//med1
			}//med_2
		}//var_1
	}//med_1

	*/
	imagenOut.data = h_out;
	imshow("Imagen Ruido", image);
	imshow("Imagen Filtrada", imagenOut);

	//Noise
	imagenOut_Noise.data = h_out_Noise;

	imshow("Imagen Noise", imagenOut_Noise);	
	waitKey(0);

	//vector<int> compression_params;
	//compression_params.push_back(IMWRITE_PNG_COMPRESSION);
	//compression_params.push_back(9);
	//	imwrite( "C:/Users/AgustinTortolero/Google Drive/Trabajo Doctorado/Resultados/Imagenes Filtradas/LenaPropuestaPeerMMF_SyP_05.bmp",imagenOut, compression_params );
	//imwrite("D:/Google Drive/Trabajo Doctorado/Resultados/Imagenes Filtradas/Noise_Lena05.bmp", imagenOut_Noise, compression_params);
	///////////




	EscribirCriterios(valPSNR, valMCRE, valNCD, valMAE, nExperimentos);
	hipFree(dev_in);		hipFree(dev_out);
	//hipDeviceReset();

	free(h_in);				free(h_out);



	return 0;
}

